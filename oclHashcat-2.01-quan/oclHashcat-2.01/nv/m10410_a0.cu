#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _MD5_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 0
#define DGST_R1 1
#define DGST_R2 2
#define DGST_R3 3

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"
#include "include/rp_gpu.h"
#include "rp_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_S "check_single_vect1_comp4.c"
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_S "check_single_vect2_comp4.c"
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

__device__ __constant__ u32 padding[8] =
{
  0x5e4ebf28,
  0x418a754e,
  0x564e0064,
  0x0801faff,
  0xb6002e2e,
  0x803e68d0,
  0xfea90c2f,
  0x7a695364
};

typedef struct
{
  u8 S[256];

  u32 wtf_its_faster;

} RC4_KEY;

__device__ static void swap (RC4_KEY *rc4_key, const u8 i, const u8 j)
{
  u8 tmp;

  tmp           = rc4_key->S[i];
  rc4_key->S[i] = rc4_key->S[j];
  rc4_key->S[j] = tmp;
}

__device__ static void rc4_init_16 (RC4_KEY *rc4_key, const u32 data[4])
{
  u32 v = 0x03020100;
  u32 a = 0x04040404;

  u32 *ptr = (u32 *) rc4_key->S;

  #pragma unroll 64
  for (u32 i = 0; i < 64; i++)
  {
    *ptr++ = v; v += a;
  }

  const u32 d0 = data[0] >>  0;
  const u32 d1 = data[0] >>  8;
  const u32 d2 = data[0] >> 16;
  const u32 d3 = data[0] >> 24;
  const u32 d4 = data[1] >>  0;

  u32 i = 0;
  u32 j = 0;

  #pragma unroll 52
  for (i = 0; i < 255; i += 5)
  {
    j += rc4_key->S[i + 0]; j += d0; swap (rc4_key, i + 0, j);
    j += rc4_key->S[i + 1]; j += d1; swap (rc4_key, i + 1, j);
    j += rc4_key->S[i + 2]; j += d2; swap (rc4_key, i + 2, j);
    j += rc4_key->S[i + 3]; j += d3; swap (rc4_key, i + 3, j);
    j += rc4_key->S[i + 4]; j += d4; swap (rc4_key, i + 4, j);
  }

  j += rc4_key->S[i + 0]; j += d0; swap (rc4_key, i + 0, j);
}

__device__ static u8 rc4_next_16 (RC4_KEY *rc4_key, u8 i, u8 j, const u32 in[4], u32 out[4])
{
  for (u32 k = 0; k < 4; k++)
  {
    u32 xor4 = 0;

    u8 idx;

    i += 1;
    j += rc4_key->S[i];

    swap (rc4_key, i, j);

    idx = rc4_key->S[i] + rc4_key->S[j];

    xor4 |= rc4_key->S[idx] <<  0;

    i += 1;
    j += rc4_key->S[i];

    swap (rc4_key, i, j);

    idx = rc4_key->S[i] + rc4_key->S[j];

    xor4 |= rc4_key->S[idx] <<  8;

    i += 1;
    j += rc4_key->S[i];

    swap (rc4_key, i, j);

    idx = rc4_key->S[i] + rc4_key->S[j];

    xor4 |= rc4_key->S[idx] << 16;

    i += 1;
    j += rc4_key->S[i];

    swap (rc4_key, i, j);

    idx = rc4_key->S[i] + rc4_key->S[j];

    xor4 |= rc4_key->S[idx] << 24;

    out[k] = in[k] ^ xor4;
  }

  return j;
}

__device__ __constant__ gpu_rule_t c_rules[1024];

extern "C" __global__ void __launch_bounds__ (64, 1) m10410_m04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const pdf_t *pdf_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x pw_buf0[4];

  pw_buf0[0] = pws[gid].i[ 0];
  pw_buf0[1] = pws[gid].i[ 1];
  pw_buf0[2] = pws[gid].i[ 2];
  pw_buf0[3] = pws[gid].i[ 3];

  u32x pw_buf1[4];

  pw_buf1[0] = pws[gid].i[ 4];
  pw_buf1[1] = pws[gid].i[ 5];
  pw_buf1[2] = pws[gid].i[ 6];
  pw_buf1[3] = pws[gid].i[ 7];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * key
   */

  __shared__ RC4_KEY rc4_keys[64];
  RC4_KEY *rc4_key = &rc4_keys[lid];

  /**
   * loop
   */

  for (u32 il_pos = 0; il_pos < rules_cnt; il_pos++)
  {
    u32x w0[4];

    w0[0] = pw_buf0[0];
    w0[1] = pw_buf0[1];
    w0[2] = pw_buf0[2];
    w0[3] = pw_buf0[3];

    u32x w1[4];

    w1[0] = pw_buf1[0];
    w1[1] = pw_buf1[1];
    w1[2] = pw_buf1[2];
    w1[3] = pw_buf1[3];

    u32x w2[4];

    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = 0;

    const u32 out_len = apply_rules (c_rules[il_pos].cmds, w0, w1, pw_len);

    // now the RC4 part

    u32x key[4];

    key[0] = w0[0];
    key[1] = w0[1];
    key[2] = 0;
    key[3] = 0;

    rc4_init_16 (rc4_key, key);

    u32x out[4];

    rc4_next_16 (rc4_key, 0, 0, padding, out);

    const u32x r0 = out[0];
    const u32x r1 = out[1];
    const u32x r2 = out[2];
    const u32x r3 = out[3];

    #include VECT_COMPARE_M
  }
}

extern "C" __global__ void __launch_bounds__ (64, 1) m10410_m08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const pdf_t *pdf_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (64, 1) m10410_m16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const pdf_t *pdf_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (64, 1) m10410_s04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const pdf_t *pdf_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x pw_buf0[4];

  pw_buf0[0] = pws[gid].i[ 0];
  pw_buf0[1] = pws[gid].i[ 1];
  pw_buf0[2] = pws[gid].i[ 2];
  pw_buf0[3] = pws[gid].i[ 3];

  u32x pw_buf1[4];

  pw_buf1[0] = pws[gid].i[ 4];
  pw_buf1[1] = pws[gid].i[ 5];
  pw_buf1[2] = pws[gid].i[ 6];
  pw_buf1[3] = pws[gid].i[ 7];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * digest
   */

  const u32 search[4] =
  {
    digests_buf[digests_offset].digest_buf[DGST_R0],
    digests_buf[digests_offset].digest_buf[DGST_R1],
    digests_buf[digests_offset].digest_buf[DGST_R2],
    digests_buf[digests_offset].digest_buf[DGST_R3]
  };

  /**
   * key
   */

  __shared__ RC4_KEY rc4_keys[64];
  RC4_KEY *rc4_key = &rc4_keys[lid];

  /**
   * loop
   */

  for (u32 il_pos = 0; il_pos < rules_cnt; il_pos++)
  {
    u32x w0[4];

    w0[0] = pw_buf0[0];
    w0[1] = pw_buf0[1];
    w0[2] = pw_buf0[2];
    w0[3] = pw_buf0[3];

    u32x w1[4];

    w1[0] = pw_buf1[0];
    w1[1] = pw_buf1[1];
    w1[2] = pw_buf1[2];
    w1[3] = pw_buf1[3];

    u32x w2[4];

    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = 0;

    const u32 out_len = apply_rules (c_rules[il_pos].cmds, w0, w1, pw_len);

    // now the RC4 part

    u32x key[4];

    key[0] = w0[0];
    key[1] = w0[1];
    key[2] = 0;
    key[3] = 0;

    rc4_init_16 (rc4_key, key);

    u32x out[4];

    rc4_next_16 (rc4_key, 0, 0, padding, out);

    const u32x r0 = out[0];
    const u32x r1 = out[1];
    const u32x r2 = out[2];
    const u32x r3 = out[3];

    #include VECT_COMPARE_S
  }
}

extern "C" __global__ void __launch_bounds__ (64, 1) m10410_s08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const pdf_t *pdf_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (64, 1) m10410_s16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const pdf_t *pdf_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}
