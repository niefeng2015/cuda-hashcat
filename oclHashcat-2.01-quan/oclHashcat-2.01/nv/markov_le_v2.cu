#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define CHARSIZ 256

#define VECT_SIZE2

#include "types_nv.c"

__device__ static void generate_pw (u32 pw_buf[16], cs_t *root_css_buf, cs_t *markov_css_buf, const u32 pw_l_len, const u32 pw_r_len, const u32 mask80, const u32 bits14, const u32 bits15, u64 val)
{
  pw_buf[ 0] = 0;
  pw_buf[ 1] = 0;
  pw_buf[ 2] = 0;
  pw_buf[ 3] = 0;
  pw_buf[ 4] = 0;
  pw_buf[ 5] = 0;
  pw_buf[ 6] = 0;
  pw_buf[ 7] = 0;
  pw_buf[ 8] = 0;
  pw_buf[ 9] = 0;
  pw_buf[10] = 0;
  pw_buf[11] = 0;
  pw_buf[12] = 0;
  pw_buf[13] = 0;
  pw_buf[14] = 0;
  pw_buf[15] = 0;

  cs_t *cs = &root_css_buf[pw_r_len];

  u32 i;
  u32 j;

  for (i = 0, j = pw_r_len; i < pw_l_len; i++, j++)
  {
    const u32 len = cs->cs_len;

    const u64 next = val / len;
    const u64 pos  = val % len;

    val = next;

    const u32 key = cs->cs_buf[pos];

    const u32 jd4 = j / 4;
    const u32 jm4 = j % 4;

    pw_buf[jd4] |= key << (jm4 * 8);

    cs = &markov_css_buf[(j * CHARSIZ) + key];
  }

  const u32 jd4 = j / 4;
  const u32 jm4 = j % 4;

  pw_buf[jd4] |= (0xff << (jm4 * 8)) & mask80;

  if (bits14) pw_buf[14] = (pw_l_len + pw_r_len) * 8;
  if (bits15) pw_buf[15] = (pw_l_len + pw_r_len) * 8;
}

extern "C" __global__ void __launch_bounds__ (256, 1) l_markov (pw_t *pws_buf_l, cs_t *root_css_buf, cs_t *markov_css_buf, const u64 off, const u32 pw_l_len, const u32 pw_r_len, const u32 mask80, const u32 bits14, const u32 bits15, const u32 gid_max)
{
  const u32 gid  = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  const u32 gid2 = gid * 2;

  u32 pw_buf0[16];
  u32 pw_buf1[16];

  generate_pw (pw_buf0, root_css_buf, markov_css_buf, pw_l_len, pw_r_len, mask80, bits14, bits15, off + gid2 + 0);
  generate_pw (pw_buf1, root_css_buf, markov_css_buf, pw_l_len, pw_r_len, mask80, bits14, bits15, off + gid2 + 1);

  #pragma unroll 16
  for (int i = 0; i < 16; i++)
  {
    pws_buf_l[gid].i[i].x = pw_buf0[i];
    pws_buf_l[gid].i[i].y = pw_buf1[i];
  }

  pws_buf_l[gid].pw_len = pw_l_len + pw_r_len;
}

extern "C" __global__ void __launch_bounds__ (256, 1) r_markov (bf_t *pws_buf_r, cs_t *root_css_buf, cs_t *markov_css_buf, const u64 off, const u32 pw_r_len, const u32 mask80, const u32 bits14, const u32 bits15, const u32 gid_max)
{
  const u32 gid  = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  const u32 gid2 = gid * 2;

  u32 pw_buf[16];

  generate_pw (pw_buf, root_css_buf, markov_css_buf, pw_r_len, 0, 0, 0, 0, off + gid2 + 0);

  pws_buf_r[gid2 + 0].i = pw_buf[0];

  generate_pw (pw_buf, root_css_buf, markov_css_buf, pw_r_len, 0, 0, 0, 0, off + gid2 + 1);

  pws_buf_r[gid2 + 1].i = pw_buf[0];
}

extern "C" __global__ void __launch_bounds__ (256, 1) C_markov (comb_t *pws_buf, cs_t *root_css_buf, cs_t *markov_css_buf, const u64 off, const u32 pw_len, const u32 mask80, const u32 bits14, const u32 bits15, const u32 gid_max)
{
  const u32 gid  = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  const u32 gid2 = gid * 2;

  u32 pw_buf[16];

  generate_pw (pw_buf, root_css_buf, markov_css_buf, pw_len, 0, mask80, bits14, bits15, off + gid2 + 0);

  pws_buf[gid2 + 0].i[ 0] = pw_buf[ 0];
  pws_buf[gid2 + 0].i[ 1] = pw_buf[ 1];
  pws_buf[gid2 + 0].i[ 2] = pw_buf[ 2];
  pws_buf[gid2 + 0].i[ 3] = pw_buf[ 3];
  pws_buf[gid2 + 0].i[ 4] = pw_buf[ 4];
  pws_buf[gid2 + 0].i[ 5] = pw_buf[ 5];
  pws_buf[gid2 + 0].i[ 6] = pw_buf[ 6];
  pws_buf[gid2 + 0].i[ 7] = pw_buf[ 7];

  pws_buf[gid2 + 0].pw_len = pw_len;

  generate_pw (pw_buf, root_css_buf, markov_css_buf, pw_len, 0, mask80, bits14, bits15, off + gid2 + 1);

  pws_buf[gid2 + 1].i[ 0] = pw_buf[ 0];
  pws_buf[gid2 + 1].i[ 1] = pw_buf[ 1];
  pws_buf[gid2 + 1].i[ 2] = pw_buf[ 2];
  pws_buf[gid2 + 1].i[ 3] = pw_buf[ 3];
  pws_buf[gid2 + 1].i[ 4] = pw_buf[ 4];
  pws_buf[gid2 + 1].i[ 5] = pw_buf[ 5];
  pws_buf[gid2 + 1].i[ 6] = pw_buf[ 6];
  pws_buf[gid2 + 1].i[ 7] = pw_buf[ 7];

  pws_buf[gid2 + 1].pw_len = pw_len;
}
