#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _SHA256_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 0
#define DGST_R1 1
#define DGST_R2 2
#define DGST_R3 3

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

__device__ static void sha256_transform (const u32x w0[4], const u32x w1[4], const u32x w2[4], const u32x w3[4], u32x digest[8])
{
  u32x a = digest[0];
  u32x b = digest[1];
  u32x c = digest[2];
  u32x d = digest[3];
  u32x e = digest[4];
  u32x f = digest[5];
  u32x g = digest[6];
  u32x h = digest[7];

  u32x w0_t = w0[0];
  u32x w1_t = w0[1];
  u32x w2_t = w0[2];
  u32x w3_t = w0[3];
  u32x w4_t = w1[0];
  u32x w5_t = w1[1];
  u32x w6_t = w1[2];
  u32x w7_t = w1[3];
  u32x w8_t = w2[0];
  u32x w9_t = w2[1];
  u32x wa_t = w2[2];
  u32x wb_t = w2[3];
  u32x wc_t = w3[0];
  u32x wd_t = w3[1];
  u32x we_t = w3[2];
  u32x wf_t = w3[3];

  SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C00);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C01);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C02);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C03);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C04);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C05);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C06);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C07);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C08);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C09);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C0a);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C0b);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C0c);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C0d);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C0e);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C0f);
  w0_t = SHA256_S1(we_t) + w9_t + SHA256_S0(w1_t) + w0_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C10);
  w1_t = SHA256_S1(wf_t) + wa_t + SHA256_S0(w2_t) + w1_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C11);
  w2_t = SHA256_S1(w0_t) + wb_t + SHA256_S0(w3_t) + w2_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C12);
  w3_t = SHA256_S1(w1_t) + wc_t + SHA256_S0(w4_t) + w3_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C13);
  w4_t = SHA256_S1(w2_t) + wd_t + SHA256_S0(w5_t) + w4_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C14);
  w5_t = SHA256_S1(w3_t) + we_t + SHA256_S0(w6_t) + w5_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C15);
  w6_t = SHA256_S1(w4_t) + wf_t + SHA256_S0(w7_t) + w6_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C16);
  w7_t = SHA256_S1(w5_t) + w0_t + SHA256_S0(w8_t) + w7_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C17);
  w8_t = SHA256_S1(w6_t) + w1_t + SHA256_S0(w9_t) + w8_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C18);
  w9_t = SHA256_S1(w7_t) + w2_t + SHA256_S0(wa_t) + w9_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C19);
  wa_t = SHA256_S1(w8_t) + w3_t + SHA256_S0(wb_t) + wa_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C1a);
  wb_t = SHA256_S1(w9_t) + w4_t + SHA256_S0(wc_t) + wb_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C1b);
  wc_t = SHA256_S1(wa_t) + w5_t + SHA256_S0(wd_t) + wc_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C1c);
  wd_t = SHA256_S1(wb_t) + w6_t + SHA256_S0(we_t) + wd_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C1d);
  we_t = SHA256_S1(wc_t) + w7_t + SHA256_S0(wf_t) + we_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C1e);
  wf_t = SHA256_S1(wd_t) + w8_t + SHA256_S0(w0_t) + wf_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C1f);
  w0_t = SHA256_S1(we_t) + w9_t + SHA256_S0(w1_t) + w0_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C20);
  w1_t = SHA256_S1(wf_t) + wa_t + SHA256_S0(w2_t) + w1_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C21);
  w2_t = SHA256_S1(w0_t) + wb_t + SHA256_S0(w3_t) + w2_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C22);
  w3_t = SHA256_S1(w1_t) + wc_t + SHA256_S0(w4_t) + w3_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C23);
  w4_t = SHA256_S1(w2_t) + wd_t + SHA256_S0(w5_t) + w4_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C24);
  w5_t = SHA256_S1(w3_t) + we_t + SHA256_S0(w6_t) + w5_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C25);
  w6_t = SHA256_S1(w4_t) + wf_t + SHA256_S0(w7_t) + w6_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C26);
  w7_t = SHA256_S1(w5_t) + w0_t + SHA256_S0(w8_t) + w7_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C27);
  w8_t = SHA256_S1(w6_t) + w1_t + SHA256_S0(w9_t) + w8_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C28);
  w9_t = SHA256_S1(w7_t) + w2_t + SHA256_S0(wa_t) + w9_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C29);
  wa_t = SHA256_S1(w8_t) + w3_t + SHA256_S0(wb_t) + wa_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C2a);
  wb_t = SHA256_S1(w9_t) + w4_t + SHA256_S0(wc_t) + wb_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C2b);
  wc_t = SHA256_S1(wa_t) + w5_t + SHA256_S0(wd_t) + wc_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C2c);
  wd_t = SHA256_S1(wb_t) + w6_t + SHA256_S0(we_t) + wd_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C2d);
  we_t = SHA256_S1(wc_t) + w7_t + SHA256_S0(wf_t) + we_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C2e);
  wf_t = SHA256_S1(wd_t) + w8_t + SHA256_S0(w0_t) + wf_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C2f);
  w0_t = SHA256_S1(we_t) + w9_t + SHA256_S0(w1_t) + w0_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C30);
  w1_t = SHA256_S1(wf_t) + wa_t + SHA256_S0(w2_t) + w1_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C31);
  w2_t = SHA256_S1(w0_t) + wb_t + SHA256_S0(w3_t) + w2_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C32);
  w3_t = SHA256_S1(w1_t) + wc_t + SHA256_S0(w4_t) + w3_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C33);
  w4_t = SHA256_S1(w2_t) + wd_t + SHA256_S0(w5_t) + w4_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C34);
  w5_t = SHA256_S1(w3_t) + we_t + SHA256_S0(w6_t) + w5_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C35);
  w6_t = SHA256_S1(w4_t) + wf_t + SHA256_S0(w7_t) + w6_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C36);
  w7_t = SHA256_S1(w5_t) + w0_t + SHA256_S0(w8_t) + w7_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C37);
  w8_t = SHA256_S1(w6_t) + w1_t + SHA256_S0(w9_t) + w8_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C38);
  w9_t = SHA256_S1(w7_t) + w2_t + SHA256_S0(wa_t) + w9_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C39);
  wa_t = SHA256_S1(w8_t) + w3_t + SHA256_S0(wb_t) + wa_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C3a);
  wb_t = SHA256_S1(w9_t) + w4_t + SHA256_S0(wc_t) + wb_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C3b);
  wc_t = SHA256_S1(wa_t) + w5_t + SHA256_S0(wd_t) + wc_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C3c);
  wd_t = SHA256_S1(wb_t) + w6_t + SHA256_S0(we_t) + wd_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C3d);
  we_t = SHA256_S1(wc_t) + w7_t + SHA256_S0(wf_t) + we_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C3e);
  wf_t = SHA256_S1(wd_t) + w8_t + SHA256_S0(w0_t) + wf_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C3f);

  digest[0] += a;
  digest[1] += b;
  digest[2] += c;
  digest[3] += d;
  digest[4] += e;
  digest[5] += f;
  digest[6] += g;
  digest[7] += h;
}

__device__ static void hmac_sha256_pad (u32x w0[4], u32x w1[4], u32x w2[4], u32x w3[4], u32x ipad[8], u32x opad[8])
{
  w0[0] = w0[0] ^ 0x36363636;
  w0[1] = w0[1] ^ 0x36363636;
  w0[2] = w0[2] ^ 0x36363636;
  w0[3] = w0[3] ^ 0x36363636;
  w1[0] = w1[0] ^ 0x36363636;
  w1[1] = w1[1] ^ 0x36363636;
  w1[2] = w1[2] ^ 0x36363636;
  w1[3] = w1[3] ^ 0x36363636;
  w2[0] = w2[0] ^ 0x36363636;
  w2[1] = w2[1] ^ 0x36363636;
  w2[2] = w2[2] ^ 0x36363636;
  w2[3] = w2[3] ^ 0x36363636;
  w3[0] = w3[0] ^ 0x36363636;
  w3[1] = w3[1] ^ 0x36363636;
  w3[2] = w3[2] ^ 0x36363636;
  w3[3] = w3[3] ^ 0x36363636;

  ipad[0] = SHA256M_A;
  ipad[1] = SHA256M_B;
  ipad[2] = SHA256M_C;
  ipad[3] = SHA256M_D;
  ipad[4] = SHA256M_E;
  ipad[5] = SHA256M_F;
  ipad[6] = SHA256M_G;
  ipad[7] = SHA256M_H;

  sha256_transform (w0, w1, w2, w3, ipad);

  w0[0] = w0[0] ^ 0x6a6a6a6a;
  w0[1] = w0[1] ^ 0x6a6a6a6a;
  w0[2] = w0[2] ^ 0x6a6a6a6a;
  w0[3] = w0[3] ^ 0x6a6a6a6a;
  w1[0] = w1[0] ^ 0x6a6a6a6a;
  w1[1] = w1[1] ^ 0x6a6a6a6a;
  w1[2] = w1[2] ^ 0x6a6a6a6a;
  w1[3] = w1[3] ^ 0x6a6a6a6a;
  w2[0] = w2[0] ^ 0x6a6a6a6a;
  w2[1] = w2[1] ^ 0x6a6a6a6a;
  w2[2] = w2[2] ^ 0x6a6a6a6a;
  w2[3] = w2[3] ^ 0x6a6a6a6a;
  w3[0] = w3[0] ^ 0x6a6a6a6a;
  w3[1] = w3[1] ^ 0x6a6a6a6a;
  w3[2] = w3[2] ^ 0x6a6a6a6a;
  w3[3] = w3[3] ^ 0x6a6a6a6a;

  opad[0] = SHA256M_A;
  opad[1] = SHA256M_B;
  opad[2] = SHA256M_C;
  opad[3] = SHA256M_D;
  opad[4] = SHA256M_E;
  opad[5] = SHA256M_F;
  opad[6] = SHA256M_G;
  opad[7] = SHA256M_H;

  sha256_transform (w0, w1, w2, w3, opad);
}

__device__ static void hmac_sha256_run (u32x w0[4], u32x w1[4], u32x w2[4], u32x w3[4], u32x ipad[8], u32x opad[8], u32x digest[8])
{
  digest[0] = ipad[0];
  digest[1] = ipad[1];
  digest[2] = ipad[2];
  digest[3] = ipad[3];
  digest[4] = ipad[4];
  digest[5] = ipad[5];
  digest[6] = ipad[6];
  digest[7] = ipad[7];

  sha256_transform (w0, w1, w2, w3, digest);

  w0[0] = digest[0];
  w0[1] = digest[1];
  w0[2] = digest[2];
  w0[3] = digest[3];
  w1[0] = digest[4];
  w1[1] = digest[5];
  w1[2] = digest[6];
  w1[3] = digest[7];
  w2[0] = 0x80000000;
  w2[1] = 0;
  w2[2] = 0;
  w2[3] = 0;
  w3[0] = 0;
  w3[1] = 0;
  w3[2] = 0;
  w3[3] = (64 + 32) * 8;

  digest[0] = opad[0];
  digest[1] = opad[1];
  digest[2] = opad[2];
  digest[3] = opad[3];
  digest[4] = opad[4];
  digest[5] = opad[5];
  digest[6] = opad[6];
  digest[7] = opad[7];

  sha256_transform (w0, w1, w2, w3, digest);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m06400_init (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, sha256aix_tmp_t *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x w0[4];

  w0[0] = pws[gid].i[ 0];
  w0[1] = pws[gid].i[ 1];
  w0[2] = pws[gid].i[ 2];
  w0[3] = pws[gid].i[ 3];

  u32x w1[4];

  w1[0] = pws[gid].i[ 4];
  w1[1] = pws[gid].i[ 5];
  w1[2] = pws[gid].i[ 6];
  w1[3] = pws[gid].i[ 7];

  u32x w2[4];

  w2[0] = pws[gid].i[ 8];
  w2[1] = pws[gid].i[ 9];
  w2[2] = pws[gid].i[10];
  w2[3] = pws[gid].i[11];

  u32x w3[4];

  w3[0] = pws[gid].i[12];
  w3[1] = pws[gid].i[13];
  w3[2] = pws[gid].i[14];
  w3[3] = pws[gid].i[15];

  /**
   * salt
   */

  u32 salt_len = salt_bufs[salt_pos].salt_len;

  u32 salt_buf0[4];

  salt_buf0[0] = salt_bufs[salt_pos].salt_buf[ 0];
  salt_buf0[1] = salt_bufs[salt_pos].salt_buf[ 1];
  salt_buf0[2] = salt_bufs[salt_pos].salt_buf[ 2];
  salt_buf0[3] = salt_bufs[salt_pos].salt_buf[ 3];

  u32 salt_buf1[4];

  salt_buf1[0] = salt_bufs[salt_pos].salt_buf[ 4];
  salt_buf1[1] = salt_bufs[salt_pos].salt_buf[ 5];
  salt_buf1[2] = salt_bufs[salt_pos].salt_buf[ 6];
  salt_buf1[3] = salt_bufs[salt_pos].salt_buf[ 7];

  u32 salt_buf2[4];

  salt_buf2[0] = salt_bufs[salt_pos].salt_buf[ 8];
  salt_buf2[1] = salt_bufs[salt_pos].salt_buf[ 9];
  salt_buf2[2] = salt_bufs[salt_pos].salt_buf[10];
  salt_buf2[3] = salt_bufs[salt_pos].salt_buf[11];

  u32 salt_buf3[4];

  salt_buf3[0] = 0;
  salt_buf3[1] = 0;
  salt_buf3[2] = 0;
  salt_buf3[3] = 0;

  append_0x01_4 (salt_buf0, salt_buf1, salt_buf2, salt_buf3, salt_len + 3);

  append_0x80_4 (salt_buf0, salt_buf1, salt_buf2, salt_buf3, salt_len + 4);

  /**
   * pads
   */

  w0[0] = swap_workaround (w0[0]);
  w0[1] = swap_workaround (w0[1]);
  w0[2] = swap_workaround (w0[2]);
  w0[3] = swap_workaround (w0[3]);
  w1[0] = swap_workaround (w1[0]);
  w1[1] = swap_workaround (w1[1]);
  w1[2] = swap_workaround (w1[2]);
  w1[3] = swap_workaround (w1[3]);
  w2[0] = swap_workaround (w2[0]);
  w2[1] = swap_workaround (w2[1]);
  w2[2] = swap_workaround (w2[2]);
  w2[3] = swap_workaround (w2[3]);
  w3[0] = swap_workaround (w3[0]);
  w3[1] = swap_workaround (w3[1]);
  w3[2] = swap_workaround (w3[2]);
  w3[3] = swap_workaround (w3[3]);

  u32x ipad[8];
  u32x opad[8];

  hmac_sha256_pad (w0, w1, w2, w3, ipad, opad);

  tmps[gid].ipad[0] = ipad[0];
  tmps[gid].ipad[1] = ipad[1];
  tmps[gid].ipad[2] = ipad[2];
  tmps[gid].ipad[3] = ipad[3];
  tmps[gid].ipad[4] = ipad[4];
  tmps[gid].ipad[5] = ipad[5];
  tmps[gid].ipad[6] = ipad[6];
  tmps[gid].ipad[7] = ipad[7];

  tmps[gid].opad[0] = opad[0];
  tmps[gid].opad[1] = opad[1];
  tmps[gid].opad[2] = opad[2];
  tmps[gid].opad[3] = opad[3];
  tmps[gid].opad[4] = opad[4];
  tmps[gid].opad[5] = opad[5];
  tmps[gid].opad[6] = opad[6];
  tmps[gid].opad[7] = opad[7];

  w0[0] = salt_buf0[0];
  w0[1] = salt_buf0[1];
  w0[2] = salt_buf0[2];
  w0[3] = salt_buf0[3];
  w1[0] = salt_buf1[0];
  w1[1] = salt_buf1[1];
  w1[2] = salt_buf1[2];
  w1[3] = salt_buf1[3];
  w2[0] = salt_buf2[0];
  w2[1] = salt_buf2[1];
  w2[2] = salt_buf2[2];
  w2[3] = salt_buf2[3];
  w3[0] = salt_buf3[0];
  w3[1] = salt_buf3[1];
  w3[2] = salt_buf3[2];
  // w3[3] = 0;

  w0[0] = swap_workaround (w0[0]);
  w0[1] = swap_workaround (w0[1]);
  w0[2] = swap_workaround (w0[2]);
  w0[3] = swap_workaround (w0[3]);
  w1[0] = swap_workaround (w1[0]);
  w1[1] = swap_workaround (w1[1]);
  w1[2] = swap_workaround (w1[2]);
  w1[3] = swap_workaround (w1[3]);
  w2[0] = swap_workaround (w2[0]);
  w2[1] = swap_workaround (w2[1]);
  w2[2] = swap_workaround (w2[2]);
  w2[3] = swap_workaround (w2[3]);
  w3[0] = swap_workaround (w3[0]);
  w3[1] = swap_workaround (w3[1]);
  w3[2] = swap_workaround (w3[2]);
  w3[3] = (64 + salt_len + 4) * 8;

  u32x dgst[8];

  hmac_sha256_run (w0, w1, w2, w3, ipad, opad, dgst);

  tmps[gid].dgst[0] = dgst[0];
  tmps[gid].dgst[1] = dgst[1];
  tmps[gid].dgst[2] = dgst[2];
  tmps[gid].dgst[3] = dgst[3];
  tmps[gid].dgst[4] = dgst[4];
  tmps[gid].dgst[5] = dgst[5];
  tmps[gid].dgst[6] = dgst[6];
  tmps[gid].dgst[7] = dgst[7];

  tmps[gid].out[0] = dgst[0];
  tmps[gid].out[1] = dgst[1];
  tmps[gid].out[2] = dgst[2];
  tmps[gid].out[3] = dgst[3];
  tmps[gid].out[4] = dgst[4];
  tmps[gid].out[5] = dgst[5];
  tmps[gid].out[6] = dgst[6];
  tmps[gid].out[7] = dgst[7];
}

extern "C" __global__ void __launch_bounds__ (256, 1) m06400_loop (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, sha256aix_tmp_t *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x ipad[8];
  u32x opad[8];

  ipad[0] = tmps[gid].ipad[0];
  ipad[1] = tmps[gid].ipad[1];
  ipad[2] = tmps[gid].ipad[2];
  ipad[3] = tmps[gid].ipad[3];
  ipad[4] = tmps[gid].ipad[4];
  ipad[5] = tmps[gid].ipad[5];
  ipad[6] = tmps[gid].ipad[6];
  ipad[7] = tmps[gid].ipad[7];

  opad[0] = tmps[gid].opad[0];
  opad[1] = tmps[gid].opad[1];
  opad[2] = tmps[gid].opad[2];
  opad[3] = tmps[gid].opad[3];
  opad[4] = tmps[gid].opad[4];
  opad[5] = tmps[gid].opad[5];
  opad[6] = tmps[gid].opad[6];
  opad[7] = tmps[gid].opad[7];

  u32x dgst[8];
  u32x out[8];

  dgst[0] = tmps[gid].dgst[0];
  dgst[1] = tmps[gid].dgst[1];
  dgst[2] = tmps[gid].dgst[2];
  dgst[3] = tmps[gid].dgst[3];
  dgst[4] = tmps[gid].dgst[4];
  dgst[5] = tmps[gid].dgst[5];
  dgst[6] = tmps[gid].dgst[6];
  dgst[7] = tmps[gid].dgst[7];

  out[0] = tmps[gid].out[0];
  out[1] = tmps[gid].out[1];
  out[2] = tmps[gid].out[2];
  out[3] = tmps[gid].out[3];
  out[4] = tmps[gid].out[4];
  out[5] = tmps[gid].out[5];
  out[6] = tmps[gid].out[6];
  out[7] = tmps[gid].out[7];

  for (u32 j = 0; j < loop_cnt; j++)
  {
    u32x w0[4];
    u32x w1[4];
    u32x w2[4];
    u32x w3[4];

    w0[0] = dgst[0];
    w0[1] = dgst[1];
    w0[2] = dgst[2];
    w0[3] = dgst[3];
    w1[0] = dgst[4];
    w1[1] = dgst[5];
    w1[2] = dgst[6];
    w1[3] = dgst[7];
    w2[0] = 0x80000000;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;
    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = (64 + 32) * 8;

    hmac_sha256_run (w0, w1, w2, w3, ipad, opad, dgst);

    out[0] ^= dgst[0];
    out[1] ^= dgst[1];
    out[2] ^= dgst[2];
    out[3] ^= dgst[3];
    out[4] ^= dgst[4];
    out[5] ^= dgst[5];
    out[6] ^= dgst[6];
    out[7] ^= dgst[7];
  }

  tmps[gid].dgst[0] = dgst[0];
  tmps[gid].dgst[1] = dgst[1];
  tmps[gid].dgst[2] = dgst[2];
  tmps[gid].dgst[3] = dgst[3];
  tmps[gid].dgst[4] = dgst[4];
  tmps[gid].dgst[5] = dgst[5];
  tmps[gid].dgst[6] = dgst[6];
  tmps[gid].dgst[7] = dgst[7];

  tmps[gid].out[0] = out[0];
  tmps[gid].out[1] = out[1];
  tmps[gid].out[2] = out[2];
  tmps[gid].out[3] = out[3];
  tmps[gid].out[4] = out[4];
  tmps[gid].out[5] = out[5];
  tmps[gid].out[6] = out[6];
  tmps[gid].out[7] = out[7];
}

extern "C" __global__ void __launch_bounds__ (256, 1) m06400_comp (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, sha256aix_tmp_t *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  const u32 lid = threadIdx.x;

  /**
   * digest
   */

  const u32x r0 = tmps[gid].out[DGST_R0];
  const u32x r1 = tmps[gid].out[DGST_R1];
  const u32x r2 = tmps[gid].out[DGST_R2];
  const u32x r3 = tmps[gid].out[DGST_R3];

  /*
  u32x a = tmps[gid].out[0];
  u32x b = tmps[gid].out[1];
  u32x c = tmps[gid].out[2];
  u32x d = tmps[gid].out[3];
  u32x e = tmps[gid].out[4];
  u32x f = tmps[gid].out[5];
  u32x g = tmps[gid].out[6];
  u32x h = tmps[gid].out[7] & 0xffff03ff;
  */

  #define il_pos 0

  #include VECT_COMPARE_M
}
