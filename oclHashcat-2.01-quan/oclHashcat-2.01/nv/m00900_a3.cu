#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _MD4_
#define _SCALAR_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE4
#endif

#define DGST_R0 0
#define DGST_R1 3
#define DGST_R2 2
#define DGST_R3 1

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_S "check_single_vect1_comp4_warp.c"
#define VECT_COMPARE_M "check_multi_vect1_comp4_warp.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_S "check_single_vect2_comp4_warp.c"
#define VECT_COMPARE_M "check_multi_vect2_comp4_warp.c"
#endif

#ifdef  VECT_SIZE4
#define VECT_COMPARE_S "check_single_vect4_comp4_warp.c"
#define VECT_COMPARE_M "check_multi_vect4_comp4_warp.c"
#endif

#define MD4_STEP_REV(f,a,b,c,d,x,t,s)   \
{                                       \
  a  = rotr32 (a, s);                   \
  a -= f (b, c, d);                     \
  a -= x;                               \
  a -= t;                               \
}

#define MD4_STEP_REV1(f,a,b,c,d,x,t,s)  \
{                                       \
  a  = rotr32 (a, s);                   \
  a -= x;                               \
  a -= t;                               \
}

__device__ __constant__ u32x c_bfs[1024];

__device__ static void m00900m (u32 w[16], const u32 pw_len, const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x * words_buf_r, void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset)
{
  /**
   * modifier
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;
  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 F_w0c00 =     0 + MD4C00;
  const u32 F_w1c00 = w[ 1] + MD4C00;
  const u32 F_w2c00 = w[ 2] + MD4C00;
  const u32 F_w3c00 = w[ 3] + MD4C00;
  const u32 F_w4c00 = w[ 4] + MD4C00;
  const u32 F_w5c00 = w[ 5] + MD4C00;
  const u32 F_w6c00 = w[ 6] + MD4C00;
  const u32 F_w7c00 = w[ 7] + MD4C00;
  const u32 F_w8c00 = w[ 8] + MD4C00;
  const u32 F_w9c00 = w[ 9] + MD4C00;
  const u32 F_wac00 = w[10] + MD4C00;
  const u32 F_wbc00 = w[11] + MD4C00;
  const u32 F_wcc00 = w[12] + MD4C00;
  const u32 F_wdc00 = w[13] + MD4C00;
  const u32 F_wec00 = w[14] + MD4C00;
  const u32 F_wfc00 = w[15] + MD4C00;

  const u32 G_w0c01 =     0 + MD4C01;
  const u32 G_w4c01 = w[ 4] + MD4C01;
  const u32 G_w8c01 = w[ 8] + MD4C01;
  const u32 G_wcc01 = w[12] + MD4C01;
  const u32 G_w1c01 = w[ 1] + MD4C01;
  const u32 G_w5c01 = w[ 5] + MD4C01;
  const u32 G_w9c01 = w[ 9] + MD4C01;
  const u32 G_wdc01 = w[13] + MD4C01;
  const u32 G_w2c01 = w[ 2] + MD4C01;
  const u32 G_w6c01 = w[ 6] + MD4C01;
  const u32 G_wac01 = w[10] + MD4C01;
  const u32 G_wec01 = w[14] + MD4C01;
  const u32 G_w3c01 = w[ 3] + MD4C01;
  const u32 G_w7c01 = w[ 7] + MD4C01;
  const u32 G_wbc01 = w[11] + MD4C01;
  const u32 G_wfc01 = w[15] + MD4C01;

  const u32 H_w0c02 =     0 + MD4C02;
  const u32 H_w8c02 = w[ 8] + MD4C02;
  const u32 H_w4c02 = w[ 4] + MD4C02;
  const u32 H_wcc02 = w[12] + MD4C02;
  const u32 H_w2c02 = w[ 2] + MD4C02;
  const u32 H_wac02 = w[10] + MD4C02;
  const u32 H_w6c02 = w[ 6] + MD4C02;
  const u32 H_wec02 = w[14] + MD4C02;
  const u32 H_w1c02 = w[ 1] + MD4C02;
  const u32 H_w9c02 = w[ 9] + MD4C02;
  const u32 H_w5c02 = w[ 5] + MD4C02;
  const u32 H_wdc02 = w[13] + MD4C02;
  const u32 H_w3c02 = w[ 3] + MD4C02;
  const u32 H_wbc02 = w[11] + MD4C02;
  const u32 H_w7c02 = w[ 7] + MD4C02;
  const u32 H_wfc02 = w[15] + MD4C02;

  /**
   * loop
   */

  const u32 bf_loops = ceil ((float) bfs_cnt / VECT_DIV);

  u32x w0l = w[0];

  for (u32 il_pos = 0; il_pos < bf_loops; il_pos++)
  {
    const u32x w0r = c_bfs[il_pos];

    const u32x w0 = w0l | w0r;

    u32x a = MD4M_A;
    u32x b = MD4M_B;
    u32x c = MD4M_C;
    u32x d = MD4M_D;

    MD4_STEP (MD4_Fo, a, b, c, d, w0, F_w0c00, MD4S00);
    MD4_STEP0(MD4_Fo, d, a, b, c,     F_w1c00, MD4S01);
    MD4_STEP0(MD4_Fo, c, d, a, b,     F_w2c00, MD4S02);
    MD4_STEP0(MD4_Fo, b, c, d, a,     F_w3c00, MD4S03);
    MD4_STEP0(MD4_Fo, a, b, c, d,     F_w4c00, MD4S00);
    MD4_STEP0(MD4_Fo, d, a, b, c,     F_w5c00, MD4S01);
    MD4_STEP0(MD4_Fo, c, d, a, b,     F_w6c00, MD4S02);
    MD4_STEP0(MD4_Fo, b, c, d, a,     F_w7c00, MD4S03);
    MD4_STEP0(MD4_Fo, a, b, c, d,     F_w8c00, MD4S00);
    MD4_STEP0(MD4_Fo, d, a, b, c,     F_w9c00, MD4S01);
    MD4_STEP0(MD4_Fo, c, d, a, b,     F_wac00, MD4S02);
    MD4_STEP0(MD4_Fo, b, c, d, a,     F_wbc00, MD4S03);
    MD4_STEP0(MD4_Fo, a, b, c, d,     F_wcc00, MD4S00);
    MD4_STEP0(MD4_Fo, d, a, b, c,     F_wdc00, MD4S01);
    MD4_STEP0(MD4_Fo, c, d, a, b,     F_wec00, MD4S02);
    MD4_STEP0(MD4_Fo, b, c, d, a,     F_wfc00, MD4S03);

    MD4_STEP (MD4_Go, a, b, c, d, w0, G_w0c01, MD4S10);
    MD4_STEP0(MD4_Go, d, a, b, c,     G_w4c01, MD4S11);
    MD4_STEP0(MD4_Go, c, d, a, b,     G_w8c01, MD4S12);
    MD4_STEP0(MD4_Go, b, c, d, a,     G_wcc01, MD4S13);
    MD4_STEP0(MD4_Go, a, b, c, d,     G_w1c01, MD4S10);
    MD4_STEP0(MD4_Go, d, a, b, c,     G_w5c01, MD4S11);
    MD4_STEP0(MD4_Go, c, d, a, b,     G_w9c01, MD4S12);
    MD4_STEP0(MD4_Go, b, c, d, a,     G_wdc01, MD4S13);
    MD4_STEP0(MD4_Go, a, b, c, d,     G_w2c01, MD4S10);
    MD4_STEP0(MD4_Go, d, a, b, c,     G_w6c01, MD4S11);
    MD4_STEP0(MD4_Go, c, d, a, b,     G_wac01, MD4S12);
    MD4_STEP0(MD4_Go, b, c, d, a,     G_wec01, MD4S13);
    MD4_STEP0(MD4_Go, a, b, c, d,     G_w3c01, MD4S10);
    MD4_STEP0(MD4_Go, d, a, b, c,     G_w7c01, MD4S11);
    MD4_STEP0(MD4_Go, c, d, a, b,     G_wbc01, MD4S12);
    MD4_STEP0(MD4_Go, b, c, d, a,     G_wfc01, MD4S13);

    MD4_STEP (MD4_H , a, b, c, d, w0, H_w0c02, MD4S20);
    MD4_STEP0(MD4_H , d, a, b, c,     H_w8c02, MD4S21);
    MD4_STEP0(MD4_H , c, d, a, b,     H_w4c02, MD4S22);
    MD4_STEP0(MD4_H , b, c, d, a,     H_wcc02, MD4S23);
    MD4_STEP0(MD4_H , a, b, c, d,     H_w2c02, MD4S20);
    MD4_STEP0(MD4_H , d, a, b, c,     H_wac02, MD4S21);
    MD4_STEP0(MD4_H , c, d, a, b,     H_w6c02, MD4S22);
    MD4_STEP0(MD4_H , b, c, d, a,     H_wec02, MD4S23);
    MD4_STEP0(MD4_H , a, b, c, d,     H_w1c02, MD4S20);
    MD4_STEP0(MD4_H , d, a, b, c,     H_w9c02, MD4S21);
    MD4_STEP0(MD4_H , c, d, a, b,     H_w5c02, MD4S22);
    MD4_STEP0(MD4_H , b, c, d, a,     H_wdc02, MD4S23);
    MD4_STEP0(MD4_H , a, b, c, d,     H_w3c02, MD4S20);
    MD4_STEP0(MD4_H , d, a, b, c,     H_wbc02, MD4S21);
    MD4_STEP0(MD4_H , c, d, a, b,     H_w7c02, MD4S22);
    MD4_STEP0(MD4_H , b, c, d, a,     H_wfc02, MD4S23);

    const u32x r0 = a;
    const u32x r1 = d;
    const u32x r2 = c;
    const u32x r3 = b;

    #include VECT_COMPARE_M
  }
}

__device__ static void m00900s (u32 w[16], const u32 pw_len, const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x * words_buf_r, void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset)
{
  /**
   * modifier
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;
  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 F_w0c00 =     0 + MD4C00;
  const u32 F_w1c00 = w[ 1] + MD4C00;
  const u32 F_w2c00 = w[ 2] + MD4C00;
  const u32 F_w3c00 = w[ 3] + MD4C00;
  const u32 F_w4c00 = w[ 4] + MD4C00;
  const u32 F_w5c00 = w[ 5] + MD4C00;
  const u32 F_w6c00 = w[ 6] + MD4C00;
  const u32 F_w7c00 = w[ 7] + MD4C00;
  const u32 F_w8c00 = w[ 8] + MD4C00;
  const u32 F_w9c00 = w[ 9] + MD4C00;
  const u32 F_wac00 = w[10] + MD4C00;
  const u32 F_wbc00 = w[11] + MD4C00;
  const u32 F_wcc00 = w[12] + MD4C00;
  const u32 F_wdc00 = w[13] + MD4C00;
  const u32 F_wec00 = w[14] + MD4C00;
  const u32 F_wfc00 = w[15] + MD4C00;

  const u32 G_w0c01 =     0 + MD4C01;
  const u32 G_w4c01 = w[ 4] + MD4C01;
  const u32 G_w8c01 = w[ 8] + MD4C01;
  const u32 G_wcc01 = w[12] + MD4C01;
  const u32 G_w1c01 = w[ 1] + MD4C01;
  const u32 G_w5c01 = w[ 5] + MD4C01;
  const u32 G_w9c01 = w[ 9] + MD4C01;
  const u32 G_wdc01 = w[13] + MD4C01;
  const u32 G_w2c01 = w[ 2] + MD4C01;
  const u32 G_w6c01 = w[ 6] + MD4C01;
  const u32 G_wac01 = w[10] + MD4C01;
  const u32 G_wec01 = w[14] + MD4C01;
  const u32 G_w3c01 = w[ 3] + MD4C01;
  const u32 G_w7c01 = w[ 7] + MD4C01;
  const u32 G_wbc01 = w[11] + MD4C01;
  const u32 G_wfc01 = w[15] + MD4C01;

  const u32 H_w0c02 =     0 + MD4C02;
  const u32 H_w8c02 = w[ 8] + MD4C02;
  const u32 H_w4c02 = w[ 4] + MD4C02;
  const u32 H_wcc02 = w[12] + MD4C02;
  const u32 H_w2c02 = w[ 2] + MD4C02;
  const u32 H_wac02 = w[10] + MD4C02;
  const u32 H_w6c02 = w[ 6] + MD4C02;
  const u32 H_wec02 = w[14] + MD4C02;
  const u32 H_w1c02 = w[ 1] + MD4C02;
  const u32 H_w9c02 = w[ 9] + MD4C02;
  const u32 H_w5c02 = w[ 5] + MD4C02;
  const u32 H_wdc02 = w[13] + MD4C02;
  const u32 H_w3c02 = w[ 3] + MD4C02;
  const u32 H_wbc02 = w[11] + MD4C02;
  const u32 H_w7c02 = w[ 7] + MD4C02;
  const u32 H_wfc02 = w[15] + MD4C02;

  /**
   * digest
   */

  const u32 search[4] =
  {
    digests_buf[digests_offset].digest_buf[DGST_R0],
    digests_buf[digests_offset].digest_buf[DGST_R1],
    digests_buf[digests_offset].digest_buf[DGST_R2],
    digests_buf[digests_offset].digest_buf[DGST_R3]
  };

  /**
   * reverse
   */

  u32 a_rev = digests_buf[digests_offset].digest_buf[0];
  u32 b_rev = digests_buf[digests_offset].digest_buf[1];
  u32 c_rev = digests_buf[digests_offset].digest_buf[2];
  u32 d_rev = digests_buf[digests_offset].digest_buf[3];

  MD4_STEP_REV (MD4_H, b_rev, c_rev, d_rev, a_rev, w[15], MD4C02, MD4S23);
  MD4_STEP_REV (MD4_H, c_rev, d_rev, a_rev, b_rev, w[ 7], MD4C02, MD4S22);
  MD4_STEP_REV (MD4_H, d_rev, a_rev, b_rev, c_rev, w[11], MD4C02, MD4S21);
  MD4_STEP_REV (MD4_H, a_rev, b_rev, c_rev, d_rev, w[ 3], MD4C02, MD4S20);
  MD4_STEP_REV (MD4_H, b_rev, c_rev, d_rev, a_rev, w[13], MD4C02, MD4S23);
  MD4_STEP_REV (MD4_H, c_rev, d_rev, a_rev, b_rev, w[ 5], MD4C02, MD4S22);
  MD4_STEP_REV (MD4_H, d_rev, a_rev, b_rev, c_rev, w[ 9], MD4C02, MD4S21);
  MD4_STEP_REV (MD4_H, a_rev, b_rev, c_rev, d_rev, w[ 1], MD4C02, MD4S20);
  MD4_STEP_REV (MD4_H, b_rev, c_rev, d_rev, a_rev, w[14], MD4C02, MD4S23);
  MD4_STEP_REV (MD4_H, c_rev, d_rev, a_rev, b_rev, w[ 6], MD4C02, MD4S22);
  MD4_STEP_REV (MD4_H, d_rev, a_rev, b_rev, c_rev, w[10], MD4C02, MD4S21);
  MD4_STEP_REV (MD4_H, a_rev, b_rev, c_rev, d_rev, w[ 2], MD4C02, MD4S20);
  MD4_STEP_REV (MD4_H, b_rev, c_rev, d_rev, a_rev, w[12], MD4C02, MD4S23);
  MD4_STEP_REV (MD4_H, c_rev, d_rev, a_rev, b_rev, w[ 4], MD4C02, MD4S22);
  MD4_STEP_REV (MD4_H, d_rev, a_rev, b_rev, c_rev, w[ 8], MD4C02, MD4S21);
  MD4_STEP_REV (MD4_H, a_rev, b_rev, c_rev, d_rev,     0, MD4C02, MD4S20);

  const u32x sav_c = c_rev;
  const u32x sav_d = d_rev;

  MD4_STEP_REV1(MD4_G, b_rev, c_rev, d_rev, a_rev, w[15], MD4C01, MD4S13);
  MD4_STEP_REV1(MD4_G, c_rev, d_rev, a_rev, b_rev, w[11], MD4C01, MD4S12);

  /**
   * loop
   */

  const u32 bf_loops = ceil ((float) bfs_cnt / VECT_DIV);

  u32x w0l = w[0];

  for (u32 il_pos = 0; il_pos < bf_loops; il_pos++)
  {
    const u32x w0r = c_bfs[il_pos];

    const u32x w0 = w0l | w0r;

    u32x pre_a = a_rev;
    u32x pre_b = b_rev;
    u32x pre_c = c_rev;

    pre_a = pre_a - w0;
    pre_b = pre_b - MD4_G (sav_c, sav_d, pre_a);
    pre_c = pre_c - MD4_G (sav_d, pre_a, pre_b);

    u32x a = MD4M_A;
    u32x b = MD4M_B;
    u32x c = MD4M_C;
    u32x d = MD4M_D;

    MD4_STEP (MD4_Fo, a, b, c, d, w0, F_w0c00, MD4S00);
    MD4_STEP0(MD4_Fo, d, a, b, c,     F_w1c00, MD4S01);
    MD4_STEP0(MD4_Fo, c, d, a, b,     F_w2c00, MD4S02);
    MD4_STEP0(MD4_Fo, b, c, d, a,     F_w3c00, MD4S03);
    MD4_STEP0(MD4_Fo, a, b, c, d,     F_w4c00, MD4S00);
    MD4_STEP0(MD4_Fo, d, a, b, c,     F_w5c00, MD4S01);
    MD4_STEP0(MD4_Fo, c, d, a, b,     F_w6c00, MD4S02);
    MD4_STEP0(MD4_Fo, b, c, d, a,     F_w7c00, MD4S03);
    MD4_STEP0(MD4_Fo, a, b, c, d,     F_w8c00, MD4S00);
    MD4_STEP0(MD4_Fo, d, a, b, c,     F_w9c00, MD4S01);
    MD4_STEP0(MD4_Fo, c, d, a, b,     F_wac00, MD4S02);
    MD4_STEP0(MD4_Fo, b, c, d, a,     F_wbc00, MD4S03);
    MD4_STEP0(MD4_Fo, a, b, c, d,     F_wcc00, MD4S00);
    MD4_STEP0(MD4_Fo, d, a, b, c,     F_wdc00, MD4S01);
    MD4_STEP0(MD4_Fo, c, d, a, b,     F_wec00, MD4S02);
    MD4_STEP0(MD4_Fo, b, c, d, a,     F_wfc00, MD4S03);

    MD4_STEP (MD4_Go, a, b, c, d, w0, G_w0c01, MD4S10);
    MD4_STEP0(MD4_Go, d, a, b, c,     G_w4c01, MD4S11);
    MD4_STEP0(MD4_Go, c, d, a, b,     G_w8c01, MD4S12);
    MD4_STEP0(MD4_Go, b, c, d, a,     G_wcc01, MD4S13);
    MD4_STEP0(MD4_Go, a, b, c, d,     G_w1c01, MD4S10);
    MD4_STEP0(MD4_Go, d, a, b, c,     G_w5c01, MD4S11);
    MD4_STEP0(MD4_Go, c, d, a, b,     G_w9c01, MD4S12);
    MD4_STEP0(MD4_Go, b, c, d, a,     G_wdc01, MD4S13);
    MD4_STEP0(MD4_Go, a, b, c, d,     G_w2c01, MD4S10);
    MD4_STEP0(MD4_Go, d, a, b, c,     G_w6c01, MD4S11);
    MD4_STEP0(MD4_Go, c, d, a, b,     G_wac01, MD4S12);

    bool q_cond = (pre_c != c);

    if (q_cond) continue;

    MD4_STEP0(MD4_Go, b, c, d, a,     G_wec01, MD4S13);
    MD4_STEP0(MD4_Go, a, b, c, d,     G_w3c01, MD4S10);
    MD4_STEP0(MD4_Go, d, a, b, c,     G_w7c01, MD4S11);
    MD4_STEP0(MD4_Go, c, d, a, b,     G_wbc01, MD4S12);
    MD4_STEP0(MD4_Go, b, c, d, a,     G_wfc01, MD4S13);

    MD4_STEP (MD4_H , a, b, c, d, w0, H_w0c02, MD4S20);
    MD4_STEP0(MD4_H , d, a, b, c,     H_w8c02, MD4S21);
    MD4_STEP0(MD4_H , c, d, a, b,     H_w4c02, MD4S22);
    MD4_STEP0(MD4_H , b, c, d, a,     H_wcc02, MD4S23);
    MD4_STEP0(MD4_H , a, b, c, d,     H_w2c02, MD4S20);
    MD4_STEP0(MD4_H , d, a, b, c,     H_wac02, MD4S21);
    MD4_STEP0(MD4_H , c, d, a, b,     H_w6c02, MD4S22);
    MD4_STEP0(MD4_H , b, c, d, a,     H_wec02, MD4S23);
    MD4_STEP0(MD4_H , a, b, c, d,     H_w1c02, MD4S20);
    MD4_STEP0(MD4_H , d, a, b, c,     H_w9c02, MD4S21);
    MD4_STEP0(MD4_H , c, d, a, b,     H_w5c02, MD4S22);
    MD4_STEP0(MD4_H , b, c, d, a,     H_wdc02, MD4S23);
    MD4_STEP0(MD4_H , a, b, c, d,     H_w3c02, MD4S20);
    MD4_STEP0(MD4_H , d, a, b, c,     H_wbc02, MD4S21);
    MD4_STEP0(MD4_H , c, d, a, b,     H_w7c02, MD4S22);
    MD4_STEP0(MD4_H , b, c, d, a,     H_wfc02, MD4S23);

    const u32x r0 = a;
    const u32x r1 = d;
    const u32x r2 = c;
    const u32x r3 = b;

    #include VECT_COMPARE_S
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m00900_m04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r,void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = 0;
  w[ 5] = 0;
  w[ 6] = 0;
  w[ 7] = 0;
  w[ 8] = 0;
  w[ 9] = 0;
  w[10] = 0;
  w[11] = 0;
  w[12] = 0;
  w[13] = 0;
  w[14] = pws[gid].i[14];
  w[15] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m00900m (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m00900_m08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r,void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = pws[gid].i[ 4];
  w[ 5] = pws[gid].i[ 5];
  w[ 6] = pws[gid].i[ 6];
  w[ 7] = pws[gid].i[ 7];
  w[ 8] = 0;
  w[ 9] = 0;
  w[10] = 0;
  w[11] = 0;
  w[12] = 0;
  w[13] = 0;
  w[14] = pws[gid].i[14];
  w[15] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m00900m (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m00900_m16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r,void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = pws[gid].i[ 4];
  w[ 5] = pws[gid].i[ 5];
  w[ 6] = pws[gid].i[ 6];
  w[ 7] = pws[gid].i[ 7];
  w[ 8] = pws[gid].i[ 8];
  w[ 9] = pws[gid].i[ 9];
  w[10] = pws[gid].i[10];
  w[11] = pws[gid].i[11];
  w[12] = pws[gid].i[12];
  w[13] = pws[gid].i[13];
  w[14] = pws[gid].i[14];
  w[15] = pws[gid].i[15];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m00900m (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m00900_s04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r,void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = 0;
  w[ 5] = 0;
  w[ 6] = 0;
  w[ 7] = 0;
  w[ 8] = 0;
  w[ 9] = 0;
  w[10] = 0;
  w[11] = 0;
  w[12] = 0;
  w[13] = 0;
  w[14] = pws[gid].i[14];
  w[15] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m00900s (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m00900_s08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r,void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = pws[gid].i[ 4];
  w[ 5] = pws[gid].i[ 5];
  w[ 6] = pws[gid].i[ 6];
  w[ 7] = pws[gid].i[ 7];
  w[ 8] = 0;
  w[ 9] = 0;
  w[10] = 0;
  w[11] = 0;
  w[12] = 0;
  w[13] = 0;
  w[14] = pws[gid].i[14];
  w[15] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m00900s (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m00900_s16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r,void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = pws[gid].i[ 4];
  w[ 5] = pws[gid].i[ 5];
  w[ 6] = pws[gid].i[ 6];
  w[ 7] = pws[gid].i[ 7];
  w[ 8] = pws[gid].i[ 8];
  w[ 9] = pws[gid].i[ 9];
  w[10] = pws[gid].i[10];
  w[11] = pws[gid].i[11];
  w[12] = pws[gid].i[12];
  w[13] = pws[gid].i[13];
  w[14] = pws[gid].i[14];
  w[15] = pws[gid].i[15];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m00900s (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}
