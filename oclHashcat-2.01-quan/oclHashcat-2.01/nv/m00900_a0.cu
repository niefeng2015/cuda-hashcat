#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _MD4_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 0
#define DGST_R1 3
#define DGST_R2 2
#define DGST_R3 1

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"
#include "include/rp_gpu.h"
#include "rp_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_S "check_single_vect1_comp4.c"
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_S "check_single_vect2_comp4.c"
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

#ifdef  VECT_SIZE4
#define VECT_COMPARE_S "check_single_vect4_comp4.c"
#define VECT_COMPARE_M "check_multi_vect4_comp4.c"
#endif

__device__ __constant__ gpu_rule_t c_rules[1024];

extern "C" __global__ void __launch_bounds__ (256, 1) m00900_m04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x pw_buf0[4];

  pw_buf0[0] = pws[gid].i[ 0];
  pw_buf0[1] = pws[gid].i[ 1];
  pw_buf0[2] = pws[gid].i[ 2];
  pw_buf0[3] = pws[gid].i[ 3];

  u32x pw_buf1[4];

  pw_buf1[0] = pws[gid].i[ 4];
  pw_buf1[1] = pws[gid].i[ 5];
  pw_buf1[2] = pws[gid].i[ 6];
  pw_buf1[3] = pws[gid].i[ 7];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * loop
   */

  for (u32 il_pos = 0; il_pos < rules_cnt; il_pos++)
  {
    u32x w0[4];

    w0[0] = pw_buf0[0];
    w0[1] = pw_buf0[1];
    w0[2] = pw_buf0[2];
    w0[3] = pw_buf0[3];

    u32x w1[4];

    w1[0] = pw_buf1[0];
    w1[1] = pw_buf1[1];
    w1[2] = pw_buf1[2];
    w1[3] = pw_buf1[3];

    u32x w2[4];

    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = 0;

    const u32 out_len = apply_rules (c_rules[il_pos].cmds, w0, w1, pw_len);

    append_0x80_2 (w0, w1, out_len);

    w3[2] = out_len * 8;

    u32x a = MD4M_A;
    u32x b = MD4M_B;
    u32x c = MD4M_C;
    u32x d = MD4M_D;

    MD4_STEP (MD4_Fo, a, b, c, d, w0[0], MD4C00, MD4S00);
    MD4_STEP (MD4_Fo, d, a, b, c, w0[1], MD4C00, MD4S01);
    MD4_STEP (MD4_Fo, c, d, a, b, w0[2], MD4C00, MD4S02);
    MD4_STEP (MD4_Fo, b, c, d, a, w0[3], MD4C00, MD4S03);
    MD4_STEP (MD4_Fo, a, b, c, d, w1[0], MD4C00, MD4S00);
    MD4_STEP (MD4_Fo, d, a, b, c, w1[1], MD4C00, MD4S01);
    MD4_STEP (MD4_Fo, c, d, a, b, w1[2], MD4C00, MD4S02);
    MD4_STEP (MD4_Fo, b, c, d, a, w1[3], MD4C00, MD4S03);
    MD4_STEP (MD4_Fo, a, b, c, d, w2[0], MD4C00, MD4S00);
    MD4_STEP (MD4_Fo, d, a, b, c, w2[1], MD4C00, MD4S01);
    MD4_STEP (MD4_Fo, c, d, a, b, w2[2], MD4C00, MD4S02);
    MD4_STEP (MD4_Fo, b, c, d, a, w2[3], MD4C00, MD4S03);
    MD4_STEP (MD4_Fo, a, b, c, d, w3[0], MD4C00, MD4S00);
    MD4_STEP (MD4_Fo, d, a, b, c, w3[1], MD4C00, MD4S01);
    MD4_STEP (MD4_Fo, c, d, a, b, w3[2], MD4C00, MD4S02);
    MD4_STEP (MD4_Fo, b, c, d, a, w3[3], MD4C00, MD4S03);

    MD4_STEP (MD4_Go, a, b, c, d, w0[0], MD4C01, MD4S10);
    MD4_STEP (MD4_Go, d, a, b, c, w1[0], MD4C01, MD4S11);
    MD4_STEP (MD4_Go, c, d, a, b, w2[0], MD4C01, MD4S12);
    MD4_STEP (MD4_Go, b, c, d, a, w3[0], MD4C01, MD4S13);
    MD4_STEP (MD4_Go, a, b, c, d, w0[1], MD4C01, MD4S10);
    MD4_STEP (MD4_Go, d, a, b, c, w1[1], MD4C01, MD4S11);
    MD4_STEP (MD4_Go, c, d, a, b, w2[1], MD4C01, MD4S12);
    MD4_STEP (MD4_Go, b, c, d, a, w3[1], MD4C01, MD4S13);
    MD4_STEP (MD4_Go, a, b, c, d, w0[2], MD4C01, MD4S10);
    MD4_STEP (MD4_Go, d, a, b, c, w1[2], MD4C01, MD4S11);
    MD4_STEP (MD4_Go, c, d, a, b, w2[2], MD4C01, MD4S12);
    MD4_STEP (MD4_Go, b, c, d, a, w3[2], MD4C01, MD4S13);
    MD4_STEP (MD4_Go, a, b, c, d, w0[3], MD4C01, MD4S10);
    MD4_STEP (MD4_Go, d, a, b, c, w1[3], MD4C01, MD4S11);
    MD4_STEP (MD4_Go, c, d, a, b, w2[3], MD4C01, MD4S12);
    MD4_STEP (MD4_Go, b, c, d, a, w3[3], MD4C01, MD4S13);

    MD4_STEP (MD4_H , a, b, c, d, w0[0], MD4C02, MD4S20);
    MD4_STEP (MD4_H , d, a, b, c, w2[0], MD4C02, MD4S21);
    MD4_STEP (MD4_H , c, d, a, b, w1[0], MD4C02, MD4S22);
    MD4_STEP (MD4_H , b, c, d, a, w3[0], MD4C02, MD4S23);
    MD4_STEP (MD4_H , a, b, c, d, w0[2], MD4C02, MD4S20);
    MD4_STEP (MD4_H , d, a, b, c, w2[2], MD4C02, MD4S21);
    MD4_STEP (MD4_H , c, d, a, b, w1[2], MD4C02, MD4S22);
    MD4_STEP (MD4_H , b, c, d, a, w3[2], MD4C02, MD4S23);
    MD4_STEP (MD4_H , a, b, c, d, w0[1], MD4C02, MD4S20);
    MD4_STEP (MD4_H , d, a, b, c, w2[1], MD4C02, MD4S21);
    MD4_STEP (MD4_H , c, d, a, b, w1[1], MD4C02, MD4S22);
    MD4_STEP (MD4_H , b, c, d, a, w3[1], MD4C02, MD4S23);
    MD4_STEP (MD4_H , a, b, c, d, w0[3], MD4C02, MD4S20);
    MD4_STEP (MD4_H , d, a, b, c, w2[3], MD4C02, MD4S21);
    MD4_STEP (MD4_H , c, d, a, b, w1[3], MD4C02, MD4S22);
    MD4_STEP (MD4_H , b, c, d, a, w3[3], MD4C02, MD4S23);

    const u32x r0 = a;
    const u32x r1 = d;
    const u32x r2 = c;
    const u32x r3 = b;

    #include VECT_COMPARE_M
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m00900_m08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m00900_m16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m00900_s04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x pw_buf0[4];

  pw_buf0[0] = pws[gid].i[ 0];
  pw_buf0[1] = pws[gid].i[ 1];
  pw_buf0[2] = pws[gid].i[ 2];
  pw_buf0[3] = pws[gid].i[ 3];

  u32x pw_buf1[4];

  pw_buf1[0] = pws[gid].i[ 4];
  pw_buf1[1] = pws[gid].i[ 5];
  pw_buf1[2] = pws[gid].i[ 6];
  pw_buf1[3] = pws[gid].i[ 7];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * digest
   */

  const u32 search[4] =
  {
    digests_buf[digests_offset].digest_buf[DGST_R0],
    digests_buf[digests_offset].digest_buf[DGST_R1],
    digests_buf[digests_offset].digest_buf[DGST_R2],
    digests_buf[digests_offset].digest_buf[DGST_R3]
  };

  /**
   * loop
   */

  for (u32 il_pos = 0; il_pos < rules_cnt; il_pos++)
  {
    u32x w0[4];

    w0[0] = pw_buf0[0];
    w0[1] = pw_buf0[1];
    w0[2] = pw_buf0[2];
    w0[3] = pw_buf0[3];

    u32x w1[4];

    w1[0] = pw_buf1[0];
    w1[1] = pw_buf1[1];
    w1[2] = pw_buf1[2];
    w1[3] = pw_buf1[3];

    u32x w2[4];

    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = 0;

    const u32 out_len = apply_rules (c_rules[il_pos].cmds, w0, w1, pw_len);

    append_0x80_2 (w0, w1, out_len);

    w3[2] = out_len * 8;

    u32x a = MD4M_A;
    u32x b = MD4M_B;
    u32x c = MD4M_C;
    u32x d = MD4M_D;

    MD4_STEP (MD4_Fo, a, b, c, d, w0[0], MD4C00, MD4S00);
    MD4_STEP (MD4_Fo, d, a, b, c, w0[1], MD4C00, MD4S01);
    MD4_STEP (MD4_Fo, c, d, a, b, w0[2], MD4C00, MD4S02);
    MD4_STEP (MD4_Fo, b, c, d, a, w0[3], MD4C00, MD4S03);
    MD4_STEP (MD4_Fo, a, b, c, d, w1[0], MD4C00, MD4S00);
    MD4_STEP (MD4_Fo, d, a, b, c, w1[1], MD4C00, MD4S01);
    MD4_STEP (MD4_Fo, c, d, a, b, w1[2], MD4C00, MD4S02);
    MD4_STEP (MD4_Fo, b, c, d, a, w1[3], MD4C00, MD4S03);
    MD4_STEP (MD4_Fo, a, b, c, d, w2[0], MD4C00, MD4S00);
    MD4_STEP (MD4_Fo, d, a, b, c, w2[1], MD4C00, MD4S01);
    MD4_STEP (MD4_Fo, c, d, a, b, w2[2], MD4C00, MD4S02);
    MD4_STEP (MD4_Fo, b, c, d, a, w2[3], MD4C00, MD4S03);
    MD4_STEP (MD4_Fo, a, b, c, d, w3[0], MD4C00, MD4S00);
    MD4_STEP (MD4_Fo, d, a, b, c, w3[1], MD4C00, MD4S01);
    MD4_STEP (MD4_Fo, c, d, a, b, w3[2], MD4C00, MD4S02);
    MD4_STEP (MD4_Fo, b, c, d, a, w3[3], MD4C00, MD4S03);

    MD4_STEP (MD4_Go, a, b, c, d, w0[0], MD4C01, MD4S10);
    MD4_STEP (MD4_Go, d, a, b, c, w1[0], MD4C01, MD4S11);
    MD4_STEP (MD4_Go, c, d, a, b, w2[0], MD4C01, MD4S12);
    MD4_STEP (MD4_Go, b, c, d, a, w3[0], MD4C01, MD4S13);
    MD4_STEP (MD4_Go, a, b, c, d, w0[1], MD4C01, MD4S10);
    MD4_STEP (MD4_Go, d, a, b, c, w1[1], MD4C01, MD4S11);
    MD4_STEP (MD4_Go, c, d, a, b, w2[1], MD4C01, MD4S12);
    MD4_STEP (MD4_Go, b, c, d, a, w3[1], MD4C01, MD4S13);
    MD4_STEP (MD4_Go, a, b, c, d, w0[2], MD4C01, MD4S10);
    MD4_STEP (MD4_Go, d, a, b, c, w1[2], MD4C01, MD4S11);
    MD4_STEP (MD4_Go, c, d, a, b, w2[2], MD4C01, MD4S12);
    MD4_STEP (MD4_Go, b, c, d, a, w3[2], MD4C01, MD4S13);
    MD4_STEP (MD4_Go, a, b, c, d, w0[3], MD4C01, MD4S10);
    MD4_STEP (MD4_Go, d, a, b, c, w1[3], MD4C01, MD4S11);
    MD4_STEP (MD4_Go, c, d, a, b, w2[3], MD4C01, MD4S12);
    MD4_STEP (MD4_Go, b, c, d, a, w3[3], MD4C01, MD4S13);

    MD4_STEP (MD4_H , a, b, c, d, w0[0], MD4C02, MD4S20);
    MD4_STEP (MD4_H , d, a, b, c, w2[0], MD4C02, MD4S21);
    MD4_STEP (MD4_H , c, d, a, b, w1[0], MD4C02, MD4S22);
    MD4_STEP (MD4_H , b, c, d, a, w3[0], MD4C02, MD4S23);
    MD4_STEP (MD4_H , a, b, c, d, w0[2], MD4C02, MD4S20);
    MD4_STEP (MD4_H , d, a, b, c, w2[2], MD4C02, MD4S21);
    MD4_STEP (MD4_H , c, d, a, b, w1[2], MD4C02, MD4S22);
    MD4_STEP (MD4_H , b, c, d, a, w3[2], MD4C02, MD4S23);
    MD4_STEP (MD4_H , a, b, c, d, w0[1], MD4C02, MD4S20);
    MD4_STEP (MD4_H , d, a, b, c, w2[1], MD4C02, MD4S21);
    MD4_STEP (MD4_H , c, d, a, b, w1[1], MD4C02, MD4S22);
    MD4_STEP (MD4_H , b, c, d, a, w3[1], MD4C02, MD4S23);
    MD4_STEP (MD4_H , a, b, c, d, w0[3], MD4C02, MD4S20);
    MD4_STEP (MD4_H , d, a, b, c, w2[3], MD4C02, MD4S21);
    MD4_STEP (MD4_H , c, d, a, b, w1[3], MD4C02, MD4S22);
    MD4_STEP (MD4_H , b, c, d, a, w3[3], MD4C02, MD4S23);

    const u32x r0 = a;
    const u32x r1 = d;
    const u32x r2 = c;
    const u32x r3 = b;

    #include VECT_COMPARE_S
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m00900_s08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m00900_s16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}
