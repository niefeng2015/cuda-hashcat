#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _DES_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 0
#define DGST_R1 1
#define DGST_R2 2
#define DGST_R3 3

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"
#include "include/rp_gpu.h"
#include "rp_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_S "check_single_vect1_comp4.c"
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_S "check_single_vect2_comp4.c"
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

#ifdef  VECT_SIZE4
#define VECT_COMPARE_S "check_single_vect2_comp4.c"
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

#define PERM_OP(a,b,tt,n,m) \
{                           \
  tt = a >> n;              \
  tt = tt ^ b;              \
  tt = tt & m;              \
  b = b ^ tt;               \
  tt = tt << n;             \
  a = a ^ tt;               \
}

#define HPERM_OP(a,tt,n,m)  \
{                           \
  tt = a << (16 + n);       \
  tt = tt ^ a;              \
  tt = tt & m;              \
  a  = a ^ tt;              \
  tt = tt >> (16 + n);      \
  a  = a ^ tt;              \
}

__device__ __constant__ u32 c_SPtrans[8][64] =
{
  /* nibble 0 */
  0x02080800, 0x00080000, 0x02000002, 0x02080802,
  0x02000000, 0x00080802, 0x00080002, 0x02000002,
  0x00080802, 0x02080800, 0x02080000, 0x00000802,
  0x02000802, 0x02000000, 0x00000000, 0x00080002,
  0x00080000, 0x00000002, 0x02000800, 0x00080800,
  0x02080802, 0x02080000, 0x00000802, 0x02000800,
  0x00000002, 0x00000800, 0x00080800, 0x02080002,
  0x00000800, 0x02000802, 0x02080002, 0x00000000,
  0x00000000, 0x02080802, 0x02000800, 0x00080002,
  0x02080800, 0x00080000, 0x00000802, 0x02000800,
  0x02080002, 0x00000800, 0x00080800, 0x02000002,
  0x00080802, 0x00000002, 0x02000002, 0x02080000,
  0x02080802, 0x00080800, 0x02080000, 0x02000802,
  0x02000000, 0x00000802, 0x00080002, 0x00000000,
  0x00080000, 0x02000000, 0x02000802, 0x02080800,
  0x00000002, 0x02080002, 0x00000800, 0x00080802,
  /* nibble 1 */
  0x40108010, 0x00000000, 0x00108000, 0x40100000,
  0x40000010, 0x00008010, 0x40008000, 0x00108000,
  0x00008000, 0x40100010, 0x00000010, 0x40008000,
  0x00100010, 0x40108000, 0x40100000, 0x00000010,
  0x00100000, 0x40008010, 0x40100010, 0x00008000,
  0x00108010, 0x40000000, 0x00000000, 0x00100010,
  0x40008010, 0x00108010, 0x40108000, 0x40000010,
  0x40000000, 0x00100000, 0x00008010, 0x40108010,
  0x00100010, 0x40108000, 0x40008000, 0x00108010,
  0x40108010, 0x00100010, 0x40000010, 0x00000000,
  0x40000000, 0x00008010, 0x00100000, 0x40100010,
  0x00008000, 0x40000000, 0x00108010, 0x40008010,
  0x40108000, 0x00008000, 0x00000000, 0x40000010,
  0x00000010, 0x40108010, 0x00108000, 0x40100000,
  0x40100010, 0x00100000, 0x00008010, 0x40008000,
  0x40008010, 0x00000010, 0x40100000, 0x00108000,
  /* nibble 2 */
  0x04000001, 0x04040100, 0x00000100, 0x04000101,
  0x00040001, 0x04000000, 0x04000101, 0x00040100,
  0x04000100, 0x00040000, 0x04040000, 0x00000001,
  0x04040101, 0x00000101, 0x00000001, 0x04040001,
  0x00000000, 0x00040001, 0x04040100, 0x00000100,
  0x00000101, 0x04040101, 0x00040000, 0x04000001,
  0x04040001, 0x04000100, 0x00040101, 0x04040000,
  0x00040100, 0x00000000, 0x04000000, 0x00040101,
  0x04040100, 0x00000100, 0x00000001, 0x00040000,
  0x00000101, 0x00040001, 0x04040000, 0x04000101,
  0x00000000, 0x04040100, 0x00040100, 0x04040001,
  0x00040001, 0x04000000, 0x04040101, 0x00000001,
  0x00040101, 0x04000001, 0x04000000, 0x04040101,
  0x00040000, 0x04000100, 0x04000101, 0x00040100,
  0x04000100, 0x00000000, 0x04040001, 0x00000101,
  0x04000001, 0x00040101, 0x00000100, 0x04040000,
  /* nibble 3 */
  0x00401008, 0x10001000, 0x00000008, 0x10401008,
  0x00000000, 0x10400000, 0x10001008, 0x00400008,
  0x10401000, 0x10000008, 0x10000000, 0x00001008,
  0x10000008, 0x00401008, 0x00400000, 0x10000000,
  0x10400008, 0x00401000, 0x00001000, 0x00000008,
  0x00401000, 0x10001008, 0x10400000, 0x00001000,
  0x00001008, 0x00000000, 0x00400008, 0x10401000,
  0x10001000, 0x10400008, 0x10401008, 0x00400000,
  0x10400008, 0x00001008, 0x00400000, 0x10000008,
  0x00401000, 0x10001000, 0x00000008, 0x10400000,
  0x10001008, 0x00000000, 0x00001000, 0x00400008,
  0x00000000, 0x10400008, 0x10401000, 0x00001000,
  0x10000000, 0x10401008, 0x00401008, 0x00400000,
  0x10401008, 0x00000008, 0x10001000, 0x00401008,
  0x00400008, 0x00401000, 0x10400000, 0x10001008,
  0x00001008, 0x10000000, 0x10000008, 0x10401000,
  /* nibble 4 */
  0x08000000, 0x00010000, 0x00000400, 0x08010420,
  0x08010020, 0x08000400, 0x00010420, 0x08010000,
  0x00010000, 0x00000020, 0x08000020, 0x00010400,
  0x08000420, 0x08010020, 0x08010400, 0x00000000,
  0x00010400, 0x08000000, 0x00010020, 0x00000420,
  0x08000400, 0x00010420, 0x00000000, 0x08000020,
  0x00000020, 0x08000420, 0x08010420, 0x00010020,
  0x08010000, 0x00000400, 0x00000420, 0x08010400,
  0x08010400, 0x08000420, 0x00010020, 0x08010000,
  0x00010000, 0x00000020, 0x08000020, 0x08000400,
  0x08000000, 0x00010400, 0x08010420, 0x00000000,
  0x00010420, 0x08000000, 0x00000400, 0x00010020,
  0x08000420, 0x00000400, 0x00000000, 0x08010420,
  0x08010020, 0x08010400, 0x00000420, 0x00010000,
  0x00010400, 0x08010020, 0x08000400, 0x00000420,
  0x00000020, 0x00010420, 0x08010000, 0x08000020,
  /* nibble 5 */
  0x80000040, 0x00200040, 0x00000000, 0x80202000,
  0x00200040, 0x00002000, 0x80002040, 0x00200000,
  0x00002040, 0x80202040, 0x00202000, 0x80000000,
  0x80002000, 0x80000040, 0x80200000, 0x00202040,
  0x00200000, 0x80002040, 0x80200040, 0x00000000,
  0x00002000, 0x00000040, 0x80202000, 0x80200040,
  0x80202040, 0x80200000, 0x80000000, 0x00002040,
  0x00000040, 0x00202000, 0x00202040, 0x80002000,
  0x00002040, 0x80000000, 0x80002000, 0x00202040,
  0x80202000, 0x00200040, 0x00000000, 0x80002000,
  0x80000000, 0x00002000, 0x80200040, 0x00200000,
  0x00200040, 0x80202040, 0x00202000, 0x00000040,
  0x80202040, 0x00202000, 0x00200000, 0x80002040,
  0x80000040, 0x80200000, 0x00202040, 0x00000000,
  0x00002000, 0x80000040, 0x80002040, 0x80202000,
  0x80200000, 0x00002040, 0x00000040, 0x80200040,
  /* nibble 6 */
  0x00004000, 0x00000200, 0x01000200, 0x01000004,
  0x01004204, 0x00004004, 0x00004200, 0x00000000,
  0x01000000, 0x01000204, 0x00000204, 0x01004000,
  0x00000004, 0x01004200, 0x01004000, 0x00000204,
  0x01000204, 0x00004000, 0x00004004, 0x01004204,
  0x00000000, 0x01000200, 0x01000004, 0x00004200,
  0x01004004, 0x00004204, 0x01004200, 0x00000004,
  0x00004204, 0x01004004, 0x00000200, 0x01000000,
  0x00004204, 0x01004000, 0x01004004, 0x00000204,
  0x00004000, 0x00000200, 0x01000000, 0x01004004,
  0x01000204, 0x00004204, 0x00004200, 0x00000000,
  0x00000200, 0x01000004, 0x00000004, 0x01000200,
  0x00000000, 0x01000204, 0x01000200, 0x00004200,
  0x00000204, 0x00004000, 0x01004204, 0x01000000,
  0x01004200, 0x00000004, 0x00004004, 0x01004204,
  0x01000004, 0x01004200, 0x01004000, 0x00004004,
  /* nibble 7 */
  0x20800080, 0x20820000, 0x00020080, 0x00000000,
  0x20020000, 0x00800080, 0x20800000, 0x20820080,
  0x00000080, 0x20000000, 0x00820000, 0x00020080,
  0x00820080, 0x20020080, 0x20000080, 0x20800000,
  0x00020000, 0x00820080, 0x00800080, 0x20020000,
  0x20820080, 0x20000080, 0x00000000, 0x00820000,
  0x20000000, 0x00800000, 0x20020080, 0x20800080,
  0x00800000, 0x00020000, 0x20820000, 0x00000080,
  0x00800000, 0x00020000, 0x20000080, 0x20820080,
  0x00020080, 0x20000000, 0x00000000, 0x00820000,
  0x20800080, 0x20020080, 0x20020000, 0x00800080,
  0x20820000, 0x00000080, 0x00800080, 0x20020000,
  0x20820080, 0x00800000, 0x20800000, 0x20000080,
  0x00820000, 0x00020080, 0x20020080, 0x20800000,
  0x00000080, 0x20820000, 0x00820080, 0x00000000,
  0x20000000, 0x20800080, 0x00020000, 0x00820080,
};

__device__ __constant__ u32 c_skb[8][64] =
{
  /* for C bits (numbered as per FIPS 46) 1 2 3 4 5 6 */
  0x00000000, 0x00000010, 0x20000000, 0x20000010,
  0x00010000, 0x00010010, 0x20010000, 0x20010010,
  0x00000800, 0x00000810, 0x20000800, 0x20000810,
  0x00010800, 0x00010810, 0x20010800, 0x20010810,
  0x00000020, 0x00000030, 0x20000020, 0x20000030,
  0x00010020, 0x00010030, 0x20010020, 0x20010030,
  0x00000820, 0x00000830, 0x20000820, 0x20000830,
  0x00010820, 0x00010830, 0x20010820, 0x20010830,
  0x00080000, 0x00080010, 0x20080000, 0x20080010,
  0x00090000, 0x00090010, 0x20090000, 0x20090010,
  0x00080800, 0x00080810, 0x20080800, 0x20080810,
  0x00090800, 0x00090810, 0x20090800, 0x20090810,
  0x00080020, 0x00080030, 0x20080020, 0x20080030,
  0x00090020, 0x00090030, 0x20090020, 0x20090030,
  0x00080820, 0x00080830, 0x20080820, 0x20080830,
  0x00090820, 0x00090830, 0x20090820, 0x20090830,
  /* for C bits (numbered as per FIPS 46) 7 8 10 11 12 13 */
  0x00000000, 0x02000000, 0x00002000, 0x02002000,
  0x00200000, 0x02200000, 0x00202000, 0x02202000,
  0x00000004, 0x02000004, 0x00002004, 0x02002004,
  0x00200004, 0x02200004, 0x00202004, 0x02202004,
  0x00000400, 0x02000400, 0x00002400, 0x02002400,
  0x00200400, 0x02200400, 0x00202400, 0x02202400,
  0x00000404, 0x02000404, 0x00002404, 0x02002404,
  0x00200404, 0x02200404, 0x00202404, 0x02202404,
  0x10000000, 0x12000000, 0x10002000, 0x12002000,
  0x10200000, 0x12200000, 0x10202000, 0x12202000,
  0x10000004, 0x12000004, 0x10002004, 0x12002004,
  0x10200004, 0x12200004, 0x10202004, 0x12202004,
  0x10000400, 0x12000400, 0x10002400, 0x12002400,
  0x10200400, 0x12200400, 0x10202400, 0x12202400,
  0x10000404, 0x12000404, 0x10002404, 0x12002404,
  0x10200404, 0x12200404, 0x10202404, 0x12202404,
  /* for C bits (numbered as per FIPS 46) 14 15 16 17 19 20 */
  0x00000000, 0x00000001, 0x00040000, 0x00040001,
  0x01000000, 0x01000001, 0x01040000, 0x01040001,
  0x00000002, 0x00000003, 0x00040002, 0x00040003,
  0x01000002, 0x01000003, 0x01040002, 0x01040003,
  0x00000200, 0x00000201, 0x00040200, 0x00040201,
  0x01000200, 0x01000201, 0x01040200, 0x01040201,
  0x00000202, 0x00000203, 0x00040202, 0x00040203,
  0x01000202, 0x01000203, 0x01040202, 0x01040203,
  0x08000000, 0x08000001, 0x08040000, 0x08040001,
  0x09000000, 0x09000001, 0x09040000, 0x09040001,
  0x08000002, 0x08000003, 0x08040002, 0x08040003,
  0x09000002, 0x09000003, 0x09040002, 0x09040003,
  0x08000200, 0x08000201, 0x08040200, 0x08040201,
  0x09000200, 0x09000201, 0x09040200, 0x09040201,
  0x08000202, 0x08000203, 0x08040202, 0x08040203,
  0x09000202, 0x09000203, 0x09040202, 0x09040203,
  /* for C bits (numbered as per FIPS 46) 21 23 24 26 27 28 */
  0x00000000, 0x00100000, 0x00000100, 0x00100100,
  0x00000008, 0x00100008, 0x00000108, 0x00100108,
  0x00001000, 0x00101000, 0x00001100, 0x00101100,
  0x00001008, 0x00101008, 0x00001108, 0x00101108,
  0x04000000, 0x04100000, 0x04000100, 0x04100100,
  0x04000008, 0x04100008, 0x04000108, 0x04100108,
  0x04001000, 0x04101000, 0x04001100, 0x04101100,
  0x04001008, 0x04101008, 0x04001108, 0x04101108,
  0x00020000, 0x00120000, 0x00020100, 0x00120100,
  0x00020008, 0x00120008, 0x00020108, 0x00120108,
  0x00021000, 0x00121000, 0x00021100, 0x00121100,
  0x00021008, 0x00121008, 0x00021108, 0x00121108,
  0x04020000, 0x04120000, 0x04020100, 0x04120100,
  0x04020008, 0x04120008, 0x04020108, 0x04120108,
  0x04021000, 0x04121000, 0x04021100, 0x04121100,
  0x04021008, 0x04121008, 0x04021108, 0x04121108,
  /* for D bits (numbered as per FIPS 46) 1 2 3 4 5 6 */
  0x00000000, 0x10000000, 0x00010000, 0x10010000,
  0x00000004, 0x10000004, 0x00010004, 0x10010004,
  0x20000000, 0x30000000, 0x20010000, 0x30010000,
  0x20000004, 0x30000004, 0x20010004, 0x30010004,
  0x00100000, 0x10100000, 0x00110000, 0x10110000,
  0x00100004, 0x10100004, 0x00110004, 0x10110004,
  0x20100000, 0x30100000, 0x20110000, 0x30110000,
  0x20100004, 0x30100004, 0x20110004, 0x30110004,
  0x00001000, 0x10001000, 0x00011000, 0x10011000,
  0x00001004, 0x10001004, 0x00011004, 0x10011004,
  0x20001000, 0x30001000, 0x20011000, 0x30011000,
  0x20001004, 0x30001004, 0x20011004, 0x30011004,
  0x00101000, 0x10101000, 0x00111000, 0x10111000,
  0x00101004, 0x10101004, 0x00111004, 0x10111004,
  0x20101000, 0x30101000, 0x20111000, 0x30111000,
  0x20101004, 0x30101004, 0x20111004, 0x30111004,
  /* for D bits (numbered as per FIPS 46) 8 9 11 12 13 14 */
  0x00000000, 0x08000000, 0x00000008, 0x08000008,
  0x00000400, 0x08000400, 0x00000408, 0x08000408,
  0x00020000, 0x08020000, 0x00020008, 0x08020008,
  0x00020400, 0x08020400, 0x00020408, 0x08020408,
  0x00000001, 0x08000001, 0x00000009, 0x08000009,
  0x00000401, 0x08000401, 0x00000409, 0x08000409,
  0x00020001, 0x08020001, 0x00020009, 0x08020009,
  0x00020401, 0x08020401, 0x00020409, 0x08020409,
  0x02000000, 0x0A000000, 0x02000008, 0x0A000008,
  0x02000400, 0x0A000400, 0x02000408, 0x0A000408,
  0x02020000, 0x0A020000, 0x02020008, 0x0A020008,
  0x02020400, 0x0A020400, 0x02020408, 0x0A020408,
  0x02000001, 0x0A000001, 0x02000009, 0x0A000009,
  0x02000401, 0x0A000401, 0x02000409, 0x0A000409,
  0x02020001, 0x0A020001, 0x02020009, 0x0A020009,
  0x02020401, 0x0A020401, 0x02020409, 0x0A020409,
  /* for D bits (numbered as per FIPS 46) 16 17 18 19 20 21 */
  0x00000000, 0x00000100, 0x00080000, 0x00080100,
  0x01000000, 0x01000100, 0x01080000, 0x01080100,
  0x00000010, 0x00000110, 0x00080010, 0x00080110,
  0x01000010, 0x01000110, 0x01080010, 0x01080110,
  0x00200000, 0x00200100, 0x00280000, 0x00280100,
  0x01200000, 0x01200100, 0x01280000, 0x01280100,
  0x00200010, 0x00200110, 0x00280010, 0x00280110,
  0x01200010, 0x01200110, 0x01280010, 0x01280110,
  0x00000200, 0x00000300, 0x00080200, 0x00080300,
  0x01000200, 0x01000300, 0x01080200, 0x01080300,
  0x00000210, 0x00000310, 0x00080210, 0x00080310,
  0x01000210, 0x01000310, 0x01080210, 0x01080310,
  0x00200200, 0x00200300, 0x00280200, 0x00280300,
  0x01200200, 0x01200300, 0x01280200, 0x01280300,
  0x00200210, 0x00200310, 0x00280210, 0x00280310,
  0x01200210, 0x01200310, 0x01280210, 0x01280310,
  /* for D bits (numbered as per FIPS 46) 22 23 24 25 27 28 */
  0x00000000, 0x04000000, 0x00040000, 0x04040000,
  0x00000002, 0x04000002, 0x00040002, 0x04040002,
  0x00002000, 0x04002000, 0x00042000, 0x04042000,
  0x00002002, 0x04002002, 0x00042002, 0x04042002,
  0x00000020, 0x04000020, 0x00040020, 0x04040020,
  0x00000022, 0x04000022, 0x00040022, 0x04040022,
  0x00002020, 0x04002020, 0x00042020, 0x04042020,
  0x00002022, 0x04002022, 0x00042022, 0x04042022,
  0x00000800, 0x04000800, 0x00040800, 0x04040800,
  0x00000802, 0x04000802, 0x00040802, 0x04040802,
  0x00002800, 0x04002800, 0x00042800, 0x04042800,
  0x00002802, 0x04002802, 0x00042802, 0x04042802,
  0x00000820, 0x04000820, 0x00040820, 0x04040820,
  0x00000822, 0x04000822, 0x00040822, 0x04040822,
  0x00002820, 0x04002820, 0x00042820, 0x04042820,
  0x00002822, 0x04002822, 0x00042822, 0x04042822
};

#define LM_IV_0_IP_RR3 0x2400b807
#define LM_IV_1_IP_RR3 0xaa190747

#ifdef VECT_SIZE1
#define BOX(i,n,S) u32x ((S)[(n)][(i)])
#endif

#ifdef VECT_SIZE2
#define BOX(i,n,S) u32x ((S)[(n)][(i).x], (S)[(n)][(i).y])
#endif

__device__ static void _des_crypt_encrypt (u32x iv[2], u32x data[2], u32x Kc[16], u32x Kd[16], u32 s_SPtrans[8][64])
{
  asm (".reg .u8 c0, c1, c2, c3, c4, c5, c6, c7;");

  u32x tt;

  u32x r = data[0];
  u32x l = data[1];

  for (int i = 0; i < 16; i++)
  {
    u32x u = Kc[i] ^ r;
    u32x t = Kd[i] ^ rotl32 (r, 28u);

    u = (u >> 2) & 0x3f3f3f3f;
    t = (t >> 2) & 0x3f3f3f3f;

    u32 u0, u1, u2, u3;
    u32 t0, t1, t2, t3;

    asm
    (
      "mov.b32 {c0, c1, c2, c3}, %8;"
      "mov.b32 {c4, c5, c6, c7}, %9;"
      "cvt.u32.u8 %0, c0;"
      "cvt.u32.u8 %1, c1;"
      "cvt.u32.u8 %2, c2;"
      "cvt.u32.u8 %3, c3;"
      "cvt.u32.u8 %4, c4;"
      "cvt.u32.u8 %5, c5;"
      "cvt.u32.u8 %6, c6;"
      "cvt.u32.u8 %7, c7;"

      : "=r"(u0), "=r"(u1), "=r"(u2), "=r"(u3),
        "=r"(t0), "=r"(t1), "=r"(t2), "=r"(t3)
      : "r"(u),
        "r"(t)
    );

    l ^= BOX (u0, 0, s_SPtrans)
       | BOX (u1, 2, s_SPtrans)
       | BOX (u2, 4, s_SPtrans)
       | BOX (u3, 6, s_SPtrans)
       | BOX (t0, 1, s_SPtrans)
       | BOX (t1, 3, s_SPtrans)
       | BOX (t2, 5, s_SPtrans)
       | BOX (t3, 7, s_SPtrans);

    tt = l;
    l  = r;
    r  = tt;
  }

  iv[0] = rotl32 (l, 29);
  iv[1] = rotl32 (r, 29);
}

__device__ static void _des_crypt_keysetup (u32x c, u32x d, u32x Kc[16], u32x Kd[16], u32 s_skb[8][64])
{
  u32x tt;

  PERM_OP  (d, c, tt, 4, 0x0f0f0f0f);
  HPERM_OP (c,    tt, 2, 0xcccc0000);
  HPERM_OP (d,    tt, 2, 0xcccc0000);
  PERM_OP  (d, c, tt, 1, 0x55555555);
  PERM_OP  (c, d, tt, 8, 0x00ff00ff);
  PERM_OP  (d, c, tt, 1, 0x55555555);

  d = ((d & 0x000000ff) << 16)
    | ((d & 0x0000ff00) <<  0)
    | ((d & 0x00ff0000) >> 16)
    | ((c & 0xf0000000) >>  4);

  c = c & 0x0fffffff;

  for (u32 i = 0; i < 16; i++)
  {
    if ((i < 2) || (i == 8) || (i == 15))
    {
      c = ((c >> 1) | (c << 27));
      d = ((d >> 1) | (d << 27));
    }
    else
    {
      c = ((c >> 2) | (c << 26));
      d = ((d >> 2) | (d << 26));
    }

    c = c & 0x0fffffff;
    d = d & 0x0fffffff;

    const u32x c00 = (c >>  0) & 0x0000003f;
    const u32x c06 = (c >>  6) & 0x00383003;
    const u32x c07 = (c >>  7) & 0x0000003c;
    const u32x c13 = (c >> 13) & 0x0000060f;
    const u32x c20 = (c >> 20) & 0x00000001;

    u32x s = BOX (((c00 >>  0) & 0xff), 0, s_skb)
            | BOX (((c06 >>  0) & 0xff)
                  |((c07 >>  0) & 0xff), 1, s_skb)
            | BOX (((c13 >>  0) & 0xff)
                  |((c06 >>  8) & 0xff), 2, s_skb)
            | BOX (((c20 >>  0) & 0xff)
                  |((c13 >>  8) & 0xff)
                  |((c06 >> 16) & 0xff), 3, s_skb);

    const u32x d00 = (d >>  0) & 0x00003c3f;
    const u32x d07 = (d >>  7) & 0x00003f03;
    const u32x d21 = (d >> 21) & 0x0000000f;
    const u32x d22 = (d >> 22) & 0x00000030;

    u32x t = BOX (((d00 >>  0) & 0xff), 4, s_skb)
            | BOX (((d07 >>  0) & 0xff)
                  |((d00 >>  8) & 0xff), 5, s_skb)
            | BOX (((d07 >>  8) & 0xff), 6, s_skb)
            | BOX (((d21 >>  0) & 0xff)
                  |((d22 >>  0) & 0xff), 7, s_skb);

    #if __CUDA_ARCH__ >= 200
    Kc[i] = __byte_perm (s, t, 0x5410);
    Kd[i] = __byte_perm (s, t, 0x7632);
    #else
    Kc[i] = ((t << 16) | (s & 0x0000ffff));
    Kd[i] = ((s >> 16) | (t & 0xffff0000));
    #endif

    Kc[i] = rotl32 (Kc[i], 2u);
    Kd[i] = rotl32 (Kd[i], 2u);
  }
}

__device__ static void transform_netntlmv1_key (const u32x w0, const u32x w1, u32x out[2])
{
  u32x t[8];

  t[0] = (w0 >>  0) & 0xff;
  t[1] = (w0 >>  8) & 0xff;
  t[2] = (w0 >> 16) & 0xff;
  t[3] = (w0 >> 24) & 0xff;
  t[4] = (w1 >>  0) & 0xff;
  t[5] = (w1 >>  8) & 0xff;
  t[6] = (w1 >> 16) & 0xff;
  t[7] = (w1 >> 24) & 0xff;

  u32x k[8];

  k[0] =               (t[0] >> 0);
  k[1] = (t[0] << 7) | (t[1] >> 1);
  k[2] = (t[1] << 6) | (t[2] >> 2);
  k[3] = (t[2] << 5) | (t[3] >> 3);
  k[4] = (t[3] << 4) | (t[4] >> 4);
  k[5] = (t[4] << 3) | (t[5] >> 5);
  k[6] = (t[5] << 2) | (t[6] >> 6);
  k[7] = (t[6] << 1);

  out[0] = ((k[0] & 0xff) <<  0)
         | ((k[1] & 0xff) <<  8)
         | ((k[2] & 0xff) << 16)
         | ((k[3] & 0xff) << 24);

  out[1] = ((k[4] & 0xff) <<  0)
         | ((k[5] & 0xff) <<  8)
         | ((k[6] & 0xff) << 16)
         | ((k[7] & 0xff) << 24);
}

__device__ __constant__ gpu_rule_t c_rules[1024];

extern "C" __global__ void __launch_bounds__ (256, 1) m03000_m04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  u32x pw_buf[4];

  pw_buf[0] = pws[gid].i[ 0];
  pw_buf[1] = pws[gid].i[ 1];
  pw_buf[2] = 0;
  pw_buf[3] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * sbox, kbox
   */

  __shared__ u32 s_SPtrans[8][64];
  __shared__ u32 s_skb[8][64];

  if (lid < 64)
  {
    s_SPtrans[0][lid] = c_SPtrans[0][lid];
    s_SPtrans[1][lid] = c_SPtrans[1][lid];
    s_SPtrans[2][lid] = c_SPtrans[2][lid];
    s_SPtrans[3][lid] = c_SPtrans[3][lid];
    s_SPtrans[4][lid] = c_SPtrans[4][lid];
    s_SPtrans[5][lid] = c_SPtrans[5][lid];
    s_SPtrans[6][lid] = c_SPtrans[6][lid];
    s_SPtrans[7][lid] = c_SPtrans[7][lid];

    s_skb[0][lid] = c_skb[0][lid];
    s_skb[1][lid] = c_skb[1][lid];
    s_skb[2][lid] = c_skb[2][lid];
    s_skb[3][lid] = c_skb[3][lid];
    s_skb[4][lid] = c_skb[4][lid];
    s_skb[5][lid] = c_skb[5][lid];
    s_skb[6][lid] = c_skb[6][lid];
    s_skb[7][lid] = c_skb[7][lid];
  }

  __syncthreads ();

  if (gid >= gid_max) return;

  /**
   * main
   */

  for (u32 il_pos = 0; il_pos < rules_cnt; il_pos++)
  {
    u32x w0[4];

    w0[0] = pw_buf[0];
    w0[1] = pw_buf[1];
    w0[2] = pw_buf[2];
    w0[3] = pw_buf[3];

    u32x w1[4];

    w1[0] = 0;
    w1[1] = 0;
    w1[2] = 0;
    w1[3] = 0;

    u32x w2[4];

    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = 0;

    u32 out_len = apply_rules (c_rules[il_pos].cmds, w0, w1, pw_len);

    out_len = (out_len >= 7) ? 7 : out_len;

    u32x key[2];

    transform_netntlmv1_key (w0[0], w0[1], key);

    const u32x c = key[0];
    const u32x d = key[1];

    u32x Kc[16];
    u32x Kd[16];

    _des_crypt_keysetup (c, d, Kc, Kd, s_skb);

    u32x data[2];

    data[0] = LM_IV_0_IP_RR3;
    data[1] = LM_IV_1_IP_RR3;

    u32x iv[2];

    _des_crypt_encrypt (iv, data, Kc, Kd, s_SPtrans);

    const u32x r0 = iv[0];
    const u32x r1 = iv[1];
    const u32x r2 = 0;
    const u32x r3 = 0;

    #include VECT_COMPARE_M
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m03000_m08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m03000_m16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m03000_s04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  u32x pw_buf[4];

  pw_buf[0] = pws[gid].i[ 0];
  pw_buf[1] = pws[gid].i[ 1];
  pw_buf[2] = 0;
  pw_buf[3] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * sbox, kbox
   */

  __shared__ u32 s_SPtrans[8][64];
  __shared__ u32 s_skb[8][64];

  if (lid < 64)
  {
    s_SPtrans[0][lid] = c_SPtrans[0][lid];
    s_SPtrans[1][lid] = c_SPtrans[1][lid];
    s_SPtrans[2][lid] = c_SPtrans[2][lid];
    s_SPtrans[3][lid] = c_SPtrans[3][lid];
    s_SPtrans[4][lid] = c_SPtrans[4][lid];
    s_SPtrans[5][lid] = c_SPtrans[5][lid];
    s_SPtrans[6][lid] = c_SPtrans[6][lid];
    s_SPtrans[7][lid] = c_SPtrans[7][lid];

    s_skb[0][lid] = c_skb[0][lid];
    s_skb[1][lid] = c_skb[1][lid];
    s_skb[2][lid] = c_skb[2][lid];
    s_skb[3][lid] = c_skb[3][lid];
    s_skb[4][lid] = c_skb[4][lid];
    s_skb[5][lid] = c_skb[5][lid];
    s_skb[6][lid] = c_skb[6][lid];
    s_skb[7][lid] = c_skb[7][lid];
  }

  __syncthreads ();

  if (gid >= gid_max) return;

  /**
   * digest
   */

  const u32 search[4] =
  {
    digests_buf[digests_offset].digest_buf[DGST_R0],
    digests_buf[digests_offset].digest_buf[DGST_R1],
    digests_buf[digests_offset].digest_buf[DGST_R2],
    digests_buf[digests_offset].digest_buf[DGST_R3]
  };

  /**
   * main
   */

  for (u32 il_pos = 0; il_pos < rules_cnt; il_pos++)
  {
    u32x w0[4];

    w0[0] = pw_buf[0];
    w0[1] = pw_buf[1];
    w0[2] = pw_buf[2];
    w0[3] = pw_buf[3];

    u32x w1[4];

    w1[0] = 0;
    w1[1] = 0;
    w1[2] = 0;
    w1[3] = 0;

    u32x w2[4];

    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = 0;

    u32 out_len = apply_rules (c_rules[il_pos].cmds, w0, w1, pw_len);

    out_len = (out_len >= 7) ? 7 : out_len;

    u32x key[2];

    transform_netntlmv1_key (w0[0], w0[1], key);

    const u32x c = key[0];
    const u32x d = key[1];

    u32x Kc[16];
    u32x Kd[16];

    _des_crypt_keysetup (c, d, Kc, Kd, s_skb);

    u32x data[2];

    data[0] = LM_IV_0_IP_RR3;
    data[1] = LM_IV_1_IP_RR3;

    u32x iv[2];

    _des_crypt_encrypt (iv, data, Kc, Kd, s_SPtrans);

    const u32x r0 = iv[0];
    const u32x r1 = iv[1];
    const u32x r2 = 0;
    const u32x r3 = 0;

    #include VECT_COMPARE_S
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m03000_s08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m03000_s16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}
