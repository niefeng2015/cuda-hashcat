#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _SHA1_MD5_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 3
#define DGST_R1 4
#define DGST_R2 2
#define DGST_R3 1

#include "include/kernel_functions.c"
#undef _MD5_
#include "types_nv.c"
#include "common_nv.c"
#include "include/rp_gpu.h"
#include "rp_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_S "check_single_vect1_comp4.c"
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_S "check_single_vect2_comp4.c"
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

#ifdef  VECT_SIZE4
#define VECT_COMPARE_S "check_single_vect4_comp4.c"
#define VECT_COMPARE_M "check_multi_vect4_comp4.c"
#endif

#ifdef VECT_SIZE1
#define uint_to_hex_lower8_le(i) l_bin2asc[(i)]
#endif

#ifdef VECT_SIZE2
#define uint_to_hex_lower8_le(i) u32x (l_bin2asc[(i).x], l_bin2asc[(i).y])
#endif

#ifdef VECT_SIZE4
#define uint_to_hex_lower8_le(i) u32x (l_bin2asc[(i).x], l_bin2asc[(i).y], l_bin2asc[(i).z], l_bin2asc[(i).w])
#endif

__device__ __constant__ char c_bin2asc[16] = { '0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'a', 'b', 'c', 'd', 'e', 'f' };

__device__ __shared__ short l_bin2asc[256];

__device__ __constant__ gpu_rule_t c_rules[1024];

extern "C" __global__ void __launch_bounds__ (256, 1) m04700_m04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  u32x pw_buf0[4];

  pw_buf0[0] = pws[gid].i[ 0];
  pw_buf0[1] = pws[gid].i[ 1];
  pw_buf0[2] = pws[gid].i[ 2];
  pw_buf0[3] = pws[gid].i[ 3];

  u32x pw_buf1[4];

  pw_buf1[0] = pws[gid].i[ 4];
  pw_buf1[1] = pws[gid].i[ 5];
  pw_buf1[2] = pws[gid].i[ 6];
  pw_buf1[3] = pws[gid].i[ 7];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * bin2asc table
   */

  l_bin2asc[lid] = c_bin2asc[(lid >> 0) & 15] << 0
                 | c_bin2asc[(lid >> 4) & 15] << 8;

  __syncthreads ();

  if (gid >= gid_max) return;

  /**
   * loop
   */

  for (u32 il_pos = 0; il_pos < rules_cnt; il_pos++)
  {
    u32x w0[4];

    w0[0] = pw_buf0[0];
    w0[1] = pw_buf0[1];
    w0[2] = pw_buf0[2];
    w0[3] = pw_buf0[3];

    u32x w1[4];

    w1[0] = pw_buf1[0];
    w1[1] = pw_buf1[1];
    w1[2] = pw_buf1[2];
    w1[3] = pw_buf1[3];

    u32x w2[4];

    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = pw_len * 8;
    w3[3] = 0;

    const u32 out_len = apply_rules (c_rules[il_pos].cmds, w0, w1, pw_len);

    append_0x80_2 (w0, w1, out_len);

    /**
     * md5
     */

    u32x a = MD5M_A;
    u32x b = MD5M_B;
    u32x c = MD5M_C;
    u32x d = MD5M_D;

    MD5_STEP (MD5_Fo, a, b, c, d, w0[0], MD5C00, MD5S00);
    MD5_STEP (MD5_Fo, d, a, b, c, w0[1], MD5C01, MD5S01);
    MD5_STEP (MD5_Fo, c, d, a, b, w0[2], MD5C02, MD5S02);
    MD5_STEP (MD5_Fo, b, c, d, a, w0[3], MD5C03, MD5S03);
    MD5_STEP (MD5_Fo, a, b, c, d, w1[0], MD5C04, MD5S00);
    MD5_STEP (MD5_Fo, d, a, b, c, w1[1], MD5C05, MD5S01);
    MD5_STEP (MD5_Fo, c, d, a, b, w1[2], MD5C06, MD5S02);
    MD5_STEP (MD5_Fo, b, c, d, a, w1[3], MD5C07, MD5S03);
    MD5_STEP (MD5_Fo, a, b, c, d, w2[0], MD5C08, MD5S00);
    MD5_STEP (MD5_Fo, d, a, b, c, w2[1], MD5C09, MD5S01);
    MD5_STEP (MD5_Fo, c, d, a, b, w2[2], MD5C0a, MD5S02);
    MD5_STEP (MD5_Fo, b, c, d, a, w2[3], MD5C0b, MD5S03);
    MD5_STEP (MD5_Fo, a, b, c, d, w3[0], MD5C0c, MD5S00);
    MD5_STEP (MD5_Fo, d, a, b, c, w3[1], MD5C0d, MD5S01);
    MD5_STEP (MD5_Fo, c, d, a, b, w3[2], MD5C0e, MD5S02);
    MD5_STEP (MD5_Fo, b, c, d, a, w3[3], MD5C0f, MD5S03);

    MD5_STEP (MD5_Go, a, b, c, d, w0[1], MD5C10, MD5S10);
    MD5_STEP (MD5_Go, d, a, b, c, w1[2], MD5C11, MD5S11);
    MD5_STEP (MD5_Go, c, d, a, b, w2[3], MD5C12, MD5S12);
    MD5_STEP (MD5_Go, b, c, d, a, w0[0], MD5C13, MD5S13);
    MD5_STEP (MD5_Go, a, b, c, d, w1[1], MD5C14, MD5S10);
    MD5_STEP (MD5_Go, d, a, b, c, w2[2], MD5C15, MD5S11);
    MD5_STEP (MD5_Go, c, d, a, b, w3[3], MD5C16, MD5S12);
    MD5_STEP (MD5_Go, b, c, d, a, w1[0], MD5C17, MD5S13);
    MD5_STEP (MD5_Go, a, b, c, d, w2[1], MD5C18, MD5S10);
    MD5_STEP (MD5_Go, d, a, b, c, w3[2], MD5C19, MD5S11);
    MD5_STEP (MD5_Go, c, d, a, b, w0[3], MD5C1a, MD5S12);
    MD5_STEP (MD5_Go, b, c, d, a, w2[0], MD5C1b, MD5S13);
    MD5_STEP (MD5_Go, a, b, c, d, w3[1], MD5C1c, MD5S10);
    MD5_STEP (MD5_Go, d, a, b, c, w0[2], MD5C1d, MD5S11);
    MD5_STEP (MD5_Go, c, d, a, b, w1[3], MD5C1e, MD5S12);
    MD5_STEP (MD5_Go, b, c, d, a, w3[0], MD5C1f, MD5S13);

    MD5_STEP (MD5_H , a, b, c, d, w1[1], MD5C20, MD5S20);
    MD5_STEP (MD5_H , d, a, b, c, w2[0], MD5C21, MD5S21);
    MD5_STEP (MD5_H , c, d, a, b, w2[3], MD5C22, MD5S22);
    MD5_STEP (MD5_H , b, c, d, a, w3[2], MD5C23, MD5S23);
    MD5_STEP (MD5_H , a, b, c, d, w0[1], MD5C24, MD5S20);
    MD5_STEP (MD5_H , d, a, b, c, w1[0], MD5C25, MD5S21);
    MD5_STEP (MD5_H , c, d, a, b, w1[3], MD5C26, MD5S22);
    MD5_STEP (MD5_H , b, c, d, a, w2[2], MD5C27, MD5S23);
    MD5_STEP (MD5_H , a, b, c, d, w3[1], MD5C28, MD5S20);
    MD5_STEP (MD5_H , d, a, b, c, w0[0], MD5C29, MD5S21);
    MD5_STEP (MD5_H , c, d, a, b, w0[3], MD5C2a, MD5S22);
    MD5_STEP (MD5_H , b, c, d, a, w1[2], MD5C2b, MD5S23);
    MD5_STEP (MD5_H , a, b, c, d, w2[1], MD5C2c, MD5S20);
    MD5_STEP (MD5_H , d, a, b, c, w3[0], MD5C2d, MD5S21);
    MD5_STEP (MD5_H , c, d, a, b, w3[3], MD5C2e, MD5S22);
    MD5_STEP (MD5_H , b, c, d, a, w0[2], MD5C2f, MD5S23);

    MD5_STEP (MD5_I , a, b, c, d, w0[0], MD5C30, MD5S30);
    MD5_STEP (MD5_I , d, a, b, c, w1[3], MD5C31, MD5S31);
    MD5_STEP (MD5_I , c, d, a, b, w3[2], MD5C32, MD5S32);
    MD5_STEP (MD5_I , b, c, d, a, w1[1], MD5C33, MD5S33);
    MD5_STEP (MD5_I , a, b, c, d, w3[0], MD5C34, MD5S30);
    MD5_STEP (MD5_I , d, a, b, c, w0[3], MD5C35, MD5S31);
    MD5_STEP (MD5_I , c, d, a, b, w2[2], MD5C36, MD5S32);
    MD5_STEP (MD5_I , b, c, d, a, w0[1], MD5C37, MD5S33);
    MD5_STEP (MD5_I , a, b, c, d, w2[0], MD5C38, MD5S30);
    MD5_STEP (MD5_I , d, a, b, c, w3[3], MD5C39, MD5S31);
    MD5_STEP (MD5_I , c, d, a, b, w1[2], MD5C3a, MD5S32);
    MD5_STEP (MD5_I , b, c, d, a, w3[1], MD5C3b, MD5S33);
    MD5_STEP (MD5_I , a, b, c, d, w1[0], MD5C3c, MD5S30);
    MD5_STEP (MD5_I , d, a, b, c, w2[3], MD5C3d, MD5S31);
    MD5_STEP (MD5_I , c, d, a, b, w0[2], MD5C3e, MD5S32);
    MD5_STEP (MD5_I , b, c, d, a, w2[1], MD5C3f, MD5S33);

    a += MD5M_A;
    b += MD5M_B;
    c += MD5M_C;
    d += MD5M_D;

    /*
     * sha1
     */

    u32x w0_t = uint_to_hex_lower8_le ((a >>  8) & 255) <<  0
               | uint_to_hex_lower8_le ((a >>  0) & 255) << 16;
    u32x w1_t = uint_to_hex_lower8_le ((a >> 24) & 255) <<  0
               | uint_to_hex_lower8_le ((a >> 16) & 255) << 16;
    u32x w2_t = uint_to_hex_lower8_le ((b >>  8) & 255) <<  0
               | uint_to_hex_lower8_le ((b >>  0) & 255) << 16;
    u32x w3_t = uint_to_hex_lower8_le ((b >> 24) & 255) <<  0
               | uint_to_hex_lower8_le ((b >> 16) & 255) << 16;
    u32x w4_t = uint_to_hex_lower8_le ((c >>  8) & 255) <<  0
               | uint_to_hex_lower8_le ((c >>  0) & 255) << 16;
    u32x w5_t = uint_to_hex_lower8_le ((c >> 24) & 255) <<  0
               | uint_to_hex_lower8_le ((c >> 16) & 255) << 16;
    u32x w6_t = uint_to_hex_lower8_le ((d >>  8) & 255) <<  0
               | uint_to_hex_lower8_le ((d >>  0) & 255) << 16;
    u32x w7_t = uint_to_hex_lower8_le ((d >> 24) & 255) <<  0
               | uint_to_hex_lower8_le ((d >> 16) & 255) << 16;

    u32x w8_t = 0x80000000;
    u32x w9_t = 0;
    u32x wa_t = 0;
    u32x wb_t = 0;
    u32x wc_t = 0;
    u32x wd_t = 0;
    u32x we_t = 0;
    u32x wf_t = 32 * 8;

    u32x e;

    a = SHA1M_A;
    b = SHA1M_B;
    c = SHA1M_C;
    d = SHA1M_D;
    e = SHA1M_E;

    #undef K
    #define K SHA1C00

    SHA1_STEP (SHA1_F0o, a, b, c, d, e, w0_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, w1_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, w2_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, w3_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, w4_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, w5_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, w6_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, w7_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, w8_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, w9_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, wa_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, wb_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, wc_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, wd_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, we_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F0o, e, a, b, c, d, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F0o, d, e, a, b, c, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F0o, c, d, e, a, b, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F0o, b, c, d, e, a, w3_t);

    #undef K
    #define K SHA1C01

    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w7_t);

    #undef K
    #define K SHA1C02

    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, wb_t);

    #undef K
    #define K SHA1C03

    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wf_t);

    const u32x r0 = d;
    const u32x r1 = e;
    const u32x r2 = c;
    const u32x r3 = b;

    #include VECT_COMPARE_M
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m04700_m08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m04700_m16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m04700_s04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  u32x pw_buf0[4];

  pw_buf0[0] = pws[gid].i[ 0];
  pw_buf0[1] = pws[gid].i[ 1];
  pw_buf0[2] = pws[gid].i[ 2];
  pw_buf0[3] = pws[gid].i[ 3];

  u32x pw_buf1[4];

  pw_buf1[0] = pws[gid].i[ 4];
  pw_buf1[1] = pws[gid].i[ 5];
  pw_buf1[2] = pws[gid].i[ 6];
  pw_buf1[3] = pws[gid].i[ 7];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * bin2asc table
   */

  l_bin2asc[lid] = c_bin2asc[(lid >> 0) & 15] << 0
                 | c_bin2asc[(lid >> 4) & 15] << 8;

  __syncthreads ();

  if (gid >= gid_max) return;

  /**
   * digest
   */

  const u32 search[4] =
  {
    digests_buf[digests_offset].digest_buf[DGST_R0],
    digests_buf[digests_offset].digest_buf[DGST_R1],
    digests_buf[digests_offset].digest_buf[DGST_R2],
    digests_buf[digests_offset].digest_buf[DGST_R3]
  };

  /**
   * reverse
   */

  const u32 e_rev = rotl32 (search[1], 2u);

  /**
   * loop
   */

  for (u32 il_pos = 0; il_pos < rules_cnt; il_pos++)
  {
    u32x w0[4];

    w0[0] = pw_buf0[0];
    w0[1] = pw_buf0[1];
    w0[2] = pw_buf0[2];
    w0[3] = pw_buf0[3];

    u32x w1[4];

    w1[0] = pw_buf1[0];
    w1[1] = pw_buf1[1];
    w1[2] = pw_buf1[2];
    w1[3] = pw_buf1[3];

    u32x w2[4];

    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = pw_len * 8;
    w3[3] = 0;

    const u32 out_len = apply_rules (c_rules[il_pos].cmds, w0, w1, pw_len);

    append_0x80_2 (w0, w1, out_len);

    /**
     * md5
     */

    u32x a = MD5M_A;
    u32x b = MD5M_B;
    u32x c = MD5M_C;
    u32x d = MD5M_D;

    MD5_STEP (MD5_Fo, a, b, c, d, w0[0], MD5C00, MD5S00);
    MD5_STEP (MD5_Fo, d, a, b, c, w0[1], MD5C01, MD5S01);
    MD5_STEP (MD5_Fo, c, d, a, b, w0[2], MD5C02, MD5S02);
    MD5_STEP (MD5_Fo, b, c, d, a, w0[3], MD5C03, MD5S03);
    MD5_STEP (MD5_Fo, a, b, c, d, w1[0], MD5C04, MD5S00);
    MD5_STEP (MD5_Fo, d, a, b, c, w1[1], MD5C05, MD5S01);
    MD5_STEP (MD5_Fo, c, d, a, b, w1[2], MD5C06, MD5S02);
    MD5_STEP (MD5_Fo, b, c, d, a, w1[3], MD5C07, MD5S03);
    MD5_STEP (MD5_Fo, a, b, c, d, w2[0], MD5C08, MD5S00);
    MD5_STEP (MD5_Fo, d, a, b, c, w2[1], MD5C09, MD5S01);
    MD5_STEP (MD5_Fo, c, d, a, b, w2[2], MD5C0a, MD5S02);
    MD5_STEP (MD5_Fo, b, c, d, a, w2[3], MD5C0b, MD5S03);
    MD5_STEP (MD5_Fo, a, b, c, d, w3[0], MD5C0c, MD5S00);
    MD5_STEP (MD5_Fo, d, a, b, c, w3[1], MD5C0d, MD5S01);
    MD5_STEP (MD5_Fo, c, d, a, b, w3[2], MD5C0e, MD5S02);
    MD5_STEP (MD5_Fo, b, c, d, a, w3[3], MD5C0f, MD5S03);

    MD5_STEP (MD5_Go, a, b, c, d, w0[1], MD5C10, MD5S10);
    MD5_STEP (MD5_Go, d, a, b, c, w1[2], MD5C11, MD5S11);
    MD5_STEP (MD5_Go, c, d, a, b, w2[3], MD5C12, MD5S12);
    MD5_STEP (MD5_Go, b, c, d, a, w0[0], MD5C13, MD5S13);
    MD5_STEP (MD5_Go, a, b, c, d, w1[1], MD5C14, MD5S10);
    MD5_STEP (MD5_Go, d, a, b, c, w2[2], MD5C15, MD5S11);
    MD5_STEP (MD5_Go, c, d, a, b, w3[3], MD5C16, MD5S12);
    MD5_STEP (MD5_Go, b, c, d, a, w1[0], MD5C17, MD5S13);
    MD5_STEP (MD5_Go, a, b, c, d, w2[1], MD5C18, MD5S10);
    MD5_STEP (MD5_Go, d, a, b, c, w3[2], MD5C19, MD5S11);
    MD5_STEP (MD5_Go, c, d, a, b, w0[3], MD5C1a, MD5S12);
    MD5_STEP (MD5_Go, b, c, d, a, w2[0], MD5C1b, MD5S13);
    MD5_STEP (MD5_Go, a, b, c, d, w3[1], MD5C1c, MD5S10);
    MD5_STEP (MD5_Go, d, a, b, c, w0[2], MD5C1d, MD5S11);
    MD5_STEP (MD5_Go, c, d, a, b, w1[3], MD5C1e, MD5S12);
    MD5_STEP (MD5_Go, b, c, d, a, w3[0], MD5C1f, MD5S13);

    MD5_STEP (MD5_H , a, b, c, d, w1[1], MD5C20, MD5S20);
    MD5_STEP (MD5_H , d, a, b, c, w2[0], MD5C21, MD5S21);
    MD5_STEP (MD5_H , c, d, a, b, w2[3], MD5C22, MD5S22);
    MD5_STEP (MD5_H , b, c, d, a, w3[2], MD5C23, MD5S23);
    MD5_STEP (MD5_H , a, b, c, d, w0[1], MD5C24, MD5S20);
    MD5_STEP (MD5_H , d, a, b, c, w1[0], MD5C25, MD5S21);
    MD5_STEP (MD5_H , c, d, a, b, w1[3], MD5C26, MD5S22);
    MD5_STEP (MD5_H , b, c, d, a, w2[2], MD5C27, MD5S23);
    MD5_STEP (MD5_H , a, b, c, d, w3[1], MD5C28, MD5S20);
    MD5_STEP (MD5_H , d, a, b, c, w0[0], MD5C29, MD5S21);
    MD5_STEP (MD5_H , c, d, a, b, w0[3], MD5C2a, MD5S22);
    MD5_STEP (MD5_H , b, c, d, a, w1[2], MD5C2b, MD5S23);
    MD5_STEP (MD5_H , a, b, c, d, w2[1], MD5C2c, MD5S20);
    MD5_STEP (MD5_H , d, a, b, c, w3[0], MD5C2d, MD5S21);
    MD5_STEP (MD5_H , c, d, a, b, w3[3], MD5C2e, MD5S22);
    MD5_STEP (MD5_H , b, c, d, a, w0[2], MD5C2f, MD5S23);

    MD5_STEP (MD5_I , a, b, c, d, w0[0], MD5C30, MD5S30);
    MD5_STEP (MD5_I , d, a, b, c, w1[3], MD5C31, MD5S31);
    MD5_STEP (MD5_I , c, d, a, b, w3[2], MD5C32, MD5S32);
    MD5_STEP (MD5_I , b, c, d, a, w1[1], MD5C33, MD5S33);
    MD5_STEP (MD5_I , a, b, c, d, w3[0], MD5C34, MD5S30);
    MD5_STEP (MD5_I , d, a, b, c, w0[3], MD5C35, MD5S31);
    MD5_STEP (MD5_I , c, d, a, b, w2[2], MD5C36, MD5S32);
    MD5_STEP (MD5_I , b, c, d, a, w0[1], MD5C37, MD5S33);
    MD5_STEP (MD5_I , a, b, c, d, w2[0], MD5C38, MD5S30);
    MD5_STEP (MD5_I , d, a, b, c, w3[3], MD5C39, MD5S31);
    MD5_STEP (MD5_I , c, d, a, b, w1[2], MD5C3a, MD5S32);
    MD5_STEP (MD5_I , b, c, d, a, w3[1], MD5C3b, MD5S33);
    MD5_STEP (MD5_I , a, b, c, d, w1[0], MD5C3c, MD5S30);
    MD5_STEP (MD5_I , d, a, b, c, w2[3], MD5C3d, MD5S31);
    MD5_STEP (MD5_I , c, d, a, b, w0[2], MD5C3e, MD5S32);
    MD5_STEP (MD5_I , b, c, d, a, w2[1], MD5C3f, MD5S33);

    a += MD5M_A;
    b += MD5M_B;
    c += MD5M_C;
    d += MD5M_D;

    /*
     * sha1
     */

    u32x w0_t = uint_to_hex_lower8_le ((a >>  8) & 255) <<  0
               | uint_to_hex_lower8_le ((a >>  0) & 255) << 16;
    u32x w1_t = uint_to_hex_lower8_le ((a >> 24) & 255) <<  0
               | uint_to_hex_lower8_le ((a >> 16) & 255) << 16;
    u32x w2_t = uint_to_hex_lower8_le ((b >>  8) & 255) <<  0
               | uint_to_hex_lower8_le ((b >>  0) & 255) << 16;
    u32x w3_t = uint_to_hex_lower8_le ((b >> 24) & 255) <<  0
               | uint_to_hex_lower8_le ((b >> 16) & 255) << 16;
    u32x w4_t = uint_to_hex_lower8_le ((c >>  8) & 255) <<  0
               | uint_to_hex_lower8_le ((c >>  0) & 255) << 16;
    u32x w5_t = uint_to_hex_lower8_le ((c >> 24) & 255) <<  0
               | uint_to_hex_lower8_le ((c >> 16) & 255) << 16;
    u32x w6_t = uint_to_hex_lower8_le ((d >>  8) & 255) <<  0
               | uint_to_hex_lower8_le ((d >>  0) & 255) << 16;
    u32x w7_t = uint_to_hex_lower8_le ((d >> 24) & 255) <<  0
               | uint_to_hex_lower8_le ((d >> 16) & 255) << 16;

    u32x w8_t = 0x80000000;
    u32x w9_t = 0;
    u32x wa_t = 0;
    u32x wb_t = 0;
    u32x wc_t = 0;
    u32x wd_t = 0;
    u32x we_t = 0;
    u32x wf_t = 32 * 8;

    u32x e;

    a = SHA1M_A;
    b = SHA1M_B;
    c = SHA1M_C;
    d = SHA1M_D;
    e = SHA1M_E;

    #undef K
    #define K SHA1C00

    SHA1_STEP (SHA1_F0o, a, b, c, d, e, w0_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, w1_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, w2_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, w3_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, w4_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, w5_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, w6_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, w7_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, w8_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, w9_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, wa_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, wb_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, wc_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, wd_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, we_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F0o, e, a, b, c, d, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F0o, d, e, a, b, c, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F0o, c, d, e, a, b, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F0o, b, c, d, e, a, w3_t);

    #undef K
    #define K SHA1C01

    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w7_t);

    #undef K
    #define K SHA1C02

    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, wb_t);

    #undef K
    #define K SHA1C03

    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, wb_t);

    if (e != e_rev) continue;

    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wf_t);

    const u32x r0 = d;
    const u32x r1 = e;
    const u32x r2 = c;
    const u32x r3 = b;

    #include VECT_COMPARE_S
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m04700_s08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m04700_s16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}
