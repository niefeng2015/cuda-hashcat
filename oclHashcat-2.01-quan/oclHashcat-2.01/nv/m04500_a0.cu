#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _SHA1_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 3
#define DGST_R1 4
#define DGST_R2 2
#define DGST_R3 1

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"
#include "include/rp_gpu.h"
#include "rp_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_S "check_single_vect1_comp4.c"
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_S "check_single_vect2_comp4.c"
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

#ifdef  VECT_SIZE4
#define VECT_COMPARE_S "check_single_vect4_comp4.c"
#define VECT_COMPARE_M "check_multi_vect4_comp4.c"
#endif

#ifdef VECT_SIZE1
#define uint_to_hex_lower8_le(i) l_bin2asc[(i)]
#endif

#ifdef VECT_SIZE2
#define uint_to_hex_lower8_le(i) u32x (l_bin2asc[(i).x], l_bin2asc[(i).y])
#endif

#ifdef VECT_SIZE4
#define uint_to_hex_lower8_le(i) u32x (l_bin2asc[(i).x], l_bin2asc[(i).y], l_bin2asc[(i).z], l_bin2asc[(i).w])
#endif

__device__ __constant__ char c_bin2asc[16] = { '0', '1', '2', '3', '4', '5', '6', '7', '8', '9', 'a', 'b', 'c', 'd', 'e', 'f' };

__device__ __shared__ short l_bin2asc[256];

__device__ __constant__ gpu_rule_t c_rules[1024];

extern "C" __global__ void __launch_bounds__ (256, 1) m04500_m04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  u32x pw_buf0[4];

  pw_buf0[0] = pws[gid].i[ 0];
  pw_buf0[1] = pws[gid].i[ 1];
  pw_buf0[2] = pws[gid].i[ 2];
  pw_buf0[3] = pws[gid].i[ 3];

  u32x pw_buf1[4];

  pw_buf1[0] = pws[gid].i[ 4];
  pw_buf1[1] = pws[gid].i[ 5];
  pw_buf1[2] = pws[gid].i[ 6];
  pw_buf1[3] = pws[gid].i[ 7];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * bin2asc table
   */

  l_bin2asc[lid] = c_bin2asc[(lid >> 0) & 15] << 0
                 | c_bin2asc[(lid >> 4) & 15] << 8;

  __syncthreads ();

  if (gid >= gid_max) return;

  /**
   * loop
   */

  for (u32 il_pos = 0; il_pos < rules_cnt; il_pos++)
  {
    u32x w0[4];

    w0[0] = pw_buf0[0];
    w0[1] = pw_buf0[1];
    w0[2] = pw_buf0[2];
    w0[3] = pw_buf0[3];

    u32x w1[4];

    w1[0] = pw_buf1[0];
    w1[1] = pw_buf1[1];
    w1[2] = pw_buf1[2];
    w1[3] = pw_buf1[3];

    u32x w2[4];

    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = 0;

    const u32 out_len = apply_rules (c_rules[il_pos].cmds, w0, w1, pw_len);

    append_0x80_2 (w0, w1, out_len);

    /**
     * sha1
     */

    u32x w0_t = swap_workaround (w0[0]);
    u32x w1_t = swap_workaround (w0[1]);
    u32x w2_t = swap_workaround (w0[2]);
    u32x w3_t = swap_workaround (w0[3]);
    u32x w4_t = swap_workaround (w1[0]);
    u32x w5_t = swap_workaround (w1[1]);
    u32x w6_t = swap_workaround (w1[2]);
    u32x w7_t = swap_workaround (w1[3]);
    u32x w8_t = swap_workaround (w2[0]);
    u32x w9_t = swap_workaround (w2[1]);
    u32x wa_t = swap_workaround (w2[2]);
    u32x wb_t = swap_workaround (w2[3]);
    u32x wc_t = swap_workaround (w3[0]);
    u32x wd_t = swap_workaround (w3[1]);
    u32x we_t = 0;
    u32x wf_t = out_len * 8;

    u32x a = SHA1M_A;
    u32x b = SHA1M_B;
    u32x c = SHA1M_C;
    u32x d = SHA1M_D;
    u32x e = SHA1M_E;

    #undef K
    #define K SHA1C00

    SHA1_STEP (SHA1_F0o, a, b, c, d, e, w0_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, w1_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, w2_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, w3_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, w4_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, w5_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, w6_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, w7_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, w8_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, w9_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, wa_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, wb_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, wc_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, wd_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, we_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F0o, e, a, b, c, d, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F0o, d, e, a, b, c, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F0o, c, d, e, a, b, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F0o, b, c, d, e, a, w3_t);

    #undef K
    #define K SHA1C01

    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w7_t);

    #undef K
    #define K SHA1C02

    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, wb_t);

    #undef K
    #define K SHA1C03

    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wf_t);

    a += SHA1M_A;
    b += SHA1M_B;
    c += SHA1M_C;
    d += SHA1M_D;
    e += SHA1M_E;

    /**
     * 2nd SHA1
     */

    w0_t = uint_to_hex_lower8_le ((a >> 16) & 255) <<  0
         | uint_to_hex_lower8_le ((a >> 24) & 255) << 16;
    w1_t = uint_to_hex_lower8_le ((a >>  0) & 255) <<  0
         | uint_to_hex_lower8_le ((a >>  8) & 255) << 16;
    w2_t = uint_to_hex_lower8_le ((b >> 16) & 255) <<  0
         | uint_to_hex_lower8_le ((b >> 24) & 255) << 16;
    w3_t = uint_to_hex_lower8_le ((b >>  0) & 255) <<  0
         | uint_to_hex_lower8_le ((b >>  8) & 255) << 16;
    w4_t = uint_to_hex_lower8_le ((c >> 16) & 255) <<  0
         | uint_to_hex_lower8_le ((c >> 24) & 255) << 16;
    w5_t = uint_to_hex_lower8_le ((c >>  0) & 255) <<  0
         | uint_to_hex_lower8_le ((c >>  8) & 255) << 16;
    w6_t = uint_to_hex_lower8_le ((d >> 16) & 255) <<  0
         | uint_to_hex_lower8_le ((d >> 24) & 255) << 16;
    w7_t = uint_to_hex_lower8_le ((d >>  0) & 255) <<  0
         | uint_to_hex_lower8_le ((d >>  8) & 255) << 16;
    w8_t = uint_to_hex_lower8_le ((e >> 16) & 255) <<  0
         | uint_to_hex_lower8_le ((e >> 24) & 255) << 16;
    w9_t = uint_to_hex_lower8_le ((e >>  0) & 255) <<  0
         | uint_to_hex_lower8_le ((e >>  8) & 255) << 16;

    wa_t = 0x80000000;
    wb_t = 0;
    wc_t = 0;
    wd_t = 0;
    we_t = 0;
    wf_t = 40 * 8;

    a = SHA1M_A;
    b = SHA1M_B;
    c = SHA1M_C;
    d = SHA1M_D;
    e = SHA1M_E;

    #undef K
    #define K SHA1C00

    SHA1_STEP (SHA1_F0o, a, b, c, d, e, w0_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, w1_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, w2_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, w3_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, w4_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, w5_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, w6_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, w7_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, w8_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, w9_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, wa_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, wb_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, wc_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, wd_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, we_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F0o, e, a, b, c, d, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F0o, d, e, a, b, c, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F0o, c, d, e, a, b, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F0o, b, c, d, e, a, w3_t);

    #undef K
    #define K SHA1C01

    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w7_t);

    #undef K
    #define K SHA1C02

    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, wb_t);

    #undef K
    #define K SHA1C03

    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, wb_t);

    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wf_t);

    const u32x r0 = d;
    const u32x r1 = e;
    const u32x r2 = c;
    const u32x r3 = b;

    #include VECT_COMPARE_M
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m04500_m08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m04500_m16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m04500_s04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  u32x pw_buf0[4];

  pw_buf0[0] = pws[gid].i[ 0];
  pw_buf0[1] = pws[gid].i[ 1];
  pw_buf0[2] = pws[gid].i[ 2];
  pw_buf0[3] = pws[gid].i[ 3];

  u32x pw_buf1[4];

  pw_buf1[0] = pws[gid].i[ 4];
  pw_buf1[1] = pws[gid].i[ 5];
  pw_buf1[2] = pws[gid].i[ 6];
  pw_buf1[3] = pws[gid].i[ 7];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * bin2asc table
   */

  l_bin2asc[lid] = c_bin2asc[(lid >> 0) & 15] << 0
                 | c_bin2asc[(lid >> 4) & 15] << 8;

  __syncthreads ();

  if (gid >= gid_max) return;

  /**
   * digest
   */

  const u32 search[4] =
  {
    digests_buf[digests_offset].digest_buf[DGST_R0],
    digests_buf[digests_offset].digest_buf[DGST_R1],
    digests_buf[digests_offset].digest_buf[DGST_R2],
    digests_buf[digests_offset].digest_buf[DGST_R3]
  };

  /**
   * reverse
   */

  const u32 e_rev = rotl32 (search[1], 2u);

  /**
   * loop
   */

  for (u32 il_pos = 0; il_pos < rules_cnt; il_pos++)
  {
    u32x w0[4];

    w0[0] = pw_buf0[0];
    w0[1] = pw_buf0[1];
    w0[2] = pw_buf0[2];
    w0[3] = pw_buf0[3];

    u32x w1[4];

    w1[0] = pw_buf1[0];
    w1[1] = pw_buf1[1];
    w1[2] = pw_buf1[2];
    w1[3] = pw_buf1[3];

    u32x w2[4];

    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = 0;

    const u32 out_len = apply_rules (c_rules[il_pos].cmds, w0, w1, pw_len);

    append_0x80_2 (w0, w1, out_len);

    /**
     * sha1
     */

    u32x w0_t = swap_workaround (w0[0]);
    u32x w1_t = swap_workaround (w0[1]);
    u32x w2_t = swap_workaround (w0[2]);
    u32x w3_t = swap_workaround (w0[3]);
    u32x w4_t = swap_workaround (w1[0]);
    u32x w5_t = swap_workaround (w1[1]);
    u32x w6_t = swap_workaround (w1[2]);
    u32x w7_t = swap_workaround (w1[3]);
    u32x w8_t = swap_workaround (w2[0]);
    u32x w9_t = swap_workaround (w2[1]);
    u32x wa_t = swap_workaround (w2[2]);
    u32x wb_t = swap_workaround (w2[3]);
    u32x wc_t = swap_workaround (w3[0]);
    u32x wd_t = swap_workaround (w3[1]);
    u32x we_t = 0;
    u32x wf_t = out_len * 8;

    u32x a = SHA1M_A;
    u32x b = SHA1M_B;
    u32x c = SHA1M_C;
    u32x d = SHA1M_D;
    u32x e = SHA1M_E;

    #undef K
    #define K SHA1C00

    SHA1_STEP (SHA1_F0o, a, b, c, d, e, w0_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, w1_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, w2_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, w3_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, w4_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, w5_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, w6_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, w7_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, w8_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, w9_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, wa_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, wb_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, wc_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, wd_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, we_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F0o, e, a, b, c, d, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F0o, d, e, a, b, c, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F0o, c, d, e, a, b, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F0o, b, c, d, e, a, w3_t);

    #undef K
    #define K SHA1C01

    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w7_t);

    #undef K
    #define K SHA1C02

    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, wb_t);

    #undef K
    #define K SHA1C03

    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wf_t);

    a += SHA1M_A;
    b += SHA1M_B;
    c += SHA1M_C;
    d += SHA1M_D;
    e += SHA1M_E;

    /**
     * 2nd SHA1
     */

    w0_t = uint_to_hex_lower8_le ((a >> 16) & 255) <<  0
         | uint_to_hex_lower8_le ((a >> 24) & 255) << 16;
    w1_t = uint_to_hex_lower8_le ((a >>  0) & 255) <<  0
         | uint_to_hex_lower8_le ((a >>  8) & 255) << 16;
    w2_t = uint_to_hex_lower8_le ((b >> 16) & 255) <<  0
         | uint_to_hex_lower8_le ((b >> 24) & 255) << 16;
    w3_t = uint_to_hex_lower8_le ((b >>  0) & 255) <<  0
         | uint_to_hex_lower8_le ((b >>  8) & 255) << 16;
    w4_t = uint_to_hex_lower8_le ((c >> 16) & 255) <<  0
         | uint_to_hex_lower8_le ((c >> 24) & 255) << 16;
    w5_t = uint_to_hex_lower8_le ((c >>  0) & 255) <<  0
         | uint_to_hex_lower8_le ((c >>  8) & 255) << 16;
    w6_t = uint_to_hex_lower8_le ((d >> 16) & 255) <<  0
         | uint_to_hex_lower8_le ((d >> 24) & 255) << 16;
    w7_t = uint_to_hex_lower8_le ((d >>  0) & 255) <<  0
         | uint_to_hex_lower8_le ((d >>  8) & 255) << 16;
    w8_t = uint_to_hex_lower8_le ((e >> 16) & 255) <<  0
         | uint_to_hex_lower8_le ((e >> 24) & 255) << 16;
    w9_t = uint_to_hex_lower8_le ((e >>  0) & 255) <<  0
         | uint_to_hex_lower8_le ((e >>  8) & 255) << 16;

    wa_t = 0x80000000;
    wb_t = 0;
    wc_t = 0;
    wd_t = 0;
    we_t = 0;
    wf_t = 40 * 8;

    a = SHA1M_A;
    b = SHA1M_B;
    c = SHA1M_C;
    d = SHA1M_D;
    e = SHA1M_E;

    #undef K
    #define K SHA1C00

    SHA1_STEP (SHA1_F0o, a, b, c, d, e, w0_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, w1_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, w2_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, w3_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, w4_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, w5_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, w6_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, w7_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, w8_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, w9_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, wa_t);
    SHA1_STEP (SHA1_F0o, e, a, b, c, d, wb_t);
    SHA1_STEP (SHA1_F0o, d, e, a, b, c, wc_t);
    SHA1_STEP (SHA1_F0o, c, d, e, a, b, wd_t);
    SHA1_STEP (SHA1_F0o, b, c, d, e, a, we_t);
    SHA1_STEP (SHA1_F0o, a, b, c, d, e, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F0o, e, a, b, c, d, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F0o, d, e, a, b, c, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F0o, c, d, e, a, b, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F0o, b, c, d, e, a, w3_t);

    #undef K
    #define K SHA1C01

    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w7_t);

    #undef K
    #define K SHA1C02

    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, wb_t);
    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F2o, a, b, c, d, e, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F2o, e, a, b, c, d, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F2o, d, e, a, b, c, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F2o, c, d, e, a, b, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F2o, b, c, d, e, a, wb_t);

    #undef K
    #define K SHA1C03

    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, wf_t);
    w0_t = rotl32 ((wd_t ^ w8_t ^ w2_t ^ w0_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w0_t);
    w1_t = rotl32 ((we_t ^ w9_t ^ w3_t ^ w1_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w1_t);
    w2_t = rotl32 ((wf_t ^ wa_t ^ w4_t ^ w2_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w2_t);
    w3_t = rotl32 ((w0_t ^ wb_t ^ w5_t ^ w3_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w3_t);
    w4_t = rotl32 ((w1_t ^ wc_t ^ w6_t ^ w4_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w4_t);
    w5_t = rotl32 ((w2_t ^ wd_t ^ w7_t ^ w5_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, w5_t);
    w6_t = rotl32 ((w3_t ^ we_t ^ w8_t ^ w6_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, w6_t);
    w7_t = rotl32 ((w4_t ^ wf_t ^ w9_t ^ w7_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, w7_t);
    w8_t = rotl32 ((w5_t ^ w0_t ^ wa_t ^ w8_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, w8_t);
    w9_t = rotl32 ((w6_t ^ w1_t ^ wb_t ^ w9_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, w9_t);
    wa_t = rotl32 ((w7_t ^ w2_t ^ wc_t ^ wa_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wa_t);
    wb_t = rotl32 ((w8_t ^ w3_t ^ wd_t ^ wb_t), 1u); SHA1_STEP (SHA1_F1, a, b, c, d, e, wb_t);

    if (e != e_rev) continue;

    wc_t = rotl32 ((w9_t ^ w4_t ^ we_t ^ wc_t), 1u); SHA1_STEP (SHA1_F1, e, a, b, c, d, wc_t);
    wd_t = rotl32 ((wa_t ^ w5_t ^ wf_t ^ wd_t), 1u); SHA1_STEP (SHA1_F1, d, e, a, b, c, wd_t);
    we_t = rotl32 ((wb_t ^ w6_t ^ w0_t ^ we_t), 1u); SHA1_STEP (SHA1_F1, c, d, e, a, b, we_t);
    wf_t = rotl32 ((wc_t ^ w7_t ^ w1_t ^ wf_t), 1u); SHA1_STEP (SHA1_F1, b, c, d, e, a, wf_t);

    const u32x r0 = d;
    const u32x r1 = e;
    const u32x r2 = c;
    const u32x r3 = b;

    #include VECT_COMPARE_S
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m04500_s08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m04500_s16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}
