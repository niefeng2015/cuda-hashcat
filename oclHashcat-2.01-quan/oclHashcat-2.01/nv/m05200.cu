#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _SHA256_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 0
#define DGST_R1 1
#define DGST_R2 2
#define DGST_R3 3

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

#ifdef  VECT_SIZE4
#define VECT_COMPARE_M "check_multi_vect4_comp4.c"
#endif

__device__ static void sha256_64 (const u32x w0[4], const u32x w1[4], const u32x w2[4], const u32x w3[4], u32x digest[8])
{
  u32x w0_t = w0[0];
  u32x w1_t = w0[1];
  u32x w2_t = w0[2];
  u32x w3_t = w0[3];
  u32x w4_t = w1[0];
  u32x w5_t = w1[1];
  u32x w6_t = w1[2];
  u32x w7_t = w1[3];
  u32x w8_t = w2[0];
  u32x w9_t = w2[1];
  u32x wa_t = w2[2];
  u32x wb_t = w2[3];
  u32x wc_t = w3[0];
  u32x wd_t = w3[1];
  u32x we_t = w3[2];
  u32x wf_t = w3[3];

  u32x a = digest[0];
  u32x b = digest[1];
  u32x c = digest[2];
  u32x d = digest[3];
  u32x e = digest[4];
  u32x f = digest[5];
  u32x g = digest[6];
  u32x h = digest[7];

  SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C00);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C01);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C02);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C03);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C04);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C05);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C06);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C07);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C08);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C09);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C0a);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C0b);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C0c);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C0d);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C0e);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C0f);
  w0_t = SHA256_S1(we_t) + w9_t + SHA256_S0(w1_t) + w0_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C10);
  w1_t = SHA256_S1(wf_t) + wa_t + SHA256_S0(w2_t) + w1_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C11);
  w2_t = SHA256_S1(w0_t) + wb_t + SHA256_S0(w3_t) + w2_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C12);
  w3_t = SHA256_S1(w1_t) + wc_t + SHA256_S0(w4_t) + w3_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C13);
  w4_t = SHA256_S1(w2_t) + wd_t + SHA256_S0(w5_t) + w4_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C14);
  w5_t = SHA256_S1(w3_t) + we_t + SHA256_S0(w6_t) + w5_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C15);
  w6_t = SHA256_S1(w4_t) + wf_t + SHA256_S0(w7_t) + w6_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C16);
  w7_t = SHA256_S1(w5_t) + w0_t + SHA256_S0(w8_t) + w7_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C17);
  w8_t = SHA256_S1(w6_t) + w1_t + SHA256_S0(w9_t) + w8_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C18);
  w9_t = SHA256_S1(w7_t) + w2_t + SHA256_S0(wa_t) + w9_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C19);
  wa_t = SHA256_S1(w8_t) + w3_t + SHA256_S0(wb_t) + wa_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C1a);
  wb_t = SHA256_S1(w9_t) + w4_t + SHA256_S0(wc_t) + wb_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C1b);
  wc_t = SHA256_S1(wa_t) + w5_t + SHA256_S0(wd_t) + wc_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C1c);
  wd_t = SHA256_S1(wb_t) + w6_t + SHA256_S0(we_t) + wd_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C1d);
  we_t = SHA256_S1(wc_t) + w7_t + SHA256_S0(wf_t) + we_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C1e);
  wf_t = SHA256_S1(wd_t) + w8_t + SHA256_S0(w0_t) + wf_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C1f);
  w0_t = SHA256_S1(we_t) + w9_t + SHA256_S0(w1_t) + w0_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C20);
  w1_t = SHA256_S1(wf_t) + wa_t + SHA256_S0(w2_t) + w1_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C21);
  w2_t = SHA256_S1(w0_t) + wb_t + SHA256_S0(w3_t) + w2_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C22);
  w3_t = SHA256_S1(w1_t) + wc_t + SHA256_S0(w4_t) + w3_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C23);
  w4_t = SHA256_S1(w2_t) + wd_t + SHA256_S0(w5_t) + w4_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C24);
  w5_t = SHA256_S1(w3_t) + we_t + SHA256_S0(w6_t) + w5_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C25);
  w6_t = SHA256_S1(w4_t) + wf_t + SHA256_S0(w7_t) + w6_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C26);
  w7_t = SHA256_S1(w5_t) + w0_t + SHA256_S0(w8_t) + w7_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C27);
  w8_t = SHA256_S1(w6_t) + w1_t + SHA256_S0(w9_t) + w8_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C28);
  w9_t = SHA256_S1(w7_t) + w2_t + SHA256_S0(wa_t) + w9_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C29);
  wa_t = SHA256_S1(w8_t) + w3_t + SHA256_S0(wb_t) + wa_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C2a);
  wb_t = SHA256_S1(w9_t) + w4_t + SHA256_S0(wc_t) + wb_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C2b);
  wc_t = SHA256_S1(wa_t) + w5_t + SHA256_S0(wd_t) + wc_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C2c);
  wd_t = SHA256_S1(wb_t) + w6_t + SHA256_S0(we_t) + wd_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C2d);
  we_t = SHA256_S1(wc_t) + w7_t + SHA256_S0(wf_t) + we_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C2e);
  wf_t = SHA256_S1(wd_t) + w8_t + SHA256_S0(w0_t) + wf_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C2f);
  w0_t = SHA256_S1(we_t) + w9_t + SHA256_S0(w1_t) + w0_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C30);
  w1_t = SHA256_S1(wf_t) + wa_t + SHA256_S0(w2_t) + w1_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C31);
  w2_t = SHA256_S1(w0_t) + wb_t + SHA256_S0(w3_t) + w2_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C32);
  w3_t = SHA256_S1(w1_t) + wc_t + SHA256_S0(w4_t) + w3_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C33);
  w4_t = SHA256_S1(w2_t) + wd_t + SHA256_S0(w5_t) + w4_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C34);
  w5_t = SHA256_S1(w3_t) + we_t + SHA256_S0(w6_t) + w5_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C35);
  w6_t = SHA256_S1(w4_t) + wf_t + SHA256_S0(w7_t) + w6_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C36);
  w7_t = SHA256_S1(w5_t) + w0_t + SHA256_S0(w8_t) + w7_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C37);
  w8_t = SHA256_S1(w6_t) + w1_t + SHA256_S0(w9_t) + w8_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C38);
  w9_t = SHA256_S1(w7_t) + w2_t + SHA256_S0(wa_t) + w9_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C39);
  wa_t = SHA256_S1(w8_t) + w3_t + SHA256_S0(wb_t) + wa_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C3a);
  wb_t = SHA256_S1(w9_t) + w4_t + SHA256_S0(wc_t) + wb_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C3b);
  wc_t = SHA256_S1(wa_t) + w5_t + SHA256_S0(wd_t) + wc_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C3c);
  wd_t = SHA256_S1(wb_t) + w6_t + SHA256_S0(we_t) + wd_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C3d);
  we_t = SHA256_S1(wc_t) + w7_t + SHA256_S0(wf_t) + we_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C3e);
  wf_t = SHA256_S1(wd_t) + w8_t + SHA256_S0(w0_t) + wf_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C3f);

  digest[0] += a;
  digest[1] += b;
  digest[2] += c;
  digest[3] += d;
  digest[4] += e;
  digest[5] += f;
  digest[6] += g;
  digest[7] += h;
}

extern "C" __global__ void __launch_bounds__ (256, 1) m05200_init (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, pwsafe3_tmp_t *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x w0[4];

  w0[0] = pws[gid].i[ 0];
  w0[1] = pws[gid].i[ 1];
  w0[2] = pws[gid].i[ 2];
  w0[3] = pws[gid].i[ 3];

  u32x w1[4];

  w1[0] = pws[gid].i[ 4];
  w1[1] = pws[gid].i[ 5];
  w1[2] = 0;
  w1[3] = 0;

  u32x w2[4];

  w2[0] = 0;
  w2[1] = 0;
  w2[2] = 0;
  w2[3] = 0;

  u32x w3[4];

  w3[0] = 0;
  w3[1] = 0;
  w3[2] = 0;
  w3[3] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * salt
   */

  u32 salt_buf0[4];

  salt_buf0[0] = salt_bufs[salt_pos].salt_buf[0];
  salt_buf0[1] = salt_bufs[salt_pos].salt_buf[1];
  salt_buf0[2] = salt_bufs[salt_pos].salt_buf[2];
  salt_buf0[3] = salt_bufs[salt_pos].salt_buf[3];

  u32 salt_buf1[4];

  salt_buf1[0] = salt_bufs[salt_pos].salt_buf[4];
  salt_buf1[1] = salt_bufs[salt_pos].salt_buf[5];
  salt_buf1[2] = salt_bufs[salt_pos].salt_buf[6];
  salt_buf1[3] = salt_bufs[salt_pos].salt_buf[7];

  u32 salt_buf2[4];

  salt_buf2[0] = 0;
  salt_buf2[1] = 0;
  salt_buf2[2] = 0;
  salt_buf2[3] = 0;

  u32 salt_buf3[4];

  salt_buf3[0] = 0;
  salt_buf3[1] = 0;
  salt_buf3[2] = 0;
  salt_buf3[3] = 0;

  u32 salt_len = salt_bufs[salt_pos].salt_len;

  switch_buffer_by_offset (salt_buf0, salt_buf1, salt_buf2, salt_buf3, pw_len);

  w0[0] |= salt_buf0[0];
  w0[1] |= salt_buf0[1];
  w0[2] |= salt_buf0[2];
  w0[3] |= salt_buf0[3];

  w1[0] |= salt_buf1[0];
  w1[1] |= salt_buf1[1];
  w1[2] |= salt_buf1[2];
  w1[3] |= salt_buf1[3];

  w2[0] |= salt_buf2[0];
  w2[1] |= salt_buf2[1];
  w2[2] |= salt_buf2[2];
  w2[3] |= salt_buf2[3];

  w3[0] |= salt_buf3[0];
  w3[1] |= salt_buf3[1];
  w3[2] |= salt_buf3[2];
  w3[3] |= salt_buf3[3];

  const u32 block_len = pw_len + salt_len;

  append_0x80_4 (w0, w1, w2, w3, block_len);

  /**
   * init
   */

  w0[0] = swap_workaround (w0[0]);
  w0[1] = swap_workaround (w0[1]);
  w0[2] = swap_workaround (w0[2]);
  w0[3] = swap_workaround (w0[3]);

  w1[0] = swap_workaround (w1[0]);
  w1[1] = swap_workaround (w1[1]);
  w1[2] = swap_workaround (w1[2]);
  w1[3] = swap_workaround (w1[3]);

  w2[0] = swap_workaround (w2[0]);
  w2[1] = swap_workaround (w2[1]);
  w2[2] = swap_workaround (w2[2]);
  w2[3] = swap_workaround (w2[3]);

  w3[0] = swap_workaround (w3[0]);
  w3[1] = swap_workaround (w3[1]);
  w3[2] = 0;
  w3[3] = block_len * 8;

  /**
   * main
   */

  u32x digest[8];

  digest[0] = SHA256M_A;
  digest[1] = SHA256M_B;
  digest[2] = SHA256M_C;
  digest[3] = SHA256M_D;
  digest[4] = SHA256M_E;
  digest[5] = SHA256M_F;
  digest[6] = SHA256M_G;
  digest[7] = SHA256M_H;

  sha256_64 (w0, w1, w2, w3, digest);

  tmps[gid].digest_buf[0] = digest[0];
  tmps[gid].digest_buf[1] = digest[1];
  tmps[gid].digest_buf[2] = digest[2];
  tmps[gid].digest_buf[3] = digest[3];
  tmps[gid].digest_buf[4] = digest[4];
  tmps[gid].digest_buf[5] = digest[5];
  tmps[gid].digest_buf[6] = digest[6];
  tmps[gid].digest_buf[7] = digest[7];
}

extern "C" __global__ void __launch_bounds__ (256, 1) m05200_loop (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, pwsafe3_tmp_t *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x digest[8];

  digest[0] = tmps[gid].digest_buf[0];
  digest[1] = tmps[gid].digest_buf[1];
  digest[2] = tmps[gid].digest_buf[2];
  digest[3] = tmps[gid].digest_buf[3];
  digest[4] = tmps[gid].digest_buf[4];
  digest[5] = tmps[gid].digest_buf[5];
  digest[6] = tmps[gid].digest_buf[6];
  digest[7] = tmps[gid].digest_buf[7];

  for (u32 i = 0; i < loop_cnt; i++)
  {
    u32x w0[4];

    w0[0] = digest[0];
    w0[1] = digest[1];
    w0[2] = digest[2];
    w0[3] = digest[3];

    u32x w1[4];

    w1[0] = digest[4];
    w1[1] = digest[5];
    w1[2] = digest[6];
    w1[3] = digest[7];

    u32x w2[4];

    w2[0] = 0x80000000;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = 32 * 8;

    digest[0] = SHA256M_A;
    digest[1] = SHA256M_B;
    digest[2] = SHA256M_C;
    digest[3] = SHA256M_D;
    digest[4] = SHA256M_E;
    digest[5] = SHA256M_F;
    digest[6] = SHA256M_G;
    digest[7] = SHA256M_H;

    sha256_64 (w0, w1, w2, w3, digest);
  }

  tmps[gid].digest_buf[0] = digest[0];
  tmps[gid].digest_buf[1] = digest[1];
  tmps[gid].digest_buf[2] = digest[2];
  tmps[gid].digest_buf[3] = digest[3];
  tmps[gid].digest_buf[4] = digest[4];
  tmps[gid].digest_buf[5] = digest[5];
  tmps[gid].digest_buf[6] = digest[6];
  tmps[gid].digest_buf[7] = digest[7];
}

extern "C" __global__ void __launch_bounds__ (256, 1) m05200_comp (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, pwsafe3_tmp_t *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  const u32 lid = threadIdx.x;

  /**
   * digest
   */

  const u32x r0 = tmps[gid].digest_buf[DGST_R0];
  const u32x r1 = tmps[gid].digest_buf[DGST_R1];
  const u32x r2 = tmps[gid].digest_buf[DGST_R2];
  const u32x r3 = tmps[gid].digest_buf[DGST_R3];

  #define il_pos 0

  #include VECT_COMPARE_M
}
