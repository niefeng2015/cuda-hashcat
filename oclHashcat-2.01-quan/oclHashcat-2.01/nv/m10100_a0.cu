#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _SIPHASH_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 0
#define DGST_R1 1
#define DGST_R2 2
#define DGST_R3 3

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"
#include "include/rp_gpu.h"
#include "rp_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_S "check_single_vect1_comp4.c"
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_S "check_single_vect2_comp4.c"
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

#ifdef  VECT_SIZE4
#define VECT_COMPARE_S "check_single_vect4_comp4.c"
#define VECT_COMPARE_M "check_multi_vect4_comp4.c"
#endif

#define SIPROUND(v0,v1,v2,v3) \
  (v0) += (v1);               \
  (v1)  = rotl64 ((v1), 13);  \
  (v1) ^= (v0);               \
  (v0)  = rotl64 ((v0), 32);  \
  (v2) += (v3);               \
  (v3)  = rotl64 ((v3), 16);  \
  (v3) ^= (v2);               \
  (v0) += (v3);               \
  (v3)  = rotl64 ((v3), 21);  \
  (v3) ^= (v0);               \
  (v2) += (v1);               \
  (v1)  = rotl64 ((v1), 17);  \
  (v1) ^= (v2);               \
  (v2)  = rotl64 ((v2), 32);

__device__ __constant__ gpu_rule_t c_rules[1024];

extern "C" __global__ void __launch_bounds__ (256, 1) m10100_m04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x pw_buf0[4];

  pw_buf0[0] = pws[gid].i[ 0];
  pw_buf0[1] = pws[gid].i[ 1];
  pw_buf0[2] = pws[gid].i[ 2];
  pw_buf0[3] = pws[gid].i[ 3];

  u32x pw_buf1[4];

  pw_buf1[0] = pws[gid].i[ 4];
  pw_buf1[1] = pws[gid].i[ 5];
  pw_buf1[2] = pws[gid].i[ 6];
  pw_buf1[3] = pws[gid].i[ 7];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * base
   */

  u64 v0p = SIPHASHM_0;
  u64 v1p = SIPHASHM_1;
  u64 v2p = SIPHASHM_2;
  u64 v3p = SIPHASHM_3;

  v0p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[1], salt_bufs[salt_pos].salt_buf[0]);
  v1p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[3], salt_bufs[salt_pos].salt_buf[2]);
  v2p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[1], salt_bufs[salt_pos].salt_buf[0]);
  v3p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[3], salt_bufs[salt_pos].salt_buf[2]);

  /**
   * loop
   */

  for (u32 il_pos = 0; il_pos < rules_cnt; il_pos++)
  {
    u32x w[16];

    w[ 0] = pw_buf0[0];
    w[ 1] = pw_buf0[1];
    w[ 2] = pw_buf0[2];
    w[ 3] = pw_buf0[3];
    w[ 4] = pw_buf1[0];
    w[ 5] = pw_buf1[1];
    w[ 6] = pw_buf1[2];
    w[ 7] = pw_buf1[3];
    w[ 8] = 0;
    w[ 9] = 0;
    w[10] = 0;
    w[11] = 0;
    w[12] = 0;
    w[13] = 0;
    w[14] = 0;
    w[15] = 0;

    const u32x out_len = apply_rules (c_rules[il_pos].cmds, &w[0], &w[4], pw_len);

    u64 *w_ptr = (u64 *) w;

    w_ptr[out_len / 8] |= (u64) out_len << 56;

    u64x v0 = v0p;
    u64x v1 = v1p;
    u64x v2 = v2p;
    u64x v3 = v3p;

    int i;
    int j;

    for (i = 0, j = 0; i <= pw_len; i += 8, j += 2)
    {
      u64x m = hl32_to_64 (w[j + 1], w[j + 0]);

      v3 ^= m;

      SIPROUND (v0, v1, v2, v3);
      SIPROUND (v0, v1, v2, v3);

      v0 ^= m;
    }

    v2 ^= 0xff;

    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);

    const u64x v = v0 ^ v1 ^ v2 ^ v3;

    const u32x a = l32_from_64 (v);
    const u32x b = h32_from_64 (v);

    const u32x r0 = a;
    const u32x r1 = b;
    const u32x r2 = 0;
    const u32x r3 = 0;

    #include VECT_COMPARE_M
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m10100_m08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m10100_m16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m10100_s04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x pw_buf0[4];

  pw_buf0[0] = pws[gid].i[ 0];
  pw_buf0[1] = pws[gid].i[ 1];
  pw_buf0[2] = pws[gid].i[ 2];
  pw_buf0[3] = pws[gid].i[ 3];

  u32x pw_buf1[4];

  pw_buf1[0] = pws[gid].i[ 4];
  pw_buf1[1] = pws[gid].i[ 5];
  pw_buf1[2] = pws[gid].i[ 6];
  pw_buf1[3] = pws[gid].i[ 7];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * base
   */

  u64 v0p = SIPHASHM_0;
  u64 v1p = SIPHASHM_1;
  u64 v2p = SIPHASHM_2;
  u64 v3p = SIPHASHM_3;

  v0p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[1], salt_bufs[salt_pos].salt_buf[0]);
  v1p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[3], salt_bufs[salt_pos].salt_buf[2]);
  v2p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[1], salt_bufs[salt_pos].salt_buf[0]);
  v3p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[3], salt_bufs[salt_pos].salt_buf[2]);

  /**
   * digest
   */

  const u32 search[4] =
  {
    digests_buf[digests_offset].digest_buf[DGST_R0],
    digests_buf[digests_offset].digest_buf[DGST_R1],
    digests_buf[digests_offset].digest_buf[DGST_R2],
    digests_buf[digests_offset].digest_buf[DGST_R3]
  };

  /**
   * loop
   */

  for (u32 il_pos = 0; il_pos < rules_cnt; il_pos++)
  {
    u32x w[16];

    w[ 0] = pw_buf0[0];
    w[ 1] = pw_buf0[1];
    w[ 2] = pw_buf0[2];
    w[ 3] = pw_buf0[3];
    w[ 4] = pw_buf1[0];
    w[ 5] = pw_buf1[1];
    w[ 6] = pw_buf1[2];
    w[ 7] = pw_buf1[3];
    w[ 8] = 0;
    w[ 9] = 0;
    w[10] = 0;
    w[11] = 0;
    w[12] = 0;
    w[13] = 0;
    w[14] = 0;
    w[15] = 0;

    const u32x out_len = apply_rules (c_rules[il_pos].cmds, &w[0], &w[4], pw_len);

    u64 *w_ptr = (u64 *) w;

    w_ptr[out_len / 8] |= (u64) out_len << 56;

    u64x v0 = v0p;
    u64x v1 = v1p;
    u64x v2 = v2p;
    u64x v3 = v3p;

    int i;
    int j;

    for (i = 0, j = 0; i <= pw_len; i += 8, j += 2)
    {
      u64x m = hl32_to_64 (w[j + 1], w[j + 0]);

      v3 ^= m;

      SIPROUND (v0, v1, v2, v3);
      SIPROUND (v0, v1, v2, v3);

      v0 ^= m;
    }

    v2 ^= 0xff;

    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);

    const u64x v = v0 ^ v1 ^ v2 ^ v3;

    const u32x a = l32_from_64 (v);
    const u32x b = h32_from_64 (v);

    const u32x r0 = a;
    const u32x r1 = b;
    const u32x r2 = 0;
    const u32x r3 = 0;

    #include VECT_COMPARE_S
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m10100_s08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m10100_s16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}
