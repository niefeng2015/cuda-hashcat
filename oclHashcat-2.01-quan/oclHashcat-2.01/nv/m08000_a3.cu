#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _SHA256_
#define _SCALAR_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 3
#define DGST_R1 7
#define DGST_R2 2
#define DGST_R3 6

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_S "check_single_vect1_comp4_warp.c"
#define VECT_COMPARE_M "check_multi_vect1_comp4_warp.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_S "check_single_vect2_comp4_warp.c"
#define VECT_COMPARE_M "check_multi_vect2_comp4_warp.c"
#endif

__device__ static void sha256_transform (u32x digest[8], const u32x w[16])
{
  u32x w0_t = w[ 0];
  u32x w1_t = w[ 1];
  u32x w2_t = w[ 2];
  u32x w3_t = w[ 3];
  u32x w4_t = w[ 4];
  u32x w5_t = w[ 5];
  u32x w6_t = w[ 6];
  u32x w7_t = w[ 7];
  u32x w8_t = w[ 8];
  u32x w9_t = w[ 9];
  u32x wa_t = w[10];
  u32x wb_t = w[11];
  u32x wc_t = w[12];
  u32x wd_t = w[13];
  u32x we_t = w[14];
  u32x wf_t = w[15];

  u32x a = digest[0];
  u32x b = digest[1];
  u32x c = digest[2];
  u32x d = digest[3];
  u32x e = digest[4];
  u32x f = digest[5];
  u32x g = digest[6];
  u32x h = digest[7];

  SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C00);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C01);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C02);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C03);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C04);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C05);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C06);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C07);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C08);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C09);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C0a);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C0b);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C0c);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C0d);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C0e);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C0f);
  w0_t = SHA256_S1(we_t) + w9_t + SHA256_S0(w1_t) + w0_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C10);
  w1_t = SHA256_S1(wf_t) + wa_t + SHA256_S0(w2_t) + w1_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C11);
  w2_t = SHA256_S1(w0_t) + wb_t + SHA256_S0(w3_t) + w2_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C12);
  w3_t = SHA256_S1(w1_t) + wc_t + SHA256_S0(w4_t) + w3_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C13);
  w4_t = SHA256_S1(w2_t) + wd_t + SHA256_S0(w5_t) + w4_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C14);
  w5_t = SHA256_S1(w3_t) + we_t + SHA256_S0(w6_t) + w5_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C15);
  w6_t = SHA256_S1(w4_t) + wf_t + SHA256_S0(w7_t) + w6_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C16);
  w7_t = SHA256_S1(w5_t) + w0_t + SHA256_S0(w8_t) + w7_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C17);
  w8_t = SHA256_S1(w6_t) + w1_t + SHA256_S0(w9_t) + w8_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C18);
  w9_t = SHA256_S1(w7_t) + w2_t + SHA256_S0(wa_t) + w9_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C19);
  wa_t = SHA256_S1(w8_t) + w3_t + SHA256_S0(wb_t) + wa_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C1a);
  wb_t = SHA256_S1(w9_t) + w4_t + SHA256_S0(wc_t) + wb_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C1b);
  wc_t = SHA256_S1(wa_t) + w5_t + SHA256_S0(wd_t) + wc_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C1c);
  wd_t = SHA256_S1(wb_t) + w6_t + SHA256_S0(we_t) + wd_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C1d);
  we_t = SHA256_S1(wc_t) + w7_t + SHA256_S0(wf_t) + we_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C1e);
  wf_t = SHA256_S1(wd_t) + w8_t + SHA256_S0(w0_t) + wf_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C1f);
  w0_t = SHA256_S1(we_t) + w9_t + SHA256_S0(w1_t) + w0_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C20);
  w1_t = SHA256_S1(wf_t) + wa_t + SHA256_S0(w2_t) + w1_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C21);
  w2_t = SHA256_S1(w0_t) + wb_t + SHA256_S0(w3_t) + w2_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C22);
  w3_t = SHA256_S1(w1_t) + wc_t + SHA256_S0(w4_t) + w3_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C23);
  w4_t = SHA256_S1(w2_t) + wd_t + SHA256_S0(w5_t) + w4_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C24);
  w5_t = SHA256_S1(w3_t) + we_t + SHA256_S0(w6_t) + w5_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C25);
  w6_t = SHA256_S1(w4_t) + wf_t + SHA256_S0(w7_t) + w6_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C26);
  w7_t = SHA256_S1(w5_t) + w0_t + SHA256_S0(w8_t) + w7_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C27);
  w8_t = SHA256_S1(w6_t) + w1_t + SHA256_S0(w9_t) + w8_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C28);
  w9_t = SHA256_S1(w7_t) + w2_t + SHA256_S0(wa_t) + w9_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C29);
  wa_t = SHA256_S1(w8_t) + w3_t + SHA256_S0(wb_t) + wa_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C2a);
  wb_t = SHA256_S1(w9_t) + w4_t + SHA256_S0(wc_t) + wb_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C2b);
  wc_t = SHA256_S1(wa_t) + w5_t + SHA256_S0(wd_t) + wc_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C2c);
  wd_t = SHA256_S1(wb_t) + w6_t + SHA256_S0(we_t) + wd_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C2d);
  we_t = SHA256_S1(wc_t) + w7_t + SHA256_S0(wf_t) + we_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C2e);
  wf_t = SHA256_S1(wd_t) + w8_t + SHA256_S0(w0_t) + wf_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C2f);
  w0_t = SHA256_S1(we_t) + w9_t + SHA256_S0(w1_t) + w0_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C30);
  w1_t = SHA256_S1(wf_t) + wa_t + SHA256_S0(w2_t) + w1_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C31);
  w2_t = SHA256_S1(w0_t) + wb_t + SHA256_S0(w3_t) + w2_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C32);
  w3_t = SHA256_S1(w1_t) + wc_t + SHA256_S0(w4_t) + w3_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C33);
  w4_t = SHA256_S1(w2_t) + wd_t + SHA256_S0(w5_t) + w4_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C34);
  w5_t = SHA256_S1(w3_t) + we_t + SHA256_S0(w6_t) + w5_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C35);
  w6_t = SHA256_S1(w4_t) + wf_t + SHA256_S0(w7_t) + w6_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C36);
  w7_t = SHA256_S1(w5_t) + w0_t + SHA256_S0(w8_t) + w7_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C37);
  w8_t = SHA256_S1(w6_t) + w1_t + SHA256_S0(w9_t) + w8_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C38);
  w9_t = SHA256_S1(w7_t) + w2_t + SHA256_S0(wa_t) + w9_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C39);
  wa_t = SHA256_S1(w8_t) + w3_t + SHA256_S0(wb_t) + wa_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C3a);
  wb_t = SHA256_S1(w9_t) + w4_t + SHA256_S0(wc_t) + wb_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C3b);
  wc_t = SHA256_S1(wa_t) + w5_t + SHA256_S0(wd_t) + wc_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C3c);
  wd_t = SHA256_S1(wb_t) + w6_t + SHA256_S0(we_t) + wd_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C3d);
  we_t = SHA256_S1(wc_t) + w7_t + SHA256_S0(wf_t) + we_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C3e);
  wf_t = SHA256_S1(wd_t) + w8_t + SHA256_S0(w0_t) + wf_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C3f);

  digest[0] += a;
  digest[1] += b;
  digest[2] += c;
  digest[3] += d;
  digest[4] += e;
  digest[5] += f;
  digest[6] += g;
  digest[7] += h;
}

__device__ __constant__ u32x c_bfs[1024];

__device__ static void m08000m (u32 w[16], const u32 pw_len, const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset)
{
  /**
   * modifier
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;
  const u32 lid = threadIdx.x;

  /**
   * salt
   */

  const u32 salt_buf0 = swap_workaround (salt_bufs[salt_pos].salt_buf[ 0]);
  const u32 salt_buf1 = swap_workaround (salt_bufs[salt_pos].salt_buf[ 1]);
  const u32 salt_buf2 = swap_workaround (salt_bufs[salt_pos].salt_buf[ 2]); // 0x80

  /**
   * loop
   */

  const u32 bf_loops = ceil ((float) bfs_cnt / VECT_DIV);

  u32x w0l = w[0];

  for (u32 il_pos = 0; il_pos < bf_loops; il_pos++)
  {
    const u32x w0r = c_bfs[il_pos];

    const u32x w0 = w0l | w0r;

    u32x w_t[16];

    w_t[ 0] = w0    >> 8;
    w_t[ 1] = w[ 1] >> 8;
    w_t[ 2] = w[ 2] >> 8;
    w_t[ 3] = w[ 3] >> 8;
    w_t[ 4] = w[ 4] >> 8;
    w_t[ 5] = w[ 5] >> 8;
    w_t[ 6] = w[ 6] >> 8;
    w_t[ 7] = w[ 7] >> 8;
    w_t[ 8] = w[ 8] >> 8;
    w_t[ 9] = w[ 9] >> 8;
    w_t[10] = w[10] >> 8;
    w_t[11] = w[11] >> 8;
    w_t[12] = w[12] >> 8;
    w_t[13] = w[13] >> 8;
    w_t[14] = w[14] >> 8;
    w_t[15] = w[15] >> 8;

    u32x digest[8];

    digest[0] = SHA256M_A;
    digest[1] = SHA256M_B;
    digest[2] = SHA256M_C;
    digest[3] = SHA256M_D;
    digest[4] = SHA256M_E;
    digest[5] = SHA256M_F;
    digest[6] = SHA256M_G;
    digest[7] = SHA256M_H;

    sha256_transform (digest, w_t); //   0 - 64

    w_t[ 0] = 0;
    w_t[ 1] = 0;
    w_t[ 2] = 0;
    w_t[ 3] = 0;
    w_t[ 4] = 0;
    w_t[ 5] = 0;
    w_t[ 6] = 0;
    w_t[ 7] = 0;
    w_t[ 8] = 0;
    w_t[ 9] = 0;
    w_t[10] = 0;
    w_t[11] = 0;
    w_t[12] = 0;
    w_t[13] = 0;
    w_t[14] = 0;
    w_t[15] = 0;

    sha256_transform (digest, w_t); //  64 - 128
    sha256_transform (digest, w_t); // 128 - 192
    sha256_transform (digest, w_t); // 192 - 256
    sha256_transform (digest, w_t); // 256 - 320
    sha256_transform (digest, w_t); // 320 - 384
    sha256_transform (digest, w_t); // 384 - 448

    w_t[15] =               0 | salt_buf0 >> 16;

    sha256_transform (digest, w_t); // 448 - 512

    w_t[ 0] = salt_buf0 << 16 | salt_buf1 >> 16;
    w_t[ 1] = salt_buf1 << 16 | salt_buf2 >> 16;
    w_t[ 2] = salt_buf2 << 16 | 0;
    w_t[15] = (510 + 8) * 8;

    sha256_transform (digest, w_t); // 512 - 576

    const u32x r0 = digest[3];
    const u32x r1 = digest[7];
    const u32x r2 = digest[2];
    const u32x r3 = digest[6];

    #include VECT_COMPARE_M
  }
}

__device__ static void m08000s (u32 w[16], const u32 pw_len, const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset)
{
  /**
   * modifier
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;
  const u32 lid = threadIdx.x;

  /**
   * digest
   */

  const u32 search[4] =
  {
    digests_buf[digests_offset].digest_buf[DGST_R0],
    digests_buf[digests_offset].digest_buf[DGST_R1],
    digests_buf[digests_offset].digest_buf[DGST_R2],
    digests_buf[digests_offset].digest_buf[DGST_R3]
  };

  /**
   * salt
   */

  const u32 salt_buf0 = swap_workaround (salt_bufs[salt_pos].salt_buf[ 0]);
  const u32 salt_buf1 = swap_workaround (salt_bufs[salt_pos].salt_buf[ 1]);
  const u32 salt_buf2 = swap_workaround (salt_bufs[salt_pos].salt_buf[ 2]); // 0x80

  /**
   * loop
   */

  const u32 bf_loops = ceil ((float) bfs_cnt / VECT_DIV);

  u32x w0l = w[0];

  for (u32 il_pos = 0; il_pos < bf_loops; il_pos++)
  {
    const u32x w0r = c_bfs[il_pos];

    const u32x w0 = w0l | w0r;

    u32x w_t[16];

    w_t[ 0] = w0    >> 8;
    w_t[ 1] = w[ 1] >> 8;
    w_t[ 2] = w[ 2] >> 8;
    w_t[ 3] = w[ 3] >> 8;
    w_t[ 4] = w[ 4] >> 8;
    w_t[ 5] = w[ 5] >> 8;
    w_t[ 6] = w[ 6] >> 8;
    w_t[ 7] = w[ 7] >> 8;
    w_t[ 8] = w[ 8] >> 8;
    w_t[ 9] = w[ 9] >> 8;
    w_t[10] = w[10] >> 8;
    w_t[11] = w[11] >> 8;
    w_t[12] = w[12] >> 8;
    w_t[13] = w[13] >> 8;
    w_t[14] = w[14] >> 8;
    w_t[15] = w[15] >> 8;

    u32x digest[8];

    digest[0] = SHA256M_A;
    digest[1] = SHA256M_B;
    digest[2] = SHA256M_C;
    digest[3] = SHA256M_D;
    digest[4] = SHA256M_E;
    digest[5] = SHA256M_F;
    digest[6] = SHA256M_G;
    digest[7] = SHA256M_H;

    sha256_transform (digest, w_t); //   0 - 64

    w_t[ 0] = 0;
    w_t[ 1] = 0;
    w_t[ 2] = 0;
    w_t[ 3] = 0;
    w_t[ 4] = 0;
    w_t[ 5] = 0;
    w_t[ 6] = 0;
    w_t[ 7] = 0;
    w_t[ 8] = 0;
    w_t[ 9] = 0;
    w_t[10] = 0;
    w_t[11] = 0;
    w_t[12] = 0;
    w_t[13] = 0;
    w_t[14] = 0;
    w_t[15] = 0;

    sha256_transform (digest, w_t); //  64 - 128
    sha256_transform (digest, w_t); // 128 - 192
    sha256_transform (digest, w_t); // 192 - 256
    sha256_transform (digest, w_t); // 256 - 320
    sha256_transform (digest, w_t); // 320 - 384
    sha256_transform (digest, w_t); // 384 - 448

    w_t[15] =               0 | salt_buf0 >> 16;

    sha256_transform (digest, w_t); // 448 - 512

    w_t[ 0] = salt_buf0 << 16 | salt_buf1 >> 16;
    w_t[ 1] = salt_buf1 << 16 | salt_buf2 >> 16;
    w_t[ 2] = salt_buf2 << 16 | 0;
    w_t[15] = (510 + 8) * 8;

    sha256_transform (digest, w_t); // 512 - 576

    const u32x r0 = digest[3];
    const u32x r1 = digest[7];
    const u32x r2 = digest[2];
    const u32x r3 = digest[6];

    #include VECT_COMPARE_S
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m08000_m04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = 0;
  w[ 5] = 0;
  w[ 6] = 0;
  w[ 7] = 0;
  w[ 8] = 0;
  w[ 9] = 0;
  w[10] = 0;
  w[11] = 0;
  w[12] = 0;
  w[13] = 0;
  w[14] = 0;
  w[15] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m08000m (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m08000_m08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = pws[gid].i[ 4];
  w[ 5] = pws[gid].i[ 5];
  w[ 6] = pws[gid].i[ 6];
  w[ 7] = pws[gid].i[ 7];
  w[ 8] = 0;
  w[ 9] = 0;
  w[10] = 0;
  w[11] = 0;
  w[12] = 0;
  w[13] = 0;
  w[14] = 0;
  w[15] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m08000m (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m08000_m16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = pws[gid].i[ 4];
  w[ 5] = pws[gid].i[ 5];
  w[ 6] = pws[gid].i[ 6];
  w[ 7] = pws[gid].i[ 7];
  w[ 8] = pws[gid].i[ 8];
  w[ 9] = pws[gid].i[ 9];
  w[10] = pws[gid].i[10];
  w[11] = pws[gid].i[11];
  w[12] = pws[gid].i[12];
  w[13] = pws[gid].i[13];
  w[14] = pws[gid].i[14];
  w[15] = pws[gid].i[15];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m08000m (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m08000_s04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = 0;
  w[ 5] = 0;
  w[ 6] = 0;
  w[ 7] = 0;
  w[ 8] = 0;
  w[ 9] = 0;
  w[10] = 0;
  w[11] = 0;
  w[12] = 0;
  w[13] = 0;
  w[14] = 0;
  w[15] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m08000s (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m08000_s08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = pws[gid].i[ 4];
  w[ 5] = pws[gid].i[ 5];
  w[ 6] = pws[gid].i[ 6];
  w[ 7] = pws[gid].i[ 7];
  w[ 8] = 0;
  w[ 9] = 0;
  w[10] = 0;
  w[11] = 0;
  w[12] = 0;
  w[13] = 0;
  w[14] = 0;
  w[15] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m08000s (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m08000_s16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = pws[gid].i[ 4];
  w[ 5] = pws[gid].i[ 5];
  w[ 6] = pws[gid].i[ 6];
  w[ 7] = pws[gid].i[ 7];
  w[ 8] = pws[gid].i[ 8];
  w[ 9] = pws[gid].i[ 9];
  w[10] = pws[gid].i[10];
  w[11] = pws[gid].i[11];
  w[12] = pws[gid].i[12];
  w[13] = pws[gid].i[13];
  w[14] = pws[gid].i[14];
  w[15] = pws[gid].i[15];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m08000s (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}
