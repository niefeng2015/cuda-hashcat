#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _OLDOFFICE01_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 0
#define DGST_R1 1
#define DGST_R2 2
#define DGST_R3 3

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_S "check_single_vect1_comp4.c"
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE4
#define VECT_COMPARE_S "check_single_vect4_comp4.c"
#define VECT_COMPARE_M "check_multi_vect4_comp4.c"
#endif

typedef struct
{
  u8 S[256];

  u32 wtf_its_faster;

} RC4_KEY;

__device__ static void swap (RC4_KEY *rc4_key, const u8 i, const u8 j)
{
  u8 tmp;

  tmp           = rc4_key->S[i];
  rc4_key->S[i] = rc4_key->S[j];
  rc4_key->S[j] = tmp;
}

__device__ static void rc4_init_16 (RC4_KEY *rc4_key, const u32 data[4])
{
  u32 v = 0x03020100;
  u32 a = 0x04040404;

  u32 *ptr = (u32 *) rc4_key->S;

  #pragma unroll 64
  for (u32 i = 0; i < 64; i++)
  {
    *ptr++ = v; v += a;
  }

  u32 j = 0;

  for (u32 i = 0; i < 16; i++)
  {
    u32 idx = i * 16;

    u32 v;

    v = data[0];

    j += rc4_key->S[idx] + (v >>  0); swap (rc4_key, idx, j); idx++;
    j += rc4_key->S[idx] + (v >>  8); swap (rc4_key, idx, j); idx++;
    j += rc4_key->S[idx] + (v >> 16); swap (rc4_key, idx, j); idx++;
    j += rc4_key->S[idx] + (v >> 24); swap (rc4_key, idx, j); idx++;

    v = data[1];

    j += rc4_key->S[idx] + (v >>  0); swap (rc4_key, idx, j); idx++;
    j += rc4_key->S[idx] + (v >>  8); swap (rc4_key, idx, j); idx++;
    j += rc4_key->S[idx] + (v >> 16); swap (rc4_key, idx, j); idx++;
    j += rc4_key->S[idx] + (v >> 24); swap (rc4_key, idx, j); idx++;

    v = data[2];

    j += rc4_key->S[idx] + (v >>  0); swap (rc4_key, idx, j); idx++;
    j += rc4_key->S[idx] + (v >>  8); swap (rc4_key, idx, j); idx++;
    j += rc4_key->S[idx] + (v >> 16); swap (rc4_key, idx, j); idx++;
    j += rc4_key->S[idx] + (v >> 24); swap (rc4_key, idx, j); idx++;

    v = data[3];

    j += rc4_key->S[idx] + (v >>  0); swap (rc4_key, idx, j); idx++;
    j += rc4_key->S[idx] + (v >>  8); swap (rc4_key, idx, j); idx++;
    j += rc4_key->S[idx] + (v >> 16); swap (rc4_key, idx, j); idx++;
    j += rc4_key->S[idx] + (v >> 24); swap (rc4_key, idx, j); idx++;
  }
}

__device__ static u8 rc4_next_16 (RC4_KEY *rc4_key, u8 i, u8 j, const u32 in[4], u32 out[4])
{
  for (u32 k = 0; k < 4; k++)
  {
    u32 xor4 = 0;

    u8 idx;

    i += 1;
    j += rc4_key->S[i];

    swap (rc4_key, i, j);

    idx = rc4_key->S[i] + rc4_key->S[j];

    xor4 |= rc4_key->S[idx] <<  0;

    i += 1;
    j += rc4_key->S[i];

    swap (rc4_key, i, j);

    idx = rc4_key->S[i] + rc4_key->S[j];

    xor4 |= rc4_key->S[idx] <<  8;

    i += 1;
    j += rc4_key->S[i];

    swap (rc4_key, i, j);

    idx = rc4_key->S[i] + rc4_key->S[j];

    xor4 |= rc4_key->S[idx] << 16;

    i += 1;
    j += rc4_key->S[i];

    swap (rc4_key, i, j);

    idx = rc4_key->S[i] + rc4_key->S[j];

    xor4 |= rc4_key->S[idx] << 24;

    out[k] = in[k] ^ xor4;
  }

  return j;
}

__device__ static void md5_transform (const u32x w0[4], const u32x w1[4], const u32x w2[4], const u32x w3[4], u32x digest[4])
{
  u32x a = digest[0];
  u32x b = digest[1];
  u32x c = digest[2];
  u32x d = digest[3];

  u32x w0_t = w0[0];
  u32x w1_t = w0[1];
  u32x w2_t = w0[2];
  u32x w3_t = w0[3];
  u32x w4_t = w1[0];
  u32x w5_t = w1[1];
  u32x w6_t = w1[2];
  u32x w7_t = w1[3];
  u32x w8_t = w2[0];
  u32x w9_t = w2[1];
  u32x wa_t = w2[2];
  u32x wb_t = w2[3];
  u32x wc_t = w3[0];
  u32x wd_t = w3[1];
  u32x we_t = w3[2];
  u32x wf_t = w3[3];

  MD5_STEP (MD5_Fo, a, b, c, d, w0_t, MD5C00, MD5S00);
  MD5_STEP (MD5_Fo, d, a, b, c, w1_t, MD5C01, MD5S01);
  MD5_STEP (MD5_Fo, c, d, a, b, w2_t, MD5C02, MD5S02);
  MD5_STEP (MD5_Fo, b, c, d, a, w3_t, MD5C03, MD5S03);
  MD5_STEP (MD5_Fo, a, b, c, d, w4_t, MD5C04, MD5S00);
  MD5_STEP (MD5_Fo, d, a, b, c, w5_t, MD5C05, MD5S01);
  MD5_STEP (MD5_Fo, c, d, a, b, w6_t, MD5C06, MD5S02);
  MD5_STEP (MD5_Fo, b, c, d, a, w7_t, MD5C07, MD5S03);
  MD5_STEP (MD5_Fo, a, b, c, d, w8_t, MD5C08, MD5S00);
  MD5_STEP (MD5_Fo, d, a, b, c, w9_t, MD5C09, MD5S01);
  MD5_STEP (MD5_Fo, c, d, a, b, wa_t, MD5C0a, MD5S02);
  MD5_STEP (MD5_Fo, b, c, d, a, wb_t, MD5C0b, MD5S03);
  MD5_STEP (MD5_Fo, a, b, c, d, wc_t, MD5C0c, MD5S00);
  MD5_STEP (MD5_Fo, d, a, b, c, wd_t, MD5C0d, MD5S01);
  MD5_STEP (MD5_Fo, c, d, a, b, we_t, MD5C0e, MD5S02);
  MD5_STEP (MD5_Fo, b, c, d, a, wf_t, MD5C0f, MD5S03);

  MD5_STEP (MD5_Go, a, b, c, d, w1_t, MD5C10, MD5S10);
  MD5_STEP (MD5_Go, d, a, b, c, w6_t, MD5C11, MD5S11);
  MD5_STEP (MD5_Go, c, d, a, b, wb_t, MD5C12, MD5S12);
  MD5_STEP (MD5_Go, b, c, d, a, w0_t, MD5C13, MD5S13);
  MD5_STEP (MD5_Go, a, b, c, d, w5_t, MD5C14, MD5S10);
  MD5_STEP (MD5_Go, d, a, b, c, wa_t, MD5C15, MD5S11);
  MD5_STEP (MD5_Go, c, d, a, b, wf_t, MD5C16, MD5S12);
  MD5_STEP (MD5_Go, b, c, d, a, w4_t, MD5C17, MD5S13);
  MD5_STEP (MD5_Go, a, b, c, d, w9_t, MD5C18, MD5S10);
  MD5_STEP (MD5_Go, d, a, b, c, we_t, MD5C19, MD5S11);
  MD5_STEP (MD5_Go, c, d, a, b, w3_t, MD5C1a, MD5S12);
  MD5_STEP (MD5_Go, b, c, d, a, w8_t, MD5C1b, MD5S13);
  MD5_STEP (MD5_Go, a, b, c, d, wd_t, MD5C1c, MD5S10);
  MD5_STEP (MD5_Go, d, a, b, c, w2_t, MD5C1d, MD5S11);
  MD5_STEP (MD5_Go, c, d, a, b, w7_t, MD5C1e, MD5S12);
  MD5_STEP (MD5_Go, b, c, d, a, wc_t, MD5C1f, MD5S13);

  MD5_STEP (MD5_H , a, b, c, d, w5_t, MD5C20, MD5S20);
  MD5_STEP (MD5_H , d, a, b, c, w8_t, MD5C21, MD5S21);
  MD5_STEP (MD5_H , c, d, a, b, wb_t, MD5C22, MD5S22);
  MD5_STEP (MD5_H , b, c, d, a, we_t, MD5C23, MD5S23);
  MD5_STEP (MD5_H , a, b, c, d, w1_t, MD5C24, MD5S20);
  MD5_STEP (MD5_H , d, a, b, c, w4_t, MD5C25, MD5S21);
  MD5_STEP (MD5_H , c, d, a, b, w7_t, MD5C26, MD5S22);
  MD5_STEP (MD5_H , b, c, d, a, wa_t, MD5C27, MD5S23);
  MD5_STEP (MD5_H , a, b, c, d, wd_t, MD5C28, MD5S20);
  MD5_STEP (MD5_H , d, a, b, c, w0_t, MD5C29, MD5S21);
  MD5_STEP (MD5_H , c, d, a, b, w3_t, MD5C2a, MD5S22);
  MD5_STEP (MD5_H , b, c, d, a, w6_t, MD5C2b, MD5S23);
  MD5_STEP (MD5_H , a, b, c, d, w9_t, MD5C2c, MD5S20);
  MD5_STEP (MD5_H , d, a, b, c, wc_t, MD5C2d, MD5S21);
  MD5_STEP (MD5_H , c, d, a, b, wf_t, MD5C2e, MD5S22);
  MD5_STEP (MD5_H , b, c, d, a, w2_t, MD5C2f, MD5S23);

  MD5_STEP (MD5_I , a, b, c, d, w0_t, MD5C30, MD5S30);
  MD5_STEP (MD5_I , d, a, b, c, w7_t, MD5C31, MD5S31);
  MD5_STEP (MD5_I , c, d, a, b, we_t, MD5C32, MD5S32);
  MD5_STEP (MD5_I , b, c, d, a, w5_t, MD5C33, MD5S33);
  MD5_STEP (MD5_I , a, b, c, d, wc_t, MD5C34, MD5S30);
  MD5_STEP (MD5_I , d, a, b, c, w3_t, MD5C35, MD5S31);
  MD5_STEP (MD5_I , c, d, a, b, wa_t, MD5C36, MD5S32);
  MD5_STEP (MD5_I , b, c, d, a, w1_t, MD5C37, MD5S33);
  MD5_STEP (MD5_I , a, b, c, d, w8_t, MD5C38, MD5S30);
  MD5_STEP (MD5_I , d, a, b, c, wf_t, MD5C39, MD5S31);
  MD5_STEP (MD5_I , c, d, a, b, w6_t, MD5C3a, MD5S32);
  MD5_STEP (MD5_I , b, c, d, a, wd_t, MD5C3b, MD5S33);
  MD5_STEP (MD5_I , a, b, c, d, w4_t, MD5C3c, MD5S30);
  MD5_STEP (MD5_I , d, a, b, c, wb_t, MD5C3d, MD5S31);
  MD5_STEP (MD5_I , c, d, a, b, w2_t, MD5C3e, MD5S32);
  MD5_STEP (MD5_I , b, c, d, a, w9_t, MD5C3f, MD5S33);

  digest[0] += a;
  digest[1] += b;
  digest[2] += c;
  digest[3] += d;
}

__device__ __constant__ bf_t c_bfs[1024];

__device__ static void m09700m (RC4_KEY rc4_keys[64], u32x w0[4], u32x w1[4], u32x w2[4], u32x w3[4], const u32 pw_len, const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *bfs_buf, void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const oldoffice01_t *oldoffice01_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset)
{
  /**
   * modifier
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;
  const u32 lid = threadIdx.x;

  RC4_KEY *rc4_key = &rc4_keys[lid];

  /**
   * salt
   */

  u32 salt_buf_t0[4];
  u32 salt_buf_t1[5];
  u32 salt_buf_t2[5];
  u32 salt_buf_t3[5];

  salt_buf_t0[0] = salt_bufs[salt_pos].salt_buf[0];
  salt_buf_t0[1] = salt_bufs[salt_pos].salt_buf[1];
  salt_buf_t0[2] = salt_bufs[salt_pos].salt_buf[2];
  salt_buf_t0[3] = salt_bufs[salt_pos].salt_buf[3];

  salt_buf_t1[0] =                        salt_buf_t0[0] <<  8;
  salt_buf_t1[1] = salt_buf_t0[0] >> 24 | salt_buf_t0[1] <<  8;
  salt_buf_t1[2] = salt_buf_t0[1] >> 24 | salt_buf_t0[2] <<  8;
  salt_buf_t1[3] = salt_buf_t0[2] >> 24 | salt_buf_t0[3] <<  8;
  salt_buf_t1[4] = salt_buf_t0[3] >> 24;

  salt_buf_t2[0] =                        salt_buf_t0[0] << 16;
  salt_buf_t2[1] = salt_buf_t0[0] >> 16 | salt_buf_t0[1] << 16;
  salt_buf_t2[2] = salt_buf_t0[1] >> 16 | salt_buf_t0[2] << 16;
  salt_buf_t2[3] = salt_buf_t0[2] >> 16 | salt_buf_t0[3] << 16;
  salt_buf_t2[4] = salt_buf_t0[3] >> 16;

  salt_buf_t3[0] =                        salt_buf_t0[0] << 24;
  salt_buf_t3[1] = salt_buf_t0[0] >>  8 | salt_buf_t0[1] << 24;
  salt_buf_t3[2] = salt_buf_t0[1] >>  8 | salt_buf_t0[2] << 24;
  salt_buf_t3[3] = salt_buf_t0[2] >>  8 | salt_buf_t0[3] << 24;
  salt_buf_t3[4] = salt_buf_t0[3] >>  8;

  const u32 salt_len = 16;

  /**
   * esalt
   */

  const u32 version = oldoffice01_bufs[salt_pos].version;

  u32 encryptedVerifier[4];

  encryptedVerifier[0] = oldoffice01_bufs[salt_pos].encryptedVerifier[0];
  encryptedVerifier[1] = oldoffice01_bufs[salt_pos].encryptedVerifier[1];
  encryptedVerifier[2] = oldoffice01_bufs[salt_pos].encryptedVerifier[2];
  encryptedVerifier[3] = oldoffice01_bufs[salt_pos].encryptedVerifier[3];

  /**
   * loop
   */

  u32x w0l = w0[0];

  for (u32 il_pos = 0; il_pos < bfs_cnt; il_pos++)
  {
    const u32 w0r = c_bfs[il_pos].i;

    w0[0] = w0l | w0r;

    u32x w0_t[4];
    u32x w1_t[4];
    u32x w2_t[4];
    u32x w3_t[4];

    w0_t[0] = w0[0];
    w0_t[1] = w0[1];
    w0_t[2] = w0[2];
    w0_t[3] = w0[3];
    w1_t[0] = w1[0];
    w1_t[1] = w1[1];
    w1_t[2] = w1[2];
    w1_t[3] = w1[3];
    w2_t[0] = w2[0];
    w2_t[1] = w2[1];
    w2_t[2] = w2[2];
    w2_t[3] = w2[3];
    w3_t[0] = w3[0];
    w3_t[1] = w3[1];
    w3_t[2] = pw_len * 8;
    w3_t[3] = 0;

    u32x digest_t0[4];
    u32x digest_t1[2]; // need only first 5 byte
    u32x digest_t2[2];
    u32x digest_t3[2];

    digest_t0[0] = MD5M_A;
    digest_t0[1] = MD5M_B;
    digest_t0[2] = MD5M_C;
    digest_t0[3] = MD5M_D;

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest_t0);

    // prepare 16 * 21 buffer stuff

    u32x digest[4];

    digest[0] = MD5M_A;
    digest[1] = MD5M_B;
    digest[2] = MD5M_C;
    digest[3] = MD5M_D;

    // offsets

    digest_t0[0] &= 0xffffffff;
    digest_t0[1] &= 0x000000ff;
    digest_t0[2] &= 0x00000000;
    digest_t0[3] &= 0x00000000;

    digest_t1[0] =                      digest_t0[0] <<  8;
    digest_t1[1] = digest_t0[0] >> 24 | digest_t0[1] <<  8;

    digest_t2[0] =                      digest_t0[0] << 16;
    digest_t2[1] = digest_t0[0] >> 16 | digest_t0[1] << 16;

    digest_t3[0] =                      digest_t0[0] << 24;
    digest_t3[1] = digest_t0[0] >>  8 | digest_t0[1] << 24;

    // generate the 16 * 21 buffer

    w0_t[0] = 0;
    w0_t[1] = 0;
    w0_t[2] = 0;
    w0_t[3] = 0;
    w1_t[0] = 0;
    w1_t[1] = 0;
    w1_t[2] = 0;
    w1_t[3] = 0;
    w2_t[0] = 0;
    w2_t[1] = 0;
    w2_t[2] = 0;
    w2_t[3] = 0;
    w3_t[0] = 0;
    w3_t[1] = 0;
    w3_t[2] = 0;
    w3_t[3] = 0;

    // 0..5
    w0_t[0]  = digest_t0[0];
    w0_t[1]  = digest_t0[1];

    // 5..21
    w0_t[1] |= salt_buf_t1[0];
    w0_t[2]  = salt_buf_t1[1];
    w0_t[3]  = salt_buf_t1[2];
    w1_t[0]  = salt_buf_t1[3];
    w1_t[1]  = salt_buf_t1[4];

    // 21..26
    w1_t[1] |= digest_t1[0];
    w1_t[2]  = digest_t1[1];

    // 26..42
    w1_t[2] |= salt_buf_t2[0];
    w1_t[3]  = salt_buf_t2[1];
    w2_t[0]  = salt_buf_t2[2];
    w2_t[1]  = salt_buf_t2[3];
    w2_t[2]  = salt_buf_t2[4];

    // 42..47
    w2_t[2] |= digest_t2[0];
    w2_t[3]  = digest_t2[1];

    // 47..63
    w2_t[3] |= salt_buf_t3[0];
    w3_t[0]  = salt_buf_t3[1];
    w3_t[1]  = salt_buf_t3[2];
    w3_t[2]  = salt_buf_t3[3];
    w3_t[3]  = salt_buf_t3[4];

    // 63..

    w3_t[3] |= digest_t3[0];

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    w0_t[0] = 0;
    w0_t[1] = 0;
    w0_t[2] = 0;
    w0_t[3] = 0;
    w1_t[0] = 0;
    w1_t[1] = 0;
    w1_t[2] = 0;
    w1_t[3] = 0;
    w2_t[0] = 0;
    w2_t[1] = 0;
    w2_t[2] = 0;
    w2_t[3] = 0;
    w3_t[0] = 0;
    w3_t[1] = 0;
    w3_t[2] = 0;
    w3_t[3] = 0;

    // 0..4
    w0_t[0]  = digest_t3[1];

    // 4..20
    w0_t[1]  = salt_buf_t0[0];
    w0_t[2]  = salt_buf_t0[1];
    w0_t[3]  = salt_buf_t0[2];
    w1_t[0]  = salt_buf_t0[3];

    // 20..25
    w1_t[1]  = digest_t0[0];
    w1_t[2]  = digest_t0[1];

    // 25..41
    w1_t[2] |= salt_buf_t1[0];
    w1_t[3]  = salt_buf_t1[1];
    w2_t[0]  = salt_buf_t1[2];
    w2_t[1]  = salt_buf_t1[3];
    w2_t[2]  = salt_buf_t1[4];

    // 41..46
    w2_t[2] |= digest_t1[0];
    w2_t[3]  = digest_t1[1];

    // 46..62
    w2_t[3] |= salt_buf_t2[0];
    w3_t[0]  = salt_buf_t2[1];
    w3_t[1]  = salt_buf_t2[2];
    w3_t[2]  = salt_buf_t2[3];
    w3_t[3]  = salt_buf_t2[4];

    // 62..
    w3_t[3] |= digest_t2[0];

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    w0_t[0] = 0;
    w0_t[1] = 0;
    w0_t[2] = 0;
    w0_t[3] = 0;
    w1_t[0] = 0;
    w1_t[1] = 0;
    w1_t[2] = 0;
    w1_t[3] = 0;
    w2_t[0] = 0;
    w2_t[1] = 0;
    w2_t[2] = 0;
    w2_t[3] = 0;
    w3_t[0] = 0;
    w3_t[1] = 0;
    w3_t[2] = 0;
    w3_t[3] = 0;

    // 0..3
    w0_t[0]  = digest_t2[1];

    // 3..19
    w0_t[0] |= salt_buf_t3[0];
    w0_t[1]  = salt_buf_t3[1];
    w0_t[2]  = salt_buf_t3[2];
    w0_t[3]  = salt_buf_t3[3];
    w1_t[0]  = salt_buf_t3[4];

    // 19..24
    w1_t[0] |= digest_t3[0];
    w1_t[1]  = digest_t3[1];

    // 24..40
    w1_t[2]  = salt_buf_t0[0];
    w1_t[3]  = salt_buf_t0[1];
    w2_t[0]  = salt_buf_t0[2];
    w2_t[1]  = salt_buf_t0[3];

    // 40..45
    w2_t[2]  = digest_t0[0];
    w2_t[3]  = digest_t0[1];

    // 45..61
    w2_t[3] |= salt_buf_t1[0];
    w3_t[0]  = salt_buf_t1[1];
    w3_t[1]  = salt_buf_t1[2];
    w3_t[2]  = salt_buf_t1[3];
    w3_t[3]  = salt_buf_t1[4];

    // 61..
    w3_t[3] |= digest_t1[0];

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    w0_t[0] = 0;
    w0_t[1] = 0;
    w0_t[2] = 0;
    w0_t[3] = 0;
    w1_t[0] = 0;
    w1_t[1] = 0;
    w1_t[2] = 0;
    w1_t[3] = 0;
    w2_t[0] = 0;
    w2_t[1] = 0;
    w2_t[2] = 0;
    w2_t[3] = 0;
    w3_t[0] = 0;
    w3_t[1] = 0;
    w3_t[2] = 0;
    w3_t[3] = 0;

    // 0..2
    w0_t[0]  = digest_t1[1];

    // 2..18
    w0_t[0] |= salt_buf_t2[0];
    w0_t[1]  = salt_buf_t2[1];
    w0_t[2]  = salt_buf_t2[2];
    w0_t[3]  = salt_buf_t2[3];
    w1_t[0]  = salt_buf_t2[4];

    // 18..23
    w1_t[0] |= digest_t2[0];
    w1_t[1]  = digest_t2[1];

    // 23..39
    w1_t[1] |= salt_buf_t3[0];
    w1_t[2]  = salt_buf_t3[1];
    w1_t[3]  = salt_buf_t3[2];
    w2_t[0]  = salt_buf_t3[3];
    w2_t[1]  = salt_buf_t3[4];

    // 39..44
    w2_t[1] |= digest_t3[0];
    w2_t[2]  = digest_t3[1];

    // 44..60
    w2_t[3]  = salt_buf_t0[0];
    w3_t[0]  = salt_buf_t0[1];
    w3_t[1]  = salt_buf_t0[2];
    w3_t[2]  = salt_buf_t0[3];

    // 60..
    w3_t[3]  = digest_t0[0];

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    w0_t[0] = 0;
    w0_t[1] = 0;
    w0_t[2] = 0;
    w0_t[3] = 0;
    w1_t[0] = 0;
    w1_t[1] = 0;
    w1_t[2] = 0;
    w1_t[3] = 0;
    w2_t[0] = 0;
    w2_t[1] = 0;
    w2_t[2] = 0;
    w2_t[3] = 0;
    w3_t[0] = 0;
    w3_t[1] = 0;
    w3_t[2] = 0;
    w3_t[3] = 0;

    // 0..1
    w0_t[0]  = digest_t0[1];

    // 1..17
    w0_t[0] |= salt_buf_t1[0];
    w0_t[1]  = salt_buf_t1[1];
    w0_t[2]  = salt_buf_t1[2];
    w0_t[3]  = salt_buf_t1[3];
    w1_t[0]  = salt_buf_t1[4];

    // 17..22
    w1_t[0] |= digest_t1[0];
    w1_t[1]  = digest_t1[1];

    // 22..38
    w1_t[1] |= salt_buf_t2[0];
    w1_t[2]  = salt_buf_t2[1];
    w1_t[3]  = salt_buf_t2[2];
    w2_t[0]  = salt_buf_t2[3];
    w2_t[1]  = salt_buf_t2[4];

    // 38..43
    w2_t[1] |= digest_t2[0];
    w2_t[2]  = digest_t2[1];

    // 43..59
    w2_t[2] |= salt_buf_t3[0];
    w2_t[3]  = salt_buf_t3[1];
    w3_t[0]  = salt_buf_t3[2];
    w3_t[1]  = salt_buf_t3[3];
    w3_t[2]  = salt_buf_t3[4];

    // 59..
    w3_t[2] |= digest_t3[0];
    w3_t[3]  = digest_t3[1];

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    w0_t[0]  = salt_buf_t0[0];
    w0_t[1]  = salt_buf_t0[1];
    w0_t[2]  = salt_buf_t0[2];
    w0_t[3]  = salt_buf_t0[3];
    w1_t[0]  = 0x80;
    w1_t[1]  = 0;
    w1_t[2]  = 0;
    w1_t[3]  = 0;
    w2_t[0]  = 0;
    w2_t[1]  = 0;
    w2_t[2]  = 0;
    w2_t[3]  = 0;
    w3_t[0]  = 0;
    w3_t[1]  = 0;
    w3_t[2]  = 21 * 16 * 8;
    w3_t[3]  = 0;

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    // now the 40 bit input for the MD5 which then will generate the RC4 key, so it's precomputable!

    w0_t[0]  = digest[0];
    w0_t[1]  = digest[1] & 0xff;
    w0_t[2]  = 0x8000;
    w0_t[3]  = 0;
    w1_t[0]  = 0;
    w1_t[1]  = 0;
    w1_t[2]  = 0;
    w1_t[3]  = 0;
    w2_t[0]  = 0;
    w2_t[1]  = 0;
    w2_t[2]  = 0;
    w2_t[3]  = 0;
    w3_t[0]  = 0;
    w3_t[1]  = 0;
    w3_t[2]  = 9 * 8;
    w3_t[3]  = 0;

    digest[0] = MD5M_A;
    digest[1] = MD5M_B;
    digest[2] = MD5M_C;
    digest[3] = MD5M_D;

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    // now the RC4 part

    u32x key[4];

    key[0] = digest[0];
    key[1] = digest[1];
    key[2] = digest[2];
    key[3] = digest[3];

    rc4_init_16 (rc4_key, key);

    u32x out[4];

    u8 j = rc4_next_16 (rc4_key, 0, 0, encryptedVerifier, out);

    w0_t[0] = out[0];
    w0_t[1] = out[1];
    w0_t[2] = out[2];
    w0_t[3] = out[3];
    w1_t[0] = 0x80;
    w1_t[1] = 0;
    w1_t[2] = 0;
    w1_t[3] = 0;
    w2_t[0] = 0;
    w2_t[1] = 0;
    w2_t[2] = 0;
    w2_t[3] = 0;
    w3_t[0] = 0;
    w3_t[1] = 0;
    w3_t[2] = 16 * 8;
    w3_t[3] = 0;

    digest[0] = MD5M_A;
    digest[1] = MD5M_B;
    digest[2] = MD5M_C;
    digest[3] = MD5M_D;

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    rc4_next_16 (rc4_key, 16, j, digest, out);

    const u32x r0 = out[0];
    const u32x r1 = out[1];
    const u32x r2 = out[2];
    const u32x r3 = out[3];

    #include VECT_COMPARE_M
  }
}

__device__ static void m09700s (RC4_KEY rc4_keys[64], u32x w0[4], u32x w1[4], u32x w2[4], u32x w3[4], const u32 pw_len, const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *bfs_buf, void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const oldoffice01_t *oldoffice01_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset)
{
  /**
   * modifier
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;
  const u32 lid = threadIdx.x;

  RC4_KEY *rc4_key = &rc4_keys[lid];

  /**
   * digest
   */

  const u32 search[4] =
  {
    digests_buf[digests_offset].digest_buf[DGST_R0],
    digests_buf[digests_offset].digest_buf[DGST_R1],
    digests_buf[digests_offset].digest_buf[DGST_R2],
    digests_buf[digests_offset].digest_buf[DGST_R3]
  };

  /**
   * salt
   */

  u32 salt_buf_t0[4];
  u32 salt_buf_t1[5];
  u32 salt_buf_t2[5];
  u32 salt_buf_t3[5];

  salt_buf_t0[0] = salt_bufs[salt_pos].salt_buf[0];
  salt_buf_t0[1] = salt_bufs[salt_pos].salt_buf[1];
  salt_buf_t0[2] = salt_bufs[salt_pos].salt_buf[2];
  salt_buf_t0[3] = salt_bufs[salt_pos].salt_buf[3];

  salt_buf_t1[0] =                        salt_buf_t0[0] <<  8;
  salt_buf_t1[1] = salt_buf_t0[0] >> 24 | salt_buf_t0[1] <<  8;
  salt_buf_t1[2] = salt_buf_t0[1] >> 24 | salt_buf_t0[2] <<  8;
  salt_buf_t1[3] = salt_buf_t0[2] >> 24 | salt_buf_t0[3] <<  8;
  salt_buf_t1[4] = salt_buf_t0[3] >> 24;

  salt_buf_t2[0] =                        salt_buf_t0[0] << 16;
  salt_buf_t2[1] = salt_buf_t0[0] >> 16 | salt_buf_t0[1] << 16;
  salt_buf_t2[2] = salt_buf_t0[1] >> 16 | salt_buf_t0[2] << 16;
  salt_buf_t2[3] = salt_buf_t0[2] >> 16 | salt_buf_t0[3] << 16;
  salt_buf_t2[4] = salt_buf_t0[3] >> 16;

  salt_buf_t3[0] =                        salt_buf_t0[0] << 24;
  salt_buf_t3[1] = salt_buf_t0[0] >>  8 | salt_buf_t0[1] << 24;
  salt_buf_t3[2] = salt_buf_t0[1] >>  8 | salt_buf_t0[2] << 24;
  salt_buf_t3[3] = salt_buf_t0[2] >>  8 | salt_buf_t0[3] << 24;
  salt_buf_t3[4] = salt_buf_t0[3] >>  8;

  const u32 salt_len = 16;

  /**
   * esalt
   */

  const u32 version = oldoffice01_bufs[salt_pos].version;

  u32 encryptedVerifier[4];

  encryptedVerifier[0] = oldoffice01_bufs[salt_pos].encryptedVerifier[0];
  encryptedVerifier[1] = oldoffice01_bufs[salt_pos].encryptedVerifier[1];
  encryptedVerifier[2] = oldoffice01_bufs[salt_pos].encryptedVerifier[2];
  encryptedVerifier[3] = oldoffice01_bufs[salt_pos].encryptedVerifier[3];

  /**
   * loop
   */

  u32x w0l = w0[0];

  for (u32 il_pos = 0; il_pos < bfs_cnt; il_pos++)
  {
    const u32 w0r = c_bfs[il_pos].i;

    w0[0] = w0l | w0r;

    u32x w0_t[4];
    u32x w1_t[4];
    u32x w2_t[4];
    u32x w3_t[4];

    w0_t[0] = w0[0];
    w0_t[1] = w0[1];
    w0_t[2] = w0[2];
    w0_t[3] = w0[3];
    w1_t[0] = w1[0];
    w1_t[1] = w1[1];
    w1_t[2] = w1[2];
    w1_t[3] = w1[3];
    w2_t[0] = w2[0];
    w2_t[1] = w2[1];
    w2_t[2] = w2[2];
    w2_t[3] = w2[3];
    w3_t[0] = w3[0];
    w3_t[1] = w3[1];
    w3_t[2] = pw_len * 8;
    w3_t[3] = 0;

    u32x digest_t0[4];
    u32x digest_t1[2]; // need only first 5 byte
    u32x digest_t2[2];
    u32x digest_t3[2];

    digest_t0[0] = MD5M_A;
    digest_t0[1] = MD5M_B;
    digest_t0[2] = MD5M_C;
    digest_t0[3] = MD5M_D;

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest_t0);

    // prepare 16 * 21 buffer stuff

    u32x digest[4];

    digest[0] = MD5M_A;
    digest[1] = MD5M_B;
    digest[2] = MD5M_C;
    digest[3] = MD5M_D;

    // offsets

    digest_t0[0] &= 0xffffffff;
    digest_t0[1] &= 0x000000ff;
    digest_t0[2] &= 0x00000000;
    digest_t0[3] &= 0x00000000;

    digest_t1[0] =                      digest_t0[0] <<  8;
    digest_t1[1] = digest_t0[0] >> 24 | digest_t0[1] <<  8;

    digest_t2[0] =                      digest_t0[0] << 16;
    digest_t2[1] = digest_t0[0] >> 16 | digest_t0[1] << 16;

    digest_t3[0] =                      digest_t0[0] << 24;
    digest_t3[1] = digest_t0[0] >>  8 | digest_t0[1] << 24;

    // generate the 16 * 21 buffer

    w0_t[0] = 0;
    w0_t[1] = 0;
    w0_t[2] = 0;
    w0_t[3] = 0;
    w1_t[0] = 0;
    w1_t[1] = 0;
    w1_t[2] = 0;
    w1_t[3] = 0;
    w2_t[0] = 0;
    w2_t[1] = 0;
    w2_t[2] = 0;
    w2_t[3] = 0;
    w3_t[0] = 0;
    w3_t[1] = 0;
    w3_t[2] = 0;
    w3_t[3] = 0;

    // 0..5
    w0_t[0]  = digest_t0[0];
    w0_t[1]  = digest_t0[1];

    // 5..21
    w0_t[1] |= salt_buf_t1[0];
    w0_t[2]  = salt_buf_t1[1];
    w0_t[3]  = salt_buf_t1[2];
    w1_t[0]  = salt_buf_t1[3];
    w1_t[1]  = salt_buf_t1[4];

    // 21..26
    w1_t[1] |= digest_t1[0];
    w1_t[2]  = digest_t1[1];

    // 26..42
    w1_t[2] |= salt_buf_t2[0];
    w1_t[3]  = salt_buf_t2[1];
    w2_t[0]  = salt_buf_t2[2];
    w2_t[1]  = salt_buf_t2[3];
    w2_t[2]  = salt_buf_t2[4];

    // 42..47
    w2_t[2] |= digest_t2[0];
    w2_t[3]  = digest_t2[1];

    // 47..63
    w2_t[3] |= salt_buf_t3[0];
    w3_t[0]  = salt_buf_t3[1];
    w3_t[1]  = salt_buf_t3[2];
    w3_t[2]  = salt_buf_t3[3];
    w3_t[3]  = salt_buf_t3[4];

    // 63..

    w3_t[3] |= digest_t3[0];

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    w0_t[0] = 0;
    w0_t[1] = 0;
    w0_t[2] = 0;
    w0_t[3] = 0;
    w1_t[0] = 0;
    w1_t[1] = 0;
    w1_t[2] = 0;
    w1_t[3] = 0;
    w2_t[0] = 0;
    w2_t[1] = 0;
    w2_t[2] = 0;
    w2_t[3] = 0;
    w3_t[0] = 0;
    w3_t[1] = 0;
    w3_t[2] = 0;
    w3_t[3] = 0;

    // 0..4
    w0_t[0]  = digest_t3[1];

    // 4..20
    w0_t[1]  = salt_buf_t0[0];
    w0_t[2]  = salt_buf_t0[1];
    w0_t[3]  = salt_buf_t0[2];
    w1_t[0]  = salt_buf_t0[3];

    // 20..25
    w1_t[1]  = digest_t0[0];
    w1_t[2]  = digest_t0[1];

    // 25..41
    w1_t[2] |= salt_buf_t1[0];
    w1_t[3]  = salt_buf_t1[1];
    w2_t[0]  = salt_buf_t1[2];
    w2_t[1]  = salt_buf_t1[3];
    w2_t[2]  = salt_buf_t1[4];

    // 41..46
    w2_t[2] |= digest_t1[0];
    w2_t[3]  = digest_t1[1];

    // 46..62
    w2_t[3] |= salt_buf_t2[0];
    w3_t[0]  = salt_buf_t2[1];
    w3_t[1]  = salt_buf_t2[2];
    w3_t[2]  = salt_buf_t2[3];
    w3_t[3]  = salt_buf_t2[4];

    // 62..
    w3_t[3] |= digest_t2[0];

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    w0_t[0] = 0;
    w0_t[1] = 0;
    w0_t[2] = 0;
    w0_t[3] = 0;
    w1_t[0] = 0;
    w1_t[1] = 0;
    w1_t[2] = 0;
    w1_t[3] = 0;
    w2_t[0] = 0;
    w2_t[1] = 0;
    w2_t[2] = 0;
    w2_t[3] = 0;
    w3_t[0] = 0;
    w3_t[1] = 0;
    w3_t[2] = 0;
    w3_t[3] = 0;

    // 0..3
    w0_t[0]  = digest_t2[1];

    // 3..19
    w0_t[0] |= salt_buf_t3[0];
    w0_t[1]  = salt_buf_t3[1];
    w0_t[2]  = salt_buf_t3[2];
    w0_t[3]  = salt_buf_t3[3];
    w1_t[0]  = salt_buf_t3[4];

    // 19..24
    w1_t[0] |= digest_t3[0];
    w1_t[1]  = digest_t3[1];

    // 24..40
    w1_t[2]  = salt_buf_t0[0];
    w1_t[3]  = salt_buf_t0[1];
    w2_t[0]  = salt_buf_t0[2];
    w2_t[1]  = salt_buf_t0[3];

    // 40..45
    w2_t[2]  = digest_t0[0];
    w2_t[3]  = digest_t0[1];

    // 45..61
    w2_t[3] |= salt_buf_t1[0];
    w3_t[0]  = salt_buf_t1[1];
    w3_t[1]  = salt_buf_t1[2];
    w3_t[2]  = salt_buf_t1[3];
    w3_t[3]  = salt_buf_t1[4];

    // 61..
    w3_t[3] |= digest_t1[0];

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    w0_t[0] = 0;
    w0_t[1] = 0;
    w0_t[2] = 0;
    w0_t[3] = 0;
    w1_t[0] = 0;
    w1_t[1] = 0;
    w1_t[2] = 0;
    w1_t[3] = 0;
    w2_t[0] = 0;
    w2_t[1] = 0;
    w2_t[2] = 0;
    w2_t[3] = 0;
    w3_t[0] = 0;
    w3_t[1] = 0;
    w3_t[2] = 0;
    w3_t[3] = 0;

    // 0..2
    w0_t[0]  = digest_t1[1];

    // 2..18
    w0_t[0] |= salt_buf_t2[0];
    w0_t[1]  = salt_buf_t2[1];
    w0_t[2]  = salt_buf_t2[2];
    w0_t[3]  = salt_buf_t2[3];
    w1_t[0]  = salt_buf_t2[4];

    // 18..23
    w1_t[0] |= digest_t2[0];
    w1_t[1]  = digest_t2[1];

    // 23..39
    w1_t[1] |= salt_buf_t3[0];
    w1_t[2]  = salt_buf_t3[1];
    w1_t[3]  = salt_buf_t3[2];
    w2_t[0]  = salt_buf_t3[3];
    w2_t[1]  = salt_buf_t3[4];

    // 39..44
    w2_t[1] |= digest_t3[0];
    w2_t[2]  = digest_t3[1];

    // 44..60
    w2_t[3]  = salt_buf_t0[0];
    w3_t[0]  = salt_buf_t0[1];
    w3_t[1]  = salt_buf_t0[2];
    w3_t[2]  = salt_buf_t0[3];

    // 60..
    w3_t[3]  = digest_t0[0];

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    w0_t[0] = 0;
    w0_t[1] = 0;
    w0_t[2] = 0;
    w0_t[3] = 0;
    w1_t[0] = 0;
    w1_t[1] = 0;
    w1_t[2] = 0;
    w1_t[3] = 0;
    w2_t[0] = 0;
    w2_t[1] = 0;
    w2_t[2] = 0;
    w2_t[3] = 0;
    w3_t[0] = 0;
    w3_t[1] = 0;
    w3_t[2] = 0;
    w3_t[3] = 0;

    // 0..1
    w0_t[0]  = digest_t0[1];

    // 1..17
    w0_t[0] |= salt_buf_t1[0];
    w0_t[1]  = salt_buf_t1[1];
    w0_t[2]  = salt_buf_t1[2];
    w0_t[3]  = salt_buf_t1[3];
    w1_t[0]  = salt_buf_t1[4];

    // 17..22
    w1_t[0] |= digest_t1[0];
    w1_t[1]  = digest_t1[1];

    // 22..38
    w1_t[1] |= salt_buf_t2[0];
    w1_t[2]  = salt_buf_t2[1];
    w1_t[3]  = salt_buf_t2[2];
    w2_t[0]  = salt_buf_t2[3];
    w2_t[1]  = salt_buf_t2[4];

    // 38..43
    w2_t[1] |= digest_t2[0];
    w2_t[2]  = digest_t2[1];

    // 43..59
    w2_t[2] |= salt_buf_t3[0];
    w2_t[3]  = salt_buf_t3[1];
    w3_t[0]  = salt_buf_t3[2];
    w3_t[1]  = salt_buf_t3[3];
    w3_t[2]  = salt_buf_t3[4];

    // 59..
    w3_t[2] |= digest_t3[0];
    w3_t[3]  = digest_t3[1];

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    w0_t[0]  = salt_buf_t0[0];
    w0_t[1]  = salt_buf_t0[1];
    w0_t[2]  = salt_buf_t0[2];
    w0_t[3]  = salt_buf_t0[3];
    w1_t[0]  = 0x80;
    w1_t[1]  = 0;
    w1_t[2]  = 0;
    w1_t[3]  = 0;
    w2_t[0]  = 0;
    w2_t[1]  = 0;
    w2_t[2]  = 0;
    w2_t[3]  = 0;
    w3_t[0]  = 0;
    w3_t[1]  = 0;
    w3_t[2]  = 21 * 16 * 8;
    w3_t[3]  = 0;

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    // now the 40 bit input for the MD5 which then will generate the RC4 key, so it's precomputable!

    w0_t[0]  = digest[0];
    w0_t[1]  = digest[1] & 0xff;
    w0_t[2]  = 0x8000;
    w0_t[3]  = 0;
    w1_t[0]  = 0;
    w1_t[1]  = 0;
    w1_t[2]  = 0;
    w1_t[3]  = 0;
    w2_t[0]  = 0;
    w2_t[1]  = 0;
    w2_t[2]  = 0;
    w2_t[3]  = 0;
    w3_t[0]  = 0;
    w3_t[1]  = 0;
    w3_t[2]  = 9 * 8;
    w3_t[3]  = 0;

    digest[0] = MD5M_A;
    digest[1] = MD5M_B;
    digest[2] = MD5M_C;
    digest[3] = MD5M_D;

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    // now the RC4 part

    u32x key[4];

    key[0] = digest[0];
    key[1] = digest[1];
    key[2] = digest[2];
    key[3] = digest[3];

    rc4_init_16 (rc4_key, key);

    u32x out[4];

    u8 j = rc4_next_16 (rc4_key, 0, 0, encryptedVerifier, out);

    w0_t[0] = out[0];
    w0_t[1] = out[1];
    w0_t[2] = out[2];
    w0_t[3] = out[3];
    w1_t[0] = 0x80;
    w1_t[1] = 0;
    w1_t[2] = 0;
    w1_t[3] = 0;
    w2_t[0] = 0;
    w2_t[1] = 0;
    w2_t[2] = 0;
    w2_t[3] = 0;
    w3_t[0] = 0;
    w3_t[1] = 0;
    w3_t[2] = 16 * 8;
    w3_t[3] = 0;

    digest[0] = MD5M_A;
    digest[1] = MD5M_B;
    digest[2] = MD5M_C;
    digest[3] = MD5M_D;

    md5_transform (w0_t, w1_t, w2_t, w3_t, digest);

    rc4_next_16 (rc4_key, 16, j, digest, out);

    const u32x r0 = out[0];
    const u32x r1 = out[1];
    const u32x r2 = out[2];
    const u32x r3 = out[3];

    #include VECT_COMPARE_S
  }
}

extern "C" __global__ void __launch_bounds__ (64, 1) m09700_m04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *bfs_buf, void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const oldoffice01_t *oldoffice01_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x w0[4];

  w0[0] = pws[gid].i[ 0];
  w0[1] = pws[gid].i[ 1];
  w0[2] = pws[gid].i[ 2];
  w0[3] = pws[gid].i[ 3];

  u32x w1[4];

  w1[0] = 0;
  w1[1] = 0;
  w1[2] = 0;
  w1[3] = 0;

  u32x w2[4];

  w2[0] = 0;
  w2[1] = 0;
  w2[2] = 0;
  w2[3] = 0;

  u32x w3[4];

  w3[0] = 0;
  w3[1] = 0;
  w3[2] = 0;
  w3[3] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  __shared__ RC4_KEY rc4_keys[64];

  m09700m (rc4_keys, w0, w1, w2, w3, pw_len, pws, rules_buf, combs_buf, bfs_buf, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, oldoffice01_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (64, 1) m09700_m08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *bfs_buf, void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const oldoffice01_t *oldoffice01_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x w0[4];

  w0[0] = pws[gid].i[ 0];
  w0[1] = pws[gid].i[ 1];
  w0[2] = pws[gid].i[ 2];
  w0[3] = pws[gid].i[ 3];

  u32x w1[4];

  w1[0] = pws[gid].i[ 4];
  w1[1] = pws[gid].i[ 5];
  w1[2] = pws[gid].i[ 6];
  w1[3] = pws[gid].i[ 7];

  u32x w2[4];

  w2[0] = 0;
  w2[1] = 0;
  w2[2] = 0;
  w2[3] = 0;

  u32x w3[4];

  w3[0] = 0;
  w3[1] = 0;
  w3[2] = 0;
  w3[3] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  __shared__ RC4_KEY rc4_keys[64];

  m09700m (rc4_keys, w0, w1, w2, w3, pw_len, pws, rules_buf, combs_buf, bfs_buf, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, oldoffice01_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (64, 1) m09700_m16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *bfs_buf, void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const oldoffice01_t *oldoffice01_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x w0[4];

  w0[0] = pws[gid].i[ 0];
  w0[1] = pws[gid].i[ 1];
  w0[2] = pws[gid].i[ 2];
  w0[3] = pws[gid].i[ 3];

  u32x w1[4];

  w1[0] = pws[gid].i[ 4];
  w1[1] = pws[gid].i[ 5];
  w1[2] = pws[gid].i[ 6];
  w1[3] = pws[gid].i[ 7];

  u32x w2[4];

  w2[0] = pws[gid].i[ 8];
  w2[1] = pws[gid].i[ 9];
  w2[2] = pws[gid].i[10];
  w2[3] = pws[gid].i[11];

  u32x w3[4];

  w3[0] = pws[gid].i[12];
  w3[1] = pws[gid].i[13];
  w3[2] = 0;
  w3[3] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  __shared__ RC4_KEY rc4_keys[64];

  m09700m (rc4_keys, w0, w1, w2, w3, pw_len, pws, rules_buf, combs_buf, bfs_buf, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, oldoffice01_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (64, 1) m09700_s04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *bfs_buf, void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const oldoffice01_t *oldoffice01_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x w0[4];

  w0[0] = pws[gid].i[ 0];
  w0[1] = pws[gid].i[ 1];
  w0[2] = pws[gid].i[ 2];
  w0[3] = pws[gid].i[ 3];

  u32x w1[4];

  w1[0] = 0;
  w1[1] = 0;
  w1[2] = 0;
  w1[3] = 0;

  u32x w2[4];

  w2[0] = 0;
  w2[1] = 0;
  w2[2] = 0;
  w2[3] = 0;

  u32x w3[4];

  w3[0] = 0;
  w3[1] = 0;
  w3[2] = 0;
  w3[3] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  __shared__ RC4_KEY rc4_keys[64];

  m09700s (rc4_keys, w0, w1, w2, w3, pw_len, pws, rules_buf, combs_buf, bfs_buf, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, oldoffice01_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (64, 1) m09700_s08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *bfs_buf, void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const oldoffice01_t *oldoffice01_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x w0[4];

  w0[0] = pws[gid].i[ 0];
  w0[1] = pws[gid].i[ 1];
  w0[2] = pws[gid].i[ 2];
  w0[3] = pws[gid].i[ 3];

  u32x w1[4];

  w1[0] = pws[gid].i[ 4];
  w1[1] = pws[gid].i[ 5];
  w1[2] = pws[gid].i[ 6];
  w1[3] = pws[gid].i[ 7];

  u32x w2[4];

  w2[0] = 0;
  w2[1] = 0;
  w2[2] = 0;
  w2[3] = 0;

  u32x w3[4];

  w3[0] = 0;
  w3[1] = 0;
  w3[2] = 0;
  w3[3] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  __shared__ RC4_KEY rc4_keys[64];

  m09700s (rc4_keys, w0, w1, w2, w3, pw_len, pws, rules_buf, combs_buf, bfs_buf, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, oldoffice01_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (64, 1) m09700_s16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *bfs_buf, void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const oldoffice01_t *oldoffice01_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x w0[4];

  w0[0] = pws[gid].i[ 0];
  w0[1] = pws[gid].i[ 1];
  w0[2] = pws[gid].i[ 2];
  w0[3] = pws[gid].i[ 3];

  u32x w1[4];

  w1[0] = pws[gid].i[ 4];
  w1[1] = pws[gid].i[ 5];
  w1[2] = pws[gid].i[ 6];
  w1[3] = pws[gid].i[ 7];

  u32x w2[4];

  w2[0] = pws[gid].i[ 8];
  w2[1] = pws[gid].i[ 9];
  w2[2] = pws[gid].i[10];
  w2[3] = pws[gid].i[11];

  u32x w3[4];

  w3[0] = pws[gid].i[12];
  w3[1] = pws[gid].i[13];
  w3[2] = 0;
  w3[3] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  __shared__ RC4_KEY rc4_keys[64];

  m09700s (rc4_keys, w0, w1, w2, w3, pw_len, pws, rules_buf, combs_buf, bfs_buf, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, oldoffice01_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}
