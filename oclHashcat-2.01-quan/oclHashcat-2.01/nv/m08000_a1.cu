#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _SHA256_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 3
#define DGST_R1 7
#define DGST_R2 2
#define DGST_R3 6

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_S "check_single_vect1_comp4.c"
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_S "check_single_vect2_comp4.c"
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

__device__ static void sha256_transform (u32x digest[8], const u32x w[16])
{
  u32x w0_t = w[ 0];
  u32x w1_t = w[ 1];
  u32x w2_t = w[ 2];
  u32x w3_t = w[ 3];
  u32x w4_t = w[ 4];
  u32x w5_t = w[ 5];
  u32x w6_t = w[ 6];
  u32x w7_t = w[ 7];
  u32x w8_t = w[ 8];
  u32x w9_t = w[ 9];
  u32x wa_t = w[10];
  u32x wb_t = w[11];
  u32x wc_t = w[12];
  u32x wd_t = w[13];
  u32x we_t = w[14];
  u32x wf_t = w[15];

  u32x a = digest[0];
  u32x b = digest[1];
  u32x c = digest[2];
  u32x d = digest[3];
  u32x e = digest[4];
  u32x f = digest[5];
  u32x g = digest[6];
  u32x h = digest[7];

  SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C00);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C01);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C02);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C03);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C04);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C05);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C06);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C07);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C08);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C09);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C0a);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C0b);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C0c);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C0d);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C0e);
  SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C0f);
  w0_t = SHA256_S1(we_t) + w9_t + SHA256_S0(w1_t) + w0_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C10);
  w1_t = SHA256_S1(wf_t) + wa_t + SHA256_S0(w2_t) + w1_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C11);
  w2_t = SHA256_S1(w0_t) + wb_t + SHA256_S0(w3_t) + w2_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C12);
  w3_t = SHA256_S1(w1_t) + wc_t + SHA256_S0(w4_t) + w3_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C13);
  w4_t = SHA256_S1(w2_t) + wd_t + SHA256_S0(w5_t) + w4_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C14);
  w5_t = SHA256_S1(w3_t) + we_t + SHA256_S0(w6_t) + w5_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C15);
  w6_t = SHA256_S1(w4_t) + wf_t + SHA256_S0(w7_t) + w6_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C16);
  w7_t = SHA256_S1(w5_t) + w0_t + SHA256_S0(w8_t) + w7_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C17);
  w8_t = SHA256_S1(w6_t) + w1_t + SHA256_S0(w9_t) + w8_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C18);
  w9_t = SHA256_S1(w7_t) + w2_t + SHA256_S0(wa_t) + w9_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C19);
  wa_t = SHA256_S1(w8_t) + w3_t + SHA256_S0(wb_t) + wa_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C1a);
  wb_t = SHA256_S1(w9_t) + w4_t + SHA256_S0(wc_t) + wb_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C1b);
  wc_t = SHA256_S1(wa_t) + w5_t + SHA256_S0(wd_t) + wc_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C1c);
  wd_t = SHA256_S1(wb_t) + w6_t + SHA256_S0(we_t) + wd_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C1d);
  we_t = SHA256_S1(wc_t) + w7_t + SHA256_S0(wf_t) + we_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C1e);
  wf_t = SHA256_S1(wd_t) + w8_t + SHA256_S0(w0_t) + wf_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C1f);
  w0_t = SHA256_S1(we_t) + w9_t + SHA256_S0(w1_t) + w0_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C20);
  w1_t = SHA256_S1(wf_t) + wa_t + SHA256_S0(w2_t) + w1_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C21);
  w2_t = SHA256_S1(w0_t) + wb_t + SHA256_S0(w3_t) + w2_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C22);
  w3_t = SHA256_S1(w1_t) + wc_t + SHA256_S0(w4_t) + w3_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C23);
  w4_t = SHA256_S1(w2_t) + wd_t + SHA256_S0(w5_t) + w4_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C24);
  w5_t = SHA256_S1(w3_t) + we_t + SHA256_S0(w6_t) + w5_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C25);
  w6_t = SHA256_S1(w4_t) + wf_t + SHA256_S0(w7_t) + w6_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C26);
  w7_t = SHA256_S1(w5_t) + w0_t + SHA256_S0(w8_t) + w7_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C27);
  w8_t = SHA256_S1(w6_t) + w1_t + SHA256_S0(w9_t) + w8_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C28);
  w9_t = SHA256_S1(w7_t) + w2_t + SHA256_S0(wa_t) + w9_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C29);
  wa_t = SHA256_S1(w8_t) + w3_t + SHA256_S0(wb_t) + wa_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C2a);
  wb_t = SHA256_S1(w9_t) + w4_t + SHA256_S0(wc_t) + wb_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C2b);
  wc_t = SHA256_S1(wa_t) + w5_t + SHA256_S0(wd_t) + wc_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C2c);
  wd_t = SHA256_S1(wb_t) + w6_t + SHA256_S0(we_t) + wd_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C2d);
  we_t = SHA256_S1(wc_t) + w7_t + SHA256_S0(wf_t) + we_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C2e);
  wf_t = SHA256_S1(wd_t) + w8_t + SHA256_S0(w0_t) + wf_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C2f);
  w0_t = SHA256_S1(we_t) + w9_t + SHA256_S0(w1_t) + w0_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, SHA256C30);
  w1_t = SHA256_S1(wf_t) + wa_t + SHA256_S0(w2_t) + w1_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, SHA256C31);
  w2_t = SHA256_S1(w0_t) + wb_t + SHA256_S0(w3_t) + w2_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, SHA256C32);
  w3_t = SHA256_S1(w1_t) + wc_t + SHA256_S0(w4_t) + w3_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, SHA256C33);
  w4_t = SHA256_S1(w2_t) + wd_t + SHA256_S0(w5_t) + w4_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, SHA256C34);
  w5_t = SHA256_S1(w3_t) + we_t + SHA256_S0(w6_t) + w5_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, SHA256C35);
  w6_t = SHA256_S1(w4_t) + wf_t + SHA256_S0(w7_t) + w6_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, SHA256C36);
  w7_t = SHA256_S1(w5_t) + w0_t + SHA256_S0(w8_t) + w7_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, SHA256C37);
  w8_t = SHA256_S1(w6_t) + w1_t + SHA256_S0(w9_t) + w8_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, SHA256C38);
  w9_t = SHA256_S1(w7_t) + w2_t + SHA256_S0(wa_t) + w9_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, SHA256C39);
  wa_t = SHA256_S1(w8_t) + w3_t + SHA256_S0(wb_t) + wa_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, SHA256C3a);
  wb_t = SHA256_S1(w9_t) + w4_t + SHA256_S0(wc_t) + wb_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, SHA256C3b);
  wc_t = SHA256_S1(wa_t) + w5_t + SHA256_S0(wd_t) + wc_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, SHA256C3c);
  wd_t = SHA256_S1(wb_t) + w6_t + SHA256_S0(we_t) + wd_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, SHA256C3d);
  we_t = SHA256_S1(wc_t) + w7_t + SHA256_S0(wf_t) + we_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, SHA256C3e);
  wf_t = SHA256_S1(wd_t) + w8_t + SHA256_S0(w0_t) + wf_t; SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, SHA256C3f);

  digest[0] += a;
  digest[1] += b;
  digest[2] += c;
  digest[3] += d;
  digest[4] += e;
  digest[5] += f;
  digest[6] += g;
  digest[7] += h;
}

__device__ __constant__ comb_t c_combs[1024];

extern "C" __global__ void __launch_bounds__ (256, 1) m08000_m04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 combs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x wordl0[4];

  wordl0[0] = pws[gid].i[ 0];
  wordl0[1] = pws[gid].i[ 1];
  wordl0[2] = pws[gid].i[ 2];
  wordl0[3] = pws[gid].i[ 3];

  u32x wordl1[4];

  wordl1[0] = pws[gid].i[ 4];
  wordl1[1] = pws[gid].i[ 5];
  wordl1[2] = pws[gid].i[ 6];
  wordl1[3] = pws[gid].i[ 7];

  u32x wordl2[4];

  wordl2[0] = 0;
  wordl2[1] = 0;
  wordl2[2] = 0;
  wordl2[3] = 0;

  u32x wordl3[4];

  wordl3[0] = 0;
  wordl3[1] = 0;
  wordl3[2] = 0;
  wordl3[3] = 0;

  const u32 pw_l_len = pws[gid].pw_len;

  if (combs_mode == COMBINATOR_MODE_BASE_RIGHT)
  {
    switch_buffer_by_offset (wordl0, wordl1, wordl2, wordl3, c_combs[0].pw_len);
  }

  /**
   * salt
   */

  const u32 salt_buf0 = swap_workaround (salt_bufs[salt_pos].salt_buf[ 0]);
  const u32 salt_buf1 = swap_workaround (salt_bufs[salt_pos].salt_buf[ 1]);
  const u32 salt_buf2 = swap_workaround (salt_bufs[salt_pos].salt_buf[ 2]); // 0x80

  /**
   * loop
   */

  for (u32 il_pos = 0; il_pos < combs_cnt; il_pos++)
  {
    u32 wordr0[4];
    u32 wordr1[4];
    u32 wordr2[4];
    u32 wordr3[4];

    wordr0[0] = c_combs[il_pos].i[0];
    wordr0[1] = c_combs[il_pos].i[1];
    wordr0[2] = c_combs[il_pos].i[2];
    wordr0[3] = c_combs[il_pos].i[3];
    wordr1[0] = c_combs[il_pos].i[4];
    wordr1[1] = c_combs[il_pos].i[5];
    wordr1[2] = c_combs[il_pos].i[6];
    wordr1[3] = c_combs[il_pos].i[7];
    wordr2[0] = 0;
    wordr2[1] = 0;
    wordr2[2] = 0;
    wordr2[3] = 0;
    wordr3[0] = 0;
    wordr3[1] = 0;
    wordr3[2] = 0;
    wordr3[3] = 0;

    if (combs_mode == COMBINATOR_MODE_BASE_LEFT)
    {
      switch_buffer_by_offset (wordr0, wordr1, wordr2, wordr3, pw_l_len);
    }

    u32x w0[4];
    u32x w1[4];
    u32x w2[4];
    u32x w3[4];

    w0[0] = wordl0[0] | wordr0[0];
    w0[1] = wordl0[1] | wordr0[1];
    w0[2] = wordl0[2] | wordr0[2];
    w0[3] = wordl0[3] | wordr0[3];
    w1[0] = wordl1[0] | wordr1[0];
    w1[1] = wordl1[1] | wordr1[1];
    w1[2] = wordl1[2] | wordr1[2];
    w1[3] = wordl1[3] | wordr1[3];
    w2[0] = wordl2[0] | wordr2[0];
    w2[1] = wordl2[1] | wordr2[1];
    w2[2] = wordl2[2] | wordr2[2];
    w2[3] = wordl2[3] | wordr2[3];
    w3[0] = wordl3[0] | wordr3[0];
    w3[1] = wordl3[1] | wordr3[1];
    w3[2] = wordl3[2] | wordr3[2];
    w3[3] = wordl3[3] | wordr3[3];

    u32x w0_t[4];
    u32x w1_t[4];
    u32x w2_t[4];
    u32x w3_t[4];

    make_unicode (w0, w0_t, w1_t);

    make_unicode (w1, w2_t, w3_t);

    u32x w_t[16];

    w_t[ 0] = swap_workaround (w0_t[0]);
    w_t[ 1] = swap_workaround (w0_t[1]);
    w_t[ 2] = swap_workaround (w0_t[2]);
    w_t[ 3] = swap_workaround (w0_t[3]);
    w_t[ 4] = swap_workaround (w1_t[0]);
    w_t[ 5] = swap_workaround (w1_t[1]);
    w_t[ 6] = swap_workaround (w1_t[2]);
    w_t[ 7] = swap_workaround (w1_t[3]);
    w_t[ 8] = swap_workaround (w2_t[0]);
    w_t[ 9] = swap_workaround (w2_t[1]);
    w_t[10] = swap_workaround (w2_t[2]);
    w_t[11] = swap_workaround (w2_t[3]);
    w_t[12] = swap_workaround (w3_t[0]);
    w_t[13] = swap_workaround (w3_t[1]);
    w_t[14] = swap_workaround (w3_t[2]);
    w_t[15] = swap_workaround (w3_t[3]);

    w_t[ 0] = w_t[ 0] >> 8;
    w_t[ 1] = w_t[ 1] >> 8;
    w_t[ 2] = w_t[ 2] >> 8;
    w_t[ 3] = w_t[ 3] >> 8;
    w_t[ 4] = w_t[ 4] >> 8;
    w_t[ 5] = w_t[ 5] >> 8;
    w_t[ 6] = w_t[ 6] >> 8;
    w_t[ 7] = w_t[ 7] >> 8;
    w_t[ 8] = w_t[ 8] >> 8;
    w_t[ 9] = w_t[ 9] >> 8;
    w_t[10] = w_t[10] >> 8;
    w_t[11] = w_t[11] >> 8;
    w_t[12] = w_t[12] >> 8;
    w_t[13] = w_t[13] >> 8;
    w_t[14] = w_t[14] >> 8;
    w_t[15] = w_t[15] >> 8;

    u32x digest[8];

    digest[0] = SHA256M_A;
    digest[1] = SHA256M_B;
    digest[2] = SHA256M_C;
    digest[3] = SHA256M_D;
    digest[4] = SHA256M_E;
    digest[5] = SHA256M_F;
    digest[6] = SHA256M_G;
    digest[7] = SHA256M_H;

    sha256_transform (digest, w_t); //   0 - 64

    w_t[ 0] = 0;
    w_t[ 1] = 0;
    w_t[ 2] = 0;
    w_t[ 3] = 0;
    w_t[ 4] = 0;
    w_t[ 5] = 0;
    w_t[ 6] = 0;
    w_t[ 7] = 0;
    w_t[ 8] = 0;
    w_t[ 9] = 0;
    w_t[10] = 0;
    w_t[11] = 0;
    w_t[12] = 0;
    w_t[13] = 0;
    w_t[14] = 0;
    w_t[15] = 0;

    sha256_transform (digest, w_t); //  64 - 128
    sha256_transform (digest, w_t); // 128 - 192
    sha256_transform (digest, w_t); // 192 - 256
    sha256_transform (digest, w_t); // 256 - 320
    sha256_transform (digest, w_t); // 320 - 384
    sha256_transform (digest, w_t); // 384 - 448

    w_t[15] =               0 | salt_buf0 >> 16;

    sha256_transform (digest, w_t); // 448 - 512

    w_t[ 0] = salt_buf0 << 16 | salt_buf1 >> 16;
    w_t[ 1] = salt_buf1 << 16 | salt_buf2 >> 16;
    w_t[ 2] = salt_buf2 << 16 | 0;
    w_t[15] = (510 + 8) * 8;

    sha256_transform (digest, w_t); // 512 - 576

    const u32x r0 = digest[3];
    const u32x r1 = digest[7];
    const u32x r2 = digest[2];
    const u32x r3 = digest[6];

    #include VECT_COMPARE_M
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m08000_m08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m08000_m16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m08000_s04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 combs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x wordl0[4];

  wordl0[0] = pws[gid].i[ 0];
  wordl0[1] = pws[gid].i[ 1];
  wordl0[2] = pws[gid].i[ 2];
  wordl0[3] = pws[gid].i[ 3];

  u32x wordl1[4];

  wordl1[0] = pws[gid].i[ 4];
  wordl1[1] = pws[gid].i[ 5];
  wordl1[2] = pws[gid].i[ 6];
  wordl1[3] = pws[gid].i[ 7];

  u32x wordl2[4];

  wordl2[0] = 0;
  wordl2[1] = 0;
  wordl2[2] = 0;
  wordl2[3] = 0;

  u32x wordl3[4];

  wordl3[0] = 0;
  wordl3[1] = 0;
  wordl3[2] = 0;
  wordl3[3] = 0;

  const u32 pw_l_len = pws[gid].pw_len;

  if (combs_mode == COMBINATOR_MODE_BASE_RIGHT)
  {
    switch_buffer_by_offset (wordl0, wordl1, wordl2, wordl3, c_combs[0].pw_len);
  }

  /**
   * salt
   */

  const u32 salt_buf0 = swap_workaround (salt_bufs[salt_pos].salt_buf[ 0]);
  const u32 salt_buf1 = swap_workaround (salt_bufs[salt_pos].salt_buf[ 1]);
  const u32 salt_buf2 = swap_workaround (salt_bufs[salt_pos].salt_buf[ 2]); // 0x80

  /**
   * digest
   */

  const u32 search[4] =
  {
    digests_buf[digests_offset].digest_buf[DGST_R0],
    digests_buf[digests_offset].digest_buf[DGST_R1],
    digests_buf[digests_offset].digest_buf[DGST_R2],
    digests_buf[digests_offset].digest_buf[DGST_R3]
  };

  /**
   * loop
   */

  for (u32 il_pos = 0; il_pos < combs_cnt; il_pos++)
  {
    u32 wordr0[4];
    u32 wordr1[4];
    u32 wordr2[4];
    u32 wordr3[4];

    wordr0[0] = c_combs[il_pos].i[0];
    wordr0[1] = c_combs[il_pos].i[1];
    wordr0[2] = c_combs[il_pos].i[2];
    wordr0[3] = c_combs[il_pos].i[3];
    wordr1[0] = c_combs[il_pos].i[4];
    wordr1[1] = c_combs[il_pos].i[5];
    wordr1[2] = c_combs[il_pos].i[6];
    wordr1[3] = c_combs[il_pos].i[7];
    wordr2[0] = 0;
    wordr2[1] = 0;
    wordr2[2] = 0;
    wordr2[3] = 0;
    wordr3[0] = 0;
    wordr3[1] = 0;
    wordr3[2] = 0;
    wordr3[3] = 0;

    if (combs_mode == COMBINATOR_MODE_BASE_LEFT)
    {
      switch_buffer_by_offset (wordr0, wordr1, wordr2, wordr3, pw_l_len);
    }

    u32x w0[4];
    u32x w1[4];
    u32x w2[4];
    u32x w3[4];

    w0[0] = wordl0[0] | wordr0[0];
    w0[1] = wordl0[1] | wordr0[1];
    w0[2] = wordl0[2] | wordr0[2];
    w0[3] = wordl0[3] | wordr0[3];
    w1[0] = wordl1[0] | wordr1[0];
    w1[1] = wordl1[1] | wordr1[1];
    w1[2] = wordl1[2] | wordr1[2];
    w1[3] = wordl1[3] | wordr1[3];
    w2[0] = wordl2[0] | wordr2[0];
    w2[1] = wordl2[1] | wordr2[1];
    w2[2] = wordl2[2] | wordr2[2];
    w2[3] = wordl2[3] | wordr2[3];
    w3[0] = wordl3[0] | wordr3[0];
    w3[1] = wordl3[1] | wordr3[1];
    w3[2] = wordl3[2] | wordr3[2];
    w3[3] = wordl3[3] | wordr3[3];

    u32x w0_t[4];
    u32x w1_t[4];
    u32x w2_t[4];
    u32x w3_t[4];

    make_unicode (w0, w0_t, w1_t);

    make_unicode (w1, w2_t, w3_t);

    u32x w_t[16];

    w_t[ 0] = swap_workaround (w0_t[0]);
    w_t[ 1] = swap_workaround (w0_t[1]);
    w_t[ 2] = swap_workaround (w0_t[2]);
    w_t[ 3] = swap_workaround (w0_t[3]);
    w_t[ 4] = swap_workaround (w1_t[0]);
    w_t[ 5] = swap_workaround (w1_t[1]);
    w_t[ 6] = swap_workaround (w1_t[2]);
    w_t[ 7] = swap_workaround (w1_t[3]);
    w_t[ 8] = swap_workaround (w2_t[0]);
    w_t[ 9] = swap_workaround (w2_t[1]);
    w_t[10] = swap_workaround (w2_t[2]);
    w_t[11] = swap_workaround (w2_t[3]);
    w_t[12] = swap_workaround (w3_t[0]);
    w_t[13] = swap_workaround (w3_t[1]);
    w_t[14] = swap_workaround (w3_t[2]);
    w_t[15] = swap_workaround (w3_t[3]);

    w_t[ 0] = w_t[ 0] >> 8;
    w_t[ 1] = w_t[ 1] >> 8;
    w_t[ 2] = w_t[ 2] >> 8;
    w_t[ 3] = w_t[ 3] >> 8;
    w_t[ 4] = w_t[ 4] >> 8;
    w_t[ 5] = w_t[ 5] >> 8;
    w_t[ 6] = w_t[ 6] >> 8;
    w_t[ 7] = w_t[ 7] >> 8;
    w_t[ 8] = w_t[ 8] >> 8;
    w_t[ 9] = w_t[ 9] >> 8;
    w_t[10] = w_t[10] >> 8;
    w_t[11] = w_t[11] >> 8;
    w_t[12] = w_t[12] >> 8;
    w_t[13] = w_t[13] >> 8;
    w_t[14] = w_t[14] >> 8;
    w_t[15] = w_t[15] >> 8;

    u32x digest[8];

    digest[0] = SHA256M_A;
    digest[1] = SHA256M_B;
    digest[2] = SHA256M_C;
    digest[3] = SHA256M_D;
    digest[4] = SHA256M_E;
    digest[5] = SHA256M_F;
    digest[6] = SHA256M_G;
    digest[7] = SHA256M_H;

    sha256_transform (digest, w_t); //   0 - 64

    w_t[ 0] = 0;
    w_t[ 1] = 0;
    w_t[ 2] = 0;
    w_t[ 3] = 0;
    w_t[ 4] = 0;
    w_t[ 5] = 0;
    w_t[ 6] = 0;
    w_t[ 7] = 0;
    w_t[ 8] = 0;
    w_t[ 9] = 0;
    w_t[10] = 0;
    w_t[11] = 0;
    w_t[12] = 0;
    w_t[13] = 0;
    w_t[14] = 0;
    w_t[15] = 0;

    sha256_transform (digest, w_t); //  64 - 128
    sha256_transform (digest, w_t); // 128 - 192
    sha256_transform (digest, w_t); // 192 - 256
    sha256_transform (digest, w_t); // 256 - 320
    sha256_transform (digest, w_t); // 320 - 384
    sha256_transform (digest, w_t); // 384 - 448

    w_t[15] =               0 | salt_buf0 >> 16;

    sha256_transform (digest, w_t); // 448 - 512

    w_t[ 0] = salt_buf0 << 16 | salt_buf1 >> 16;
    w_t[ 1] = salt_buf1 << 16 | salt_buf2 >> 16;
    w_t[ 2] = salt_buf2 << 16 | 0;
    w_t[15] = (510 + 8) * 8;

    sha256_transform (digest, w_t); // 512 - 576

    const u32x r0 = digest[3];
    const u32x r1 = digest[7];
    const u32x r2 = digest[2];
    const u32x r3 = digest[6];

    #include VECT_COMPARE_S
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m08000_s08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m08000_s16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}
