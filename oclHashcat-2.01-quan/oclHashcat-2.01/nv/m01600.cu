#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _MD5_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 0
#define DGST_R1 1
#define DGST_R2 2
#define DGST_R3 3

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

#define md5apr1_magic0 0x72706124
#define md5apr1_magic1 0x00002431

__device__ static void md5_transform (const u32x w0[4], const u32x w1[4], const u32x w2[4], const u32x w3[4], u32x digest[4])
{
  u32x a = digest[0];
  u32x b = digest[1];
  u32x c = digest[2];
  u32x d = digest[3];

  u32x w0_t = w0[0];
  u32x w1_t = w0[1];
  u32x w2_t = w0[2];
  u32x w3_t = w0[3];
  u32x w4_t = w1[0];
  u32x w5_t = w1[1];
  u32x w6_t = w1[2];
  u32x w7_t = w1[3];
  u32x w8_t = w2[0];
  u32x w9_t = w2[1];
  u32x wa_t = w2[2];
  u32x wb_t = w2[3];
  u32x wc_t = w3[0];
  u32x wd_t = w3[1];
  u32x we_t = w3[2];
  u32x wf_t = 0;

  u32x tmp2;

  MD5_STEP (MD5_Fo, a, b, c, d, w0_t, MD5C00, MD5S00);
  MD5_STEP (MD5_Fo, d, a, b, c, w1_t, MD5C01, MD5S01);
  MD5_STEP (MD5_Fo, c, d, a, b, w2_t, MD5C02, MD5S02);
  MD5_STEP (MD5_Fo, b, c, d, a, w3_t, MD5C03, MD5S03);
  MD5_STEP (MD5_Fo, a, b, c, d, w4_t, MD5C04, MD5S00);
  MD5_STEP (MD5_Fo, d, a, b, c, w5_t, MD5C05, MD5S01);
  MD5_STEP (MD5_Fo, c, d, a, b, w6_t, MD5C06, MD5S02);
  MD5_STEP (MD5_Fo, b, c, d, a, w7_t, MD5C07, MD5S03);
  MD5_STEP (MD5_Fo, a, b, c, d, w8_t, MD5C08, MD5S00);
  MD5_STEP (MD5_Fo, d, a, b, c, w9_t, MD5C09, MD5S01);
  MD5_STEP (MD5_Fo, c, d, a, b, wa_t, MD5C0a, MD5S02);
  MD5_STEP (MD5_Fo, b, c, d, a, wb_t, MD5C0b, MD5S03);
  MD5_STEP (MD5_Fo, a, b, c, d, wc_t, MD5C0c, MD5S00);
  MD5_STEP (MD5_Fo, d, a, b, c, wd_t, MD5C0d, MD5S01);
  MD5_STEP (MD5_Fo, c, d, a, b, we_t, MD5C0e, MD5S02);
  MD5_STEP (MD5_Fo, b, c, d, a, wf_t, MD5C0f, MD5S03);

  MD5_STEP (MD5_Go, a, b, c, d, w1_t, MD5C10, MD5S10);
  MD5_STEP (MD5_Go, d, a, b, c, w6_t, MD5C11, MD5S11);
  MD5_STEP (MD5_Go, c, d, a, b, wb_t, MD5C12, MD5S12);
  MD5_STEP (MD5_Go, b, c, d, a, w0_t, MD5C13, MD5S13);
  MD5_STEP (MD5_Go, a, b, c, d, w5_t, MD5C14, MD5S10);
  MD5_STEP (MD5_Go, d, a, b, c, wa_t, MD5C15, MD5S11);
  MD5_STEP (MD5_Go, c, d, a, b, wf_t, MD5C16, MD5S12);
  MD5_STEP (MD5_Go, b, c, d, a, w4_t, MD5C17, MD5S13);
  MD5_STEP (MD5_Go, a, b, c, d, w9_t, MD5C18, MD5S10);
  MD5_STEP (MD5_Go, d, a, b, c, we_t, MD5C19, MD5S11);
  MD5_STEP (MD5_Go, c, d, a, b, w3_t, MD5C1a, MD5S12);
  MD5_STEP (MD5_Go, b, c, d, a, w8_t, MD5C1b, MD5S13);
  MD5_STEP (MD5_Go, a, b, c, d, wd_t, MD5C1c, MD5S10);
  MD5_STEP (MD5_Go, d, a, b, c, w2_t, MD5C1d, MD5S11);
  MD5_STEP (MD5_Go, c, d, a, b, w7_t, MD5C1e, MD5S12);
  MD5_STEP (MD5_Go, b, c, d, a, wc_t, MD5C1f, MD5S13);

  MD5_STEP (MD5_H1, a, b, c, d, w5_t, MD5C20, MD5S20);
  MD5_STEP (MD5_H2, d, a, b, c, w8_t, MD5C21, MD5S21);
  MD5_STEP (MD5_H1, c, d, a, b, wb_t, MD5C22, MD5S22);
  MD5_STEP (MD5_H2, b, c, d, a, we_t, MD5C23, MD5S23);
  MD5_STEP (MD5_H1, a, b, c, d, w1_t, MD5C24, MD5S20);
  MD5_STEP (MD5_H2, d, a, b, c, w4_t, MD5C25, MD5S21);
  MD5_STEP (MD5_H1, c, d, a, b, w7_t, MD5C26, MD5S22);
  MD5_STEP (MD5_H2, b, c, d, a, wa_t, MD5C27, MD5S23);
  MD5_STEP (MD5_H1, a, b, c, d, wd_t, MD5C28, MD5S20);
  MD5_STEP (MD5_H2, d, a, b, c, w0_t, MD5C29, MD5S21);
  MD5_STEP (MD5_H1, c, d, a, b, w3_t, MD5C2a, MD5S22);
  MD5_STEP (MD5_H2, b, c, d, a, w6_t, MD5C2b, MD5S23);
  MD5_STEP (MD5_H1, a, b, c, d, w9_t, MD5C2c, MD5S20);
  MD5_STEP (MD5_H2, d, a, b, c, wc_t, MD5C2d, MD5S21);
  MD5_STEP (MD5_H1, c, d, a, b, wf_t, MD5C2e, MD5S22);
  MD5_STEP (MD5_H2, b, c, d, a, w2_t, MD5C2f, MD5S23);

  MD5_STEP (MD5_I , a, b, c, d, w0_t, MD5C30, MD5S30);
  MD5_STEP (MD5_I , d, a, b, c, w7_t, MD5C31, MD5S31);
  MD5_STEP (MD5_I , c, d, a, b, we_t, MD5C32, MD5S32);
  MD5_STEP (MD5_I , b, c, d, a, w5_t, MD5C33, MD5S33);
  MD5_STEP (MD5_I , a, b, c, d, wc_t, MD5C34, MD5S30);
  MD5_STEP (MD5_I , d, a, b, c, w3_t, MD5C35, MD5S31);
  MD5_STEP (MD5_I , c, d, a, b, wa_t, MD5C36, MD5S32);
  MD5_STEP (MD5_I , b, c, d, a, w1_t, MD5C37, MD5S33);
  MD5_STEP (MD5_I , a, b, c, d, w8_t, MD5C38, MD5S30);
  MD5_STEP (MD5_I , d, a, b, c, wf_t, MD5C39, MD5S31);
  MD5_STEP (MD5_I , c, d, a, b, w6_t, MD5C3a, MD5S32);
  MD5_STEP (MD5_I , b, c, d, a, wd_t, MD5C3b, MD5S33);
  MD5_STEP (MD5_I , a, b, c, d, w4_t, MD5C3c, MD5S30);
  MD5_STEP (MD5_I , d, a, b, c, wb_t, MD5C3d, MD5S31);
  MD5_STEP (MD5_I , c, d, a, b, w2_t, MD5C3e, MD5S32);
  MD5_STEP (MD5_I , b, c, d, a, w9_t, MD5C3f, MD5S33);

  digest[0] += a;
  digest[1] += b;
  digest[2] += c;
  digest[3] += d;
}

__device__ static void memcat16 (u32x block0[4], u32x block1[4], u32x block2[4], u32x block3[4], const u32 block_len, const u32x append[4])
{
  u32x tmp0;
  u32x tmp1;
  u32x tmp2;
  u32x tmp3;
  u32x tmp4;

  #if __CUDA_ARCH__ >= 200

  const int offset_minus_4 = 4 - (block_len & 3);

  const int selector = (0x76543210 >> (offset_minus_4 * 4)) & 0xffff;

  tmp0 = __byte_perm (        0, append[0], selector);
  tmp1 = __byte_perm (append[0], append[1], selector);
  tmp2 = __byte_perm (append[1], append[2], selector);
  tmp3 = __byte_perm (append[2], append[3], selector);
  tmp4 = __byte_perm (append[3],         0, selector);

  #else

  const u32 mod = block_len & 3;

  switch (mod)
  {
    case 0: tmp0 = append[0];
            tmp1 = append[1];
            tmp2 = append[2];
            tmp3 = append[3];
            tmp4 = 0;
            break;
    case 1: tmp0 =                   append[0] <<  8;
            tmp1 = append[0] >> 24 | append[1] <<  8;
            tmp2 = append[1] >> 24 | append[2] <<  8;
            tmp3 = append[2] >> 24 | append[3] <<  8;
            tmp4 = append[3] >> 24;
            break;
    case 2: tmp0 =                   append[0] << 16;
            tmp1 = append[0] >> 16 | append[1] << 16;
            tmp2 = append[1] >> 16 | append[2] << 16;
            tmp3 = append[2] >> 16 | append[3] << 16;
            tmp4 = append[3] >> 16;
            break;
    case 3: tmp0 =                   append[0] << 24;
            tmp1 = append[0] >>  8 | append[1] << 24;
            tmp2 = append[1] >>  8 | append[2] << 24;
            tmp3 = append[2] >>  8 | append[3] << 24;
            tmp4 = append[3] >>  8;
            break;
  }

  #endif

  const u32 div = block_len / 4;

  switch (div)
  {
    case  0:  block0[0] |= tmp0;
              block0[1]  = tmp1;
              block0[2]  = tmp2;
              block0[3]  = tmp3;
              block1[0]  = tmp4;
              break;
    case  1:  block0[1] |= tmp0;
              block0[2]  = tmp1;
              block0[3]  = tmp2;
              block1[0]  = tmp3;
              block1[1]  = tmp4;
              break;
    case  2:  block0[2] |= tmp0;
              block0[3]  = tmp1;
              block1[0]  = tmp2;
              block1[1]  = tmp3;
              block1[2]  = tmp4;
              break;
    case  3:  block0[3] |= tmp0;
              block1[0]  = tmp1;
              block1[1]  = tmp2;
              block1[2]  = tmp3;
              block1[3]  = tmp4;
              break;
    case  4:  block1[0] |= tmp0;
              block1[1]  = tmp1;
              block1[2]  = tmp2;
              block1[3]  = tmp3;
              block2[0]  = tmp4;
              break;
    case  5:  block1[1] |= tmp0;
              block1[2]  = tmp1;
              block1[3]  = tmp2;
              block2[0]  = tmp3;
              block2[1]  = tmp4;
              break;
    case  6:  block1[2] |= tmp0;
              block1[3]  = tmp1;
              block2[0]  = tmp2;
              block2[1]  = tmp3;
              block2[2]  = tmp4;
              break;
    case  7:  block1[3] |= tmp0;
              block2[0]  = tmp1;
              block2[1]  = tmp2;
              block2[2]  = tmp3;
              block2[3]  = tmp4;
              break;
    case  8:  block2[0] |= tmp0;
              block2[1]  = tmp1;
              block2[2]  = tmp2;
              block2[3]  = tmp3;
              block3[0]  = tmp4;
              break;
    case  9:  block2[1] |= tmp0;
              block2[2]  = tmp1;
              block2[3]  = tmp2;
              block3[0]  = tmp3;
              block3[1]  = tmp4;
              break;
  }

  return;
}

__device__ static void memcat16_x80 (u32x block0[4], u32x block1[4], u32x block2[4], u32x block3[4], const u32 block_len, const u32x append[4])
{
  u32x tmp0;
  u32x tmp1;
  u32x tmp2;
  u32x tmp3;
  u32x tmp4;

  #if __CUDA_ARCH__ >= 200

  const int offset_minus_4 = 4 - (block_len & 3);

  const int selector = (0x76543210 >> (offset_minus_4 * 4)) & 0xffff;

  tmp0 = __byte_perm (        0, append[0], selector);
  tmp1 = __byte_perm (append[0], append[1], selector);
  tmp2 = __byte_perm (append[1], append[2], selector);
  tmp3 = __byte_perm (append[2], append[3], selector);
  tmp4 = __byte_perm (append[3],      0x80, selector);

  #else

  const u32 mod = block_len & 3;

  switch (mod)
  {
    case 0: tmp0 = append[0];
            tmp1 = append[1];
            tmp2 = append[2];
            tmp3 = append[3];
            tmp4 = 0x80;
            break;
    case 1: tmp0 =                   append[0] <<  8;
            tmp1 = append[0] >> 24 | append[1] <<  8;
            tmp2 = append[1] >> 24 | append[2] <<  8;
            tmp3 = append[2] >> 24 | append[3] <<  8;
            tmp4 = append[3] >> 24;
            break;
    case 2: tmp0 =                   append[0] << 16;
            tmp1 = append[0] >> 16 | append[1] << 16;
            tmp2 = append[1] >> 16 | append[2] << 16;
            tmp3 = append[2] >> 16 | append[3] << 16;
            tmp4 = append[3] >> 16;
            break;
    case 3: tmp0 =                   append[0] << 24;
            tmp1 = append[0] >>  8 | append[1] << 24;
            tmp2 = append[1] >>  8 | append[2] << 24;
            tmp3 = append[2] >>  8 | append[3] << 24;
            tmp4 = append[3] >>  8;
            break;
  }

  #endif

  const u32 div = block_len / 4;

  switch (div)
  {
    case  0:  block0[0] |= tmp0;
              block0[1]  = tmp1;
              block0[2]  = tmp2;
              block0[3]  = tmp3;
              block1[0]  = tmp4;
              break;
    case  1:  block0[1] |= tmp0;
              block0[2]  = tmp1;
              block0[3]  = tmp2;
              block1[0]  = tmp3;
              block1[1]  = tmp4;
              break;
    case  2:  block0[2] |= tmp0;
              block0[3]  = tmp1;
              block1[0]  = tmp2;
              block1[1]  = tmp3;
              block1[2]  = tmp4;
              break;
    case  3:  block0[3] |= tmp0;
              block1[0]  = tmp1;
              block1[1]  = tmp2;
              block1[2]  = tmp3;
              block1[3]  = tmp4;
              break;
    case  4:  block1[0] |= tmp0;
              block1[1]  = tmp1;
              block1[2]  = tmp2;
              block1[3]  = tmp3;
              block2[0]  = tmp4;
              break;
    case  5:  block1[1] |= tmp0;
              block1[2]  = tmp1;
              block1[3]  = tmp2;
              block2[0]  = tmp3;
              block2[1]  = tmp4;
              break;
    case  6:  block1[2] |= tmp0;
              block1[3]  = tmp1;
              block2[0]  = tmp2;
              block2[1]  = tmp3;
              block2[2]  = tmp4;
              break;
    case  7:  block1[3] |= tmp0;
              block2[0]  = tmp1;
              block2[1]  = tmp2;
              block2[2]  = tmp3;
              block2[3]  = tmp4;
              break;
    case  8:  block2[0] |= tmp0;
              block2[1]  = tmp1;
              block2[2]  = tmp2;
              block2[3]  = tmp3;
              block3[0]  = tmp4;
              break;
    case  9:  block2[1] |= tmp0;
              block2[2]  = tmp1;
              block2[3]  = tmp2;
              block3[0]  = tmp3;
              block3[1]  = tmp4;
              break;
  }

  return;
}

__device__ static void memcat8 (u32x block0[4], u32x block1[4], u32x block2[4], u32x block3[4], const u32 block_len, const u32 append[2])
{
  u32x tmp0;
  u32x tmp1;
  u32x tmp2;

  #if __CUDA_ARCH__ >= 200

  const int offset_minus_4 = 4 - (block_len & 3);

  const int selector = (0x76543210 >> (offset_minus_4 * 4)) & 0xffff;

  tmp0 = __byte_perm (        0, append[0], selector);
  tmp1 = __byte_perm (append[0], append[1], selector);
  tmp2 = __byte_perm (append[1],         0, selector);

  #else

  const u32 mod = block_len & 3;

  switch (mod)
  {
    case 0: tmp0 = append[0];
            tmp1 = append[1];
            tmp2 = 0;
            break;
    case 1: tmp0 =                   append[0] <<  8;
            tmp1 = append[0] >> 24 | append[1] <<  8;
            tmp2 = append[1] >> 24;
            break;
    case 2: tmp0 =                   append[0] << 16;
            tmp1 = append[0] >> 16 | append[1] << 16;
            tmp2 = append[1] >> 16;
            break;
    case 3: tmp0 =                   append[0] << 24;
            tmp1 = append[0] >>  8 | append[1] << 24;
            tmp2 = append[1] >>  8;
            break;
  }

  #endif

  const u32 div = block_len / 4;

  switch (div)
  {
    case  0:  block0[0] |= tmp0;
              block0[1]  = tmp1;
              block0[2]  = tmp2;
              break;
    case  1:  block0[1] |= tmp0;
              block0[2]  = tmp1;
              block0[3]  = tmp2;
              break;
    case  2:  block0[2] |= tmp0;
              block0[3]  = tmp1;
              block1[0]  = tmp2;
              break;
    case  3:  block0[3] |= tmp0;
              block1[0]  = tmp1;
              block1[1]  = tmp2;
              break;
    case  4:  block1[0] |= tmp0;
              block1[1]  = tmp1;
              block1[2]  = tmp2;
              break;
    case  5:  block1[1] |= tmp0;
              block1[2]  = tmp1;
              block1[3]  = tmp2;
              break;
    case  6:  block1[2] |= tmp0;
              block1[3]  = tmp1;
              block2[0]  = tmp2;
              break;
    case  7:  block1[3] |= tmp0;
              block2[0]  = tmp1;
              block2[1]  = tmp2;
              break;
    case  8:  block2[0] |= tmp0;
              block2[1]  = tmp1;
              block2[2]  = tmp2;
              break;
    case  9:  block2[1] |= tmp0;
              block2[2]  = tmp1;
              block2[3]  = tmp2;
              break;
    case 10:  block2[2] |= tmp0;
              block2[3]  = tmp1;
              block3[0]  = tmp2;
              break;
    case 11:  block2[3] |= tmp0;
              block3[0]  = tmp1;
              block3[1]  = tmp2;
              break;
  }

  return;
}

__device__ static void append_sign (u32x block0[4], u32x block1[4], const u32 block_len)
{
  switch (block_len)
  {
    case 0:
      block0[0] = md5apr1_magic0;
      block0[1] = md5apr1_magic1;
      break;

    case 1:
      block0[0] = block0[0]            | md5apr1_magic0 <<  8;
      block0[1] = md5apr1_magic0 >> 24 | md5apr1_magic1 <<  8;
      block0[2] = md5apr1_magic1 >> 24;
      break;

    case 2:
      block0[0] = block0[0]            | md5apr1_magic0 << 16;
      block0[1] = md5apr1_magic0 >> 16 | md5apr1_magic1 << 16;
      block0[2] = md5apr1_magic1 >> 16;
      break;

    case 3:
      block0[0] = block0[0]            | md5apr1_magic0 << 24;
      block0[1] = md5apr1_magic0 >>  8 | md5apr1_magic1 << 24;
      block0[2] = md5apr1_magic1 >>  8;
      break;

    case 4:
      block0[1] = md5apr1_magic0;
      block0[2] = md5apr1_magic1;
      break;

    case 5:
      block0[1] = block0[1]            | md5apr1_magic0 <<  8;
      block0[2] = md5apr1_magic0 >> 24 | md5apr1_magic1 <<  8;
      block0[3] = md5apr1_magic1 >> 24;
      break;

    case 6:
      block0[1] = block0[1]            | md5apr1_magic0 << 16;
      block0[2] = md5apr1_magic0 >> 16 | md5apr1_magic1 << 16;
      block0[3] = md5apr1_magic1 >> 16;
      break;

    case 7:
      block0[1] = block0[1]            | md5apr1_magic0 << 24;
      block0[2] = md5apr1_magic0 >>  8 | md5apr1_magic1 << 24;
      block0[3] = md5apr1_magic1 >>  8;
      break;

    case 8:
      block0[2] = md5apr1_magic0;
      block0[3] = md5apr1_magic1;
      break;

    case 9:
      block0[2] = block0[2]            | md5apr1_magic0 <<  8;
      block0[3] = md5apr1_magic0 >> 24 | md5apr1_magic1 <<  8;
      block1[0] = md5apr1_magic1 >> 24;
      break;

    case 10:
      block0[2] = block0[2]            | md5apr1_magic0 << 16;
      block0[3] = md5apr1_magic0 >> 16 | md5apr1_magic1 << 16;
      block1[0] = md5apr1_magic1 >> 16;
      break;

    case 11:
      block0[2] = block0[2]            | md5apr1_magic0 << 24;
      block0[3] = md5apr1_magic0 >>  8 | md5apr1_magic1 << 24;
      block1[0] = md5apr1_magic1 >>  8;
      break;

    case 12:
      block0[3] = md5apr1_magic0;
      block1[0] = md5apr1_magic1;
      break;

    case 13:
      block0[3] = block0[3]            | md5apr1_magic0 <<  8;
      block1[0] = md5apr1_magic0 >> 24 | md5apr1_magic1 <<  8;
      block1[1] = md5apr1_magic1 >> 24;
      break;

    case 14:
      block0[3] = block0[3]            | md5apr1_magic0 << 16;
      block1[0] = md5apr1_magic0 >> 16 | md5apr1_magic1 << 16;
      block1[1] = md5apr1_magic1 >> 16;
      break;

    case 15:
      block0[3] = block0[3]            | md5apr1_magic0 << 24;
      block1[0] = md5apr1_magic0 >>  8 | md5apr1_magic1 << 24;
      block1[1] = md5apr1_magic1 >>  8;
      break;
  }
}

__device__ static void append_1st (u32x block0[4], u32x block1[4], u32x block2[4], u32x block3[4], const u32 block_len, const u32x append)
{
  switch (block_len)
  {
    case 0:
      block0[0] = append;
      break;

    case 1:
      block0[0] = block0[0] | append <<  8;
      break;

    case 2:
      block0[0] = block0[0] | append << 16;
      break;

    case 3:
      block0[0] = block0[0] | append << 24;
      break;

    case 4:
      block0[1] = append;
      break;

    case 5:
      block0[1] = block0[1] | append <<  8;
      break;

    case 6:
      block0[1] = block0[1] | append << 16;
      break;

    case 7:
      block0[1] = block0[1] | append << 24;
      break;

    case 8:
      block0[2] = append;
      break;

    case 9:
      block0[2] = block0[2] | append <<  8;
      break;

    case 10:
      block0[2] = block0[2] | append << 16;
      break;

    case 11:
      block0[2] = block0[2] | append << 24;
      break;

    case 12:
      block0[3] = append;
      break;

    case 13:
      block0[3] = block0[3] | append <<  8;
      break;

    case 14:
      block0[3] = block0[3] | append << 16;
      break;

    case 15:
      block0[3] = block0[3] | append << 24;
      break;

    case 16:
      block1[0] = append;
      break;

    case 17:
      block1[0] = block1[0] | append <<  8;
      break;

    case 18:
      block1[0] = block1[0] | append << 16;
      break;

    case 19:
      block1[0] = block1[0] | append << 24;
      break;

    case 20:
      block1[1] = append;
      break;

    case 21:
      block1[1] = block1[1] | append <<  8;
      break;

    case 22:
      block1[1] = block1[1] | append << 16;
      break;

    case 23:
      block1[1] = block1[1] | append << 24;
      break;

    case 24:
      block1[2] = append;
      break;

    case 25:
      block1[2] = block1[2] | append <<  8;
      break;

    case 26:
      block1[2] = block1[2] | append << 16;
      break;

    case 27:
      block1[2] = block1[2] | append << 24;
      break;

    case 28:
      block1[3] = append;
      break;

    case 29:
      block1[3] = block1[3] | append <<  8;
      break;

    case 30:
      block1[3] = block1[3] | append << 16;
      break;

    case 31:
      block1[3] = block1[3] | append << 24;
      break;

    case 32:
      block2[0] = append;
      break;

    case 33:
      block2[0] = block2[0] | append <<  8;
      break;

    case 34:
      block2[0] = block2[0] | append << 16;
      break;

    case 35:
      block2[0] = block2[0] | append << 24;
      break;

    case 36:
      block2[1] = append;
      break;

    case 37:
      block2[1] = block2[1] | append <<  8;
      break;

    case 38:
      block2[1] = block2[1] | append << 16;
      break;

    case 39:
      block2[1] = block2[1] | append << 24;
      break;

    case 40:
      block2[2] = append;
      break;

    case 41:
      block2[2] = block2[2] | append <<  8;
      break;

    case 42:
      block2[2] = block2[2] | append << 16;
      break;

    case 43:
      block2[2] = block2[2] | append << 24;
      break;

    case 44:
      block2[3] = append;
      break;

    case 45:
      block2[3] = block2[3] | append <<  8;
      break;

    case 46:
      block2[3] = block2[3] | append << 16;
      break;

    case 47:
      block2[3] = block2[3] | append << 24;
      break;

    case 48:
      block3[0] = append;
      break;

    case 49:
      block3[0] = block3[0] | append <<  8;
      break;

    case 50:
      block3[0] = block3[0] | append << 16;
      break;

    case 51:
      block3[0] = block3[0] | append << 24;
      break;

    case 52:
      block3[1] = append;
      break;

    case 53:
      block3[1] = block3[1] | append <<  8;
      break;

    case 54:
      block3[1] = block3[1] | append << 16;
      break;

    case 55:
      block3[1] = block3[1] | append << 24;
      break;

    case 56:
      block3[2] = append;
      break;
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m01600_init (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, md5crypt_tmp_t *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x w0[4];

  w0[0] = pws[gid].i[0];
  w0[1] = pws[gid].i[1];
  w0[2] = pws[gid].i[2];
  w0[3] = pws[gid].i[3];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * salt
   */

  u32 salt_buf[2];

  salt_buf[0] = salt_bufs[salt_pos].salt_buf[0];
  salt_buf[1] = salt_bufs[salt_pos].salt_buf[1];

  const u32 salt_len = salt_bufs[salt_pos].salt_len;

  /**
   * init
   */

  //memcat16 (block0, block1, block2, block3, block_len, w0);
  //block_len += pw_len;

  u32 block_len = pw_len;

  u32x block0[4];

  block0[0] = w0[0];
  block0[1] = w0[1];
  block0[2] = w0[2];
  block0[3] = w0[3];

  u32x block1[4];

  block1[0] = 0;
  block1[1] = 0;
  block1[2] = 0;
  block1[3] = 0;

  u32x block2[4];

  block2[0] = 0;
  block2[1] = 0;
  block2[2] = 0;
  block2[3] = 0;

  u32x block3[4];

  block3[0] = 0;
  block3[1] = 0;
  block3[2] = 0;
  block3[3] = 0;

  memcat8 (block0, block1, block2, block3, block_len, salt_buf);

  block_len += salt_len;

  memcat16 (block0, block1, block2, block3, block_len, w0);

  block_len += pw_len;

  append_0x80_4 (block0, block1, block2, block3, block_len);

  block3[2] = block_len * 8;

  u32x digest[4];

  digest[0] = MD5M_A;
  digest[1] = MD5M_B;
  digest[2] = MD5M_C;
  digest[3] = MD5M_D;

  md5_transform (block0, block1, block2, block3, digest);

  /* The password first, since that is what is most unknown */
  /* Then our magic string */
  /* Then the raw salt */
  /* Then just as many characters of the MD5(pw,salt,pw) */

  //memcat16 (block0, block1, block2, block3, block_len, w);
  //block_len += pw_len;

  block_len = pw_len;

  block0[0] = w0[0];
  block0[1] = w0[1];
  block0[2] = w0[2];
  block0[3] = w0[3];

  block1[0] = 0;
  block1[1] = 0;
  block1[2] = 0;
  block1[3] = 0;

  block2[0] = 0;
  block2[1] = 0;
  block2[2] = 0;
  block2[3] = 0;

  block3[0] = 0;
  block3[1] = 0;
  block3[2] = 0;
  block3[3] = 0;

  append_sign (block0, block1, block_len);

  block_len += 6;

  memcat8 (block0, block1, block2, block3, block_len, salt_buf);

  block_len += salt_len;

  truncate_block (digest, pw_len);

  memcat16 (block0, block1, block2, block3, block_len, digest);

  block_len += pw_len;

  /* Then something really weird... */

  u32x append = block0[0] & 0xFF;

  for (u32 j = pw_len; j; j >>= 1)
  {
    if ((j & 1) == 0)
    {
      append_1st (block0, block1, block2, block3, block_len, append);
    }

    block_len++;
  }

  append_0x80_4 (block0, block1, block2, block3, block_len);

  block3[2] = block_len * 8;

  digest[0] = MD5M_A;
  digest[1] = MD5M_B;
  digest[2] = MD5M_C;
  digest[3] = MD5M_D;

  md5_transform (block0, block1, block2, block3, digest);

  tmps[gid].digest_buf[0] = digest[0];
  tmps[gid].digest_buf[1] = digest[1];
  tmps[gid].digest_buf[2] = digest[2];
  tmps[gid].digest_buf[3] = digest[3];
}

extern "C" __global__ void __launch_bounds__ (256, 1) m01600_loop (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, md5crypt_tmp_t *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x w0[4];

  w0[0] = pws[gid].i[0];
  w0[1] = pws[gid].i[1];
  w0[2] = pws[gid].i[2];
  w0[3] = pws[gid].i[3];

  const u32 pw_len = pws[gid].pw_len;

  u32x w0_x80[4];

  w0_x80[0] = w0[0];
  w0_x80[1] = w0[1];
  w0_x80[2] = w0[2];
  w0_x80[3] = w0[3];

  append_0x80_1 (w0_x80, pw_len);

  /**
   * salt
   */

  u32 salt_buf[2];

  salt_buf[0] = salt_bufs[salt_pos].salt_buf[0];
  salt_buf[1] = salt_bufs[salt_pos].salt_buf[1];

  const u32 salt_len = salt_bufs[salt_pos].salt_len;

  /**
   * digest
   */

  u32x digest[4];

  digest[0] = tmps[gid].digest_buf[0];
  digest[1] = tmps[gid].digest_buf[1];
  digest[2] = tmps[gid].digest_buf[2];
  digest[3] = tmps[gid].digest_buf[3];

  /**
   * loop
   */

  /* and now, just to make sure things don't run too fast */

  u32 block_len;

  u32x block0[4];

  block0[0] = 0;
  block0[1] = 0;
  block0[2] = 0;
  block0[3] = 0;

  u32x block1[4];

  block1[0] = 0;
  block1[1] = 0;
  block1[2] = 0;
  block1[3] = 0;

  u32x block2[4];

  block2[0] = 0;
  block2[1] = 0;
  block2[2] = 0;
  block2[3] = 0;

  u32x block3[4];

  block3[0] = 0;
  block3[1] = 0;
  block3[2] = 0;
  block3[3] = 0;

  for (u32 i = 0, j = loop_pos; i < loop_cnt; i++, j++)
  {
    block1[0] = 0;
    block1[1] = 0;
    block1[2] = 0;
    block1[3] = 0;
    block2[0] = 0;
    block2[1] = 0;
    block2[2] = 0;
    block2[3] = 0;
    block3[0] = 0;
    block3[1] = 0;

    const u32 j1 = (j & 1) ? 1 : 0;
    const u32 j3 = (j % 3) ? 1 : 0;
    const u32 j7 = (j % 7) ? 1 : 0;

    if (j1)
    {
      block0[0] = w0[0];
      block0[1] = w0[1];
      block0[2] = w0[2];
      block0[3] = w0[3];

      block_len = pw_len;

      if (j3)
      {
        memcat8 (block0, block1, block2, block3, block_len, salt_buf);

        block_len += salt_len;
      }

      if (j7)
      {
        memcat16 (block0, block1, block2, block3, block_len, w0);

        block_len += pw_len;
      }

      memcat16_x80 (block0, block1, block2, block3, block_len, digest);

      block_len += 16;
    }
    else
    {
      block0[0] = digest[0];
      block0[1] = digest[1];
      block0[2] = digest[2];
      block0[3] = digest[3];

      block_len = 16;

      if (j3 && j7)
      {
        block1[0] = salt_buf[0];
        block1[1] = salt_buf[1];

        block_len += salt_len;

        memcat16 (block0, block1, block2, block3, block_len, w0);

        block_len += pw_len;
      }
      else if (j3)
      {
        block1[0] = salt_buf[0];
        block1[1] = salt_buf[1];

        block_len += salt_len;
      }
      else if (j7)
      {
        block1[0] = w0[0];
        block1[1] = w0[1];
        block1[2] = w0[2];
        block1[3] = w0[3];

        block_len += pw_len;
      }

      memcat16 (block0, block1, block2, block3, block_len, w0_x80);

      block_len += pw_len;
    }

    block3[2] = block_len * 8;

    digest[0] = MD5M_A;
    digest[1] = MD5M_B;
    digest[2] = MD5M_C;
    digest[3] = MD5M_D;

    md5_transform (block0, block1, block2, block3, digest);
  }

  tmps[gid].digest_buf[0] = digest[0];
  tmps[gid].digest_buf[1] = digest[1];
  tmps[gid].digest_buf[2] = digest[2];
  tmps[gid].digest_buf[3] = digest[3];
}

extern "C" __global__ void __launch_bounds__ (256, 1) m01600_comp (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, md5crypt_tmp_t *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  const u32 lid = threadIdx.x;

  /**
   * digest
   */

  const u32x r0 = tmps[gid].digest_buf[DGST_R0];
  const u32x r1 = tmps[gid].digest_buf[DGST_R1];
  const u32x r2 = tmps[gid].digest_buf[DGST_R2];
  const u32x r3 = tmps[gid].digest_buf[DGST_R3];

  #define il_pos 0

  #include VECT_COMPARE_M
}
