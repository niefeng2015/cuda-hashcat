#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _SCRYPT_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 0
#define DGST_R1 1
#define DGST_R2 2
#define DGST_R3 3

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

#ifdef  VECT_SIZE4
#define VECT_COMPARE_M "check_multi_vect4_comp4.c"
#endif

class uintm
{
  private:
  public:

  u32 x;
  u32 y;
  u32 z;
  u32 w;

    inline __device__  uintm (const u32 a, const u32 b, const u32 c, const u32 d) : x(a), y(b), z(c), w(d) { }
    inline __device__  uintm (const u32 a)                                           : x(a), y(a), z(a), w(a) { }

    inline __device__  uintm (void) { }
    inline __device__ ~uintm (void) { }
};

typedef struct
{
  uintm P[64];

} scrypt_tmp_t;

__device__ static uintm __byte_perm (const uintm a, const uintm b, const u32 c)
{
  return uintm (__byte_perm (a.x, b.x, c),
                __byte_perm (a.y, b.y, c),
                __byte_perm (a.z, b.z, c),
                __byte_perm (a.w, b.w, c));
}

__device__ static uintm rotate (const uintm a, const unsigned int n)
{
  return uintm  (rotl32 (a.x, n),
                 rotl32 (a.y, n),
                 rotl32 (a.z, n),
                 rotl32 (a.w, n));
}

inline __device__ uintm wxyz (const uintm a) { return uintm (a.w, a.x, a.y, a.z); }
inline __device__ uintm zwxy (const uintm a) { return uintm (a.z, a.w, a.x, a.y); }

inline __device__ uintm operator << (const uintm  a, const u32  b) { return uintm ((a.x << b  ), (a.y << b  ), (a.z << b  ), (a.w << b  ));  }
inline __device__ uintm operator << (const uintm  a, const uintm b) { return uintm ((a.x << b.x), (a.y << b.y), (a.z << b.z), (a.w << b.w));  }

inline __device__ uintm operator >> (const uintm  a, const u32  b) { return uintm ((a.x >> b  ), (a.y >> b  ), (a.z >> b  ), (a.w >> b  ));  }
inline __device__ uintm operator >> (const uintm  a, const uintm b) { return uintm ((a.x >> b.x), (a.y >> b.y), (a.z >> b.z), (a.w >> b.w));  }

inline __device__ uintm operator ^  (const uintm  a, const u32  b) { return uintm ((a.x ^  b  ), (a.y ^  b  ), (a.z ^  b  ), (a.w ^  b  ));  }
inline __device__ uintm operator ^  (const uintm  a, const uintm b) { return uintm ((a.x ^  b.x), (a.y ^  b.y), (a.z ^  b.z), (a.w ^  b.w));  }

inline __device__ uintm operator |  (const uintm  a, const u32  b) { return uintm ((a.x |  b  ), (a.y |  b  ), (a.z |  b  ), (a.w |  b  ));  }
inline __device__ uintm operator |  (const uintm  a, const uintm b) { return uintm ((a.x |  b.x), (a.y |  b.y), (a.z |  b.z), (a.w |  b.w));  }

inline __device__ uintm operator &  (const uintm  a, const u32  b) { return uintm ((a.x &  b  ), (a.y &  b  ), (a.z &  b  ), (a.w &  b  ));  }
inline __device__ uintm operator &  (const uintm  a, const uintm b) { return uintm ((a.x &  b.x), (a.y &  b.y), (a.z &  b.z), (a.w &  b.w));  }

inline __device__ uintm operator +  (const uintm  a, const u32  b) { return uintm ((a.x +  b  ), (a.y +  b  ), (a.z +  b  ), (a.w +  b  ));  }
inline __device__ uintm operator +  (const uintm  a, const uintm b) { return uintm ((a.x +  b.x), (a.y +  b.y), (a.z +  b.z), (a.w +  b.w));  }

inline __device__ void  operator ^= (uintm &a, const u32  b) { a.x ^= b;   a.y ^= b;   a.z ^= b;   a.w ^= b;   }
inline __device__ void  operator ^= (uintm &a, const uintm b) { a.x ^= b.x; a.y ^= b.y; a.z ^= b.z; a.w ^= b.w; }

inline __device__ void  operator += (uintm &a, const u32  b) { a.x += b;   a.y += b;   a.z += b;   a.w += b;   }
inline __device__ void  operator += (uintm &a, const uintm b) { a.x += b.x; a.y += b.y; a.z += b.z; a.w += b.w; }

__constant__ u32 k_sha256[64] =
{
  SHA256C00, SHA256C01, SHA256C02, SHA256C03,
  SHA256C04, SHA256C05, SHA256C06, SHA256C07,
  SHA256C08, SHA256C09, SHA256C0a, SHA256C0b,
  SHA256C0c, SHA256C0d, SHA256C0e, SHA256C0f,
  SHA256C10, SHA256C11, SHA256C12, SHA256C13,
  SHA256C14, SHA256C15, SHA256C16, SHA256C17,
  SHA256C18, SHA256C19, SHA256C1a, SHA256C1b,
  SHA256C1c, SHA256C1d, SHA256C1e, SHA256C1f,
  SHA256C20, SHA256C21, SHA256C22, SHA256C23,
  SHA256C24, SHA256C25, SHA256C26, SHA256C27,
  SHA256C28, SHA256C29, SHA256C2a, SHA256C2b,
  SHA256C2c, SHA256C2d, SHA256C2e, SHA256C2f,
  SHA256C30, SHA256C31, SHA256C32, SHA256C33,
  SHA256C34, SHA256C35, SHA256C36, SHA256C37,
  SHA256C38, SHA256C39, SHA256C3a, SHA256C3b,
  SHA256C3c, SHA256C3d, SHA256C3e, SHA256C3f,
};

__device__ static void sha256_transform (const u32x w0[4], const u32x w1[4], const u32x w2[4], const u32x w3[4], u32x digest[8])
{
  u32x a = digest[0];
  u32x b = digest[1];
  u32x c = digest[2];
  u32x d = digest[3];
  u32x e = digest[4];
  u32x f = digest[5];
  u32x g = digest[6];
  u32x h = digest[7];

  u32x w0_t = w0[0];
  u32x w1_t = w0[1];
  u32x w2_t = w0[2];
  u32x w3_t = w0[3];
  u32x w4_t = w1[0];
  u32x w5_t = w1[1];
  u32x w6_t = w1[2];
  u32x w7_t = w1[3];
  u32x w8_t = w2[0];
  u32x w9_t = w2[1];
  u32x wa_t = w2[2];
  u32x wb_t = w2[3];
  u32x wc_t = w3[0];
  u32x wd_t = w3[1];
  u32x we_t = w3[2];
  u32x wf_t = w3[3];

  #define ROUND_EXPAND()                            \
  {                                                 \
    w0_t = SHA256_EXPAND (we_t, w9_t, w1_t, w0_t);  \
    w1_t = SHA256_EXPAND (wf_t, wa_t, w2_t, w1_t);  \
    w2_t = SHA256_EXPAND (w0_t, wb_t, w3_t, w2_t);  \
    w3_t = SHA256_EXPAND (w1_t, wc_t, w4_t, w3_t);  \
    w4_t = SHA256_EXPAND (w2_t, wd_t, w5_t, w4_t);  \
    w5_t = SHA256_EXPAND (w3_t, we_t, w6_t, w5_t);  \
    w6_t = SHA256_EXPAND (w4_t, wf_t, w7_t, w6_t);  \
    w7_t = SHA256_EXPAND (w5_t, w0_t, w8_t, w7_t);  \
    w8_t = SHA256_EXPAND (w6_t, w1_t, w9_t, w8_t);  \
    w9_t = SHA256_EXPAND (w7_t, w2_t, wa_t, w9_t);  \
    wa_t = SHA256_EXPAND (w8_t, w3_t, wb_t, wa_t);  \
    wb_t = SHA256_EXPAND (w9_t, w4_t, wc_t, wb_t);  \
    wc_t = SHA256_EXPAND (wa_t, w5_t, wd_t, wc_t);  \
    wd_t = SHA256_EXPAND (wb_t, w6_t, we_t, wd_t);  \
    we_t = SHA256_EXPAND (wc_t, w7_t, wf_t, we_t);  \
    wf_t = SHA256_EXPAND (wd_t, w8_t, w0_t, wf_t);  \
  }

  #define ROUND_STEP(i)                                                                   \
  {                                                                                       \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w0_t, k_sha256[i +  0]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w1_t, k_sha256[i +  1]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, w2_t, k_sha256[i +  2]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, w3_t, k_sha256[i +  3]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, w4_t, k_sha256[i +  4]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, w5_t, k_sha256[i +  5]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, w6_t, k_sha256[i +  6]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, w7_t, k_sha256[i +  7]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, a, b, c, d, e, f, g, h, w8_t, k_sha256[i +  8]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, h, a, b, c, d, e, f, g, w9_t, k_sha256[i +  9]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, g, h, a, b, c, d, e, f, wa_t, k_sha256[i + 10]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, f, g, h, a, b, c, d, e, wb_t, k_sha256[i + 11]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, e, f, g, h, a, b, c, d, wc_t, k_sha256[i + 12]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, d, e, f, g, h, a, b, c, wd_t, k_sha256[i + 13]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, c, d, e, f, g, h, a, b, we_t, k_sha256[i + 14]); \
    SHA256_STEP (SHA256_F0o, SHA256_F1o, b, c, d, e, f, g, h, a, wf_t, k_sha256[i + 15]); \
  }

  ROUND_STEP (0);

  for (int i = 16; i < 64; i += 16)
  {
    ROUND_EXPAND (); ROUND_STEP (i);
  }

  digest[0] += a;
  digest[1] += b;
  digest[2] += c;
  digest[3] += d;
  digest[4] += e;
  digest[5] += f;
  digest[6] += g;
  digest[7] += h;
}

__device__ static void hmac_sha256_pad (u32x w0[4], u32x w1[4], u32x w2[4], u32x w3[4], u32x ipad[8], u32x opad[8])
{
  w0[0] = w0[0] ^ 0x36363636;
  w0[1] = w0[1] ^ 0x36363636;
  w0[2] = w0[2] ^ 0x36363636;
  w0[3] = w0[3] ^ 0x36363636;
  w1[0] = w1[0] ^ 0x36363636;
  w1[1] = w1[1] ^ 0x36363636;
  w1[2] = w1[2] ^ 0x36363636;
  w1[3] = w1[3] ^ 0x36363636;
  w2[0] = w2[0] ^ 0x36363636;
  w2[1] = w2[1] ^ 0x36363636;
  w2[2] = w2[2] ^ 0x36363636;
  w2[3] = w2[3] ^ 0x36363636;
  w3[0] = w3[0] ^ 0x36363636;
  w3[1] = w3[1] ^ 0x36363636;
  w3[2] = w3[2] ^ 0x36363636;
  w3[3] = w3[3] ^ 0x36363636;

  ipad[0] = SHA256M_A;
  ipad[1] = SHA256M_B;
  ipad[2] = SHA256M_C;
  ipad[3] = SHA256M_D;
  ipad[4] = SHA256M_E;
  ipad[5] = SHA256M_F;
  ipad[6] = SHA256M_G;
  ipad[7] = SHA256M_H;

  sha256_transform (w0, w1, w2, w3, ipad);

  w0[0] = w0[0] ^ 0x6a6a6a6a;
  w0[1] = w0[1] ^ 0x6a6a6a6a;
  w0[2] = w0[2] ^ 0x6a6a6a6a;
  w0[3] = w0[3] ^ 0x6a6a6a6a;
  w1[0] = w1[0] ^ 0x6a6a6a6a;
  w1[1] = w1[1] ^ 0x6a6a6a6a;
  w1[2] = w1[2] ^ 0x6a6a6a6a;
  w1[3] = w1[3] ^ 0x6a6a6a6a;
  w2[0] = w2[0] ^ 0x6a6a6a6a;
  w2[1] = w2[1] ^ 0x6a6a6a6a;
  w2[2] = w2[2] ^ 0x6a6a6a6a;
  w2[3] = w2[3] ^ 0x6a6a6a6a;
  w3[0] = w3[0] ^ 0x6a6a6a6a;
  w3[1] = w3[1] ^ 0x6a6a6a6a;
  w3[2] = w3[2] ^ 0x6a6a6a6a;
  w3[3] = w3[3] ^ 0x6a6a6a6a;

  opad[0] = SHA256M_A;
  opad[1] = SHA256M_B;
  opad[2] = SHA256M_C;
  opad[3] = SHA256M_D;
  opad[4] = SHA256M_E;
  opad[5] = SHA256M_F;
  opad[6] = SHA256M_G;
  opad[7] = SHA256M_H;

  sha256_transform (w0, w1, w2, w3, opad);
}

__device__ static void hmac_sha256_run (u32x w0[4], u32x w1[4], u32x w2[4], u32x w3[4], u32x ipad[8], u32x opad[8], u32x digest[8])
{
  digest[0] = ipad[0];
  digest[1] = ipad[1];
  digest[2] = ipad[2];
  digest[3] = ipad[3];
  digest[4] = ipad[4];
  digest[5] = ipad[5];
  digest[6] = ipad[6];
  digest[7] = ipad[7];

  sha256_transform (w0, w1, w2, w3, digest);

  w0[0] = digest[0];
  w0[1] = digest[1];
  w0[2] = digest[2];
  w0[3] = digest[3];
  w1[0] = digest[4];
  w1[1] = digest[5];
  w1[2] = digest[6];
  w1[3] = digest[7];
  w2[0] = 0x80000000;
  w2[1] = 0;
  w2[2] = 0;
  w2[3] = 0;
  w3[0] = 0;
  w3[1] = 0;
  w3[2] = 0;
  w3[3] = (64 + 32) * 8;

  digest[0] = opad[0];
  digest[1] = opad[1];
  digest[2] = opad[2];
  digest[3] = opad[3];
  digest[4] = opad[4];
  digest[5] = opad[5];
  digest[6] = opad[6];
  digest[7] = opad[7];

  sha256_transform (w0, w1, w2, w3, digest);
}

__device__ static void memcat8 (u32x block0[4], u32x block1[4], u32x block2[4], u32x block3[4], const u32 block_len, const u32 append[2])
{
  switch (block_len)
  {
    case 0:
      block0[0] = append[0];
      block0[1] = append[1];
      break;

    case 1:
      block0[0] = block0[0]       | append[0] <<  8;
      block0[1] = append[0] >> 24 | append[1] <<  8;
      block0[2] = append[1] >> 24;
      break;

    case 2:
      block0[0] = block0[0]       | append[0] << 16;
      block0[1] = append[0] >> 16 | append[1] << 16;
      block0[2] = append[1] >> 16;
      break;

    case 3:
      block0[0] = block0[0]       | append[0] << 24;
      block0[1] = append[0] >>  8 | append[1] << 24;
      block0[2] = append[1] >>  8;
      break;

    case 4:
      block0[1] = append[0];
      block0[2] = append[1];
      break;

    case 5:
      block0[1] = block0[1]       | append[0] <<  8;
      block0[2] = append[0] >> 24 | append[1] <<  8;
      block0[3] = append[1] >> 24;
      break;

    case 6:
      block0[1] = block0[1]       | append[0] << 16;
      block0[2] = append[0] >> 16 | append[1] << 16;
      block0[3] = append[1] >> 16;
      break;

    case 7:
      block0[1] = block0[1]       | append[0] << 24;
      block0[2] = append[0] >>  8 | append[1] << 24;
      block0[3] = append[1] >>  8;
      break;

    case 8:
      block0[2] = append[0];
      block0[3] = append[1];
      break;

    case 9:
      block0[2] = block0[2]       | append[0] <<  8;
      block0[3] = append[0] >> 24 | append[1] <<  8;
      block1[0] = append[1] >> 24;
      break;

    case 10:
      block0[2] = block0[2]       | append[0] << 16;
      block0[3] = append[0] >> 16 | append[1] << 16;
      block1[0] = append[1] >> 16;
      break;

    case 11:
      block0[2] = block0[2]       | append[0] << 24;
      block0[3] = append[0] >>  8 | append[1] << 24;
      block1[0] = append[1] >>  8;
      break;

    case 12:
      block0[3] = append[0];
      block1[0] = append[1];
      break;

    case 13:
      block0[3] = block0[3]       | append[0] <<  8;
      block1[0] = append[0] >> 24 | append[1] <<  8;
      block1[1] = append[1] >> 24;
      break;

    case 14:
      block0[3] = block0[3]       | append[0] << 16;
      block1[0] = append[0] >> 16 | append[1] << 16;
      block1[1] = append[1] >> 16;
      break;

    case 15:
      block0[3] = block0[3]       | append[0] << 24;
      block1[0] = append[0] >>  8 | append[1] << 24;
      block1[1] = append[1] >>  8;
      break;

    case 16:
      block1[0] = append[0];
      block1[1] = append[1];
      break;

    case 17:
      block1[0] = block1[0]       | append[0] <<  8;
      block1[1] = append[0] >> 24 | append[1] <<  8;
      block1[2] = append[1] >> 24;
      break;

    case 18:
      block1[0] = block1[0]       | append[0] << 16;
      block1[1] = append[0] >> 16 | append[1] << 16;
      block1[2] = append[1] >> 16;
      break;

    case 19:
      block1[0] = block1[0]       | append[0] << 24;
      block1[1] = append[0] >>  8 | append[1] << 24;
      block1[2] = append[1] >>  8;
      break;

    case 20:
      block1[1] = append[0];
      block1[2] = append[1];
      break;

    case 21:
      block1[1] = block1[1]       | append[0] <<  8;
      block1[2] = append[0] >> 24 | append[1] <<  8;
      block1[3] = append[1] >> 24;
      break;

    case 22:
      block1[1] = block1[1]       | append[0] << 16;
      block1[2] = append[0] >> 16 | append[1] << 16;
      block1[3] = append[1] >> 16;
      break;

    case 23:
      block1[1] = block1[1]       | append[0] << 24;
      block1[2] = append[0] >>  8 | append[1] << 24;
      block1[3] = append[1] >>  8;
      break;

    case 24:
      block1[2] = append[0];
      block1[3] = append[1];
      break;

    case 25:
      block1[2] = block1[2]       | append[0] <<  8;
      block1[3] = append[0] >> 24 | append[1] <<  8;
      block2[0] = append[1] >> 24;
      break;

    case 26:
      block1[2] = block1[2]       | append[0] << 16;
      block1[3] = append[0] >> 16 | append[1] << 16;
      block2[0] = append[1] >> 16;
      break;

    case 27:
      block1[2] = block1[2]       | append[0] << 24;
      block1[3] = append[0] >>  8 | append[1] << 24;
      block2[0] = append[1] >>  8;
      break;

    case 28:
      block1[3] = append[0];
      block2[0] = append[1];
      break;

    case 29:
      block1[3] = block1[3]       | append[0] <<  8;
      block2[0] = append[0] >> 24 | append[1] <<  8;
      block2[1] = append[1] >> 24;
      break;

    case 30:
      block1[3] = block1[3]       | append[0] << 16;
      block2[0] = append[0] >> 16 | append[1] << 16;
      block2[1] = append[1] >> 16;
      break;

    case 31:
      block1[3] = block1[3]       | append[0] << 24;
      block2[0] = append[0] >>  8 | append[1] << 24;
      block2[1] = append[1] >>  8;
      break;

    case 32:
      block2[0] = append[0];
      block2[1] = append[1];
      break;

    case 33:
      block2[0] = block2[0]       | append[0] <<  8;
      block2[1] = append[0] >> 24 | append[1] <<  8;
      block2[2] = append[1] >> 24;
      break;

    case 34:
      block2[0] = block2[0]       | append[0] << 16;
      block2[1] = append[0] >> 16 | append[1] << 16;
      block2[2] = append[1] >> 16;
      break;

    case 35:
      block2[0] = block2[0]       | append[0] << 24;
      block2[1] = append[0] >>  8 | append[1] << 24;
      block2[2] = append[1] >>  8;
      break;

    case 36:
      block2[1] = append[0];
      block2[2] = append[1];
      break;

    case 37:
      block2[1] = block2[1]       | append[0] <<  8;
      block2[2] = append[0] >> 24 | append[1] <<  8;
      block2[3] = append[1] >> 24;
      break;

    case 38:
      block2[1] = block2[1]       | append[0] << 16;
      block2[2] = append[0] >> 16 | append[1] << 16;
      block2[3] = append[1] >> 16;
      break;

    case 39:
      block2[1] = block2[1]       | append[0] << 24;
      block2[2] = append[0] >>  8 | append[1] << 24;
      block2[3] = append[1] >>  8;
      break;

    case 40:
      block2[2] = append[0];
      block2[3] = append[1];
      break;

    case 41:
      block2[2] = block2[2]       | append[0] <<  8;
      block2[3] = append[0] >> 24 | append[1] <<  8;
      block3[0] = append[1] >> 24;
      break;

    case 42:
      block2[2] = block2[2]       | append[0] << 16;
      block2[3] = append[0] >> 16 | append[1] << 16;
      block3[0] = append[1] >> 16;
      break;

    case 43:
      block2[2] = block2[2]       | append[0] << 24;
      block2[3] = append[0] >>  8 | append[1] << 24;
      block3[0] = append[1] >>  8;
      break;

    case 44:
      block2[3] = append[0];
      block3[0] = append[1];
      break;

    case 45:
      block2[3] = block2[3]       | append[0] <<  8;
      block3[0] = append[0] >> 24 | append[1] <<  8;
      block3[1] = append[1] >> 24;
      break;

    case 46:
      block2[3] = block2[3]       | append[0] << 16;
      block3[0] = append[0] >> 16 | append[1] << 16;
      block3[1] = append[1] >> 16;
      break;

    case 47:
      block2[3] = block2[3]       | append[0] << 24;
      block3[0] = append[0] >>  8 | append[1] << 24;
      block3[1] = append[1] >>  8;
      break;

    case 48:
      block3[0] = append[0];
      block3[1] = append[1];
      break;

    case 49:
      block3[0] = block3[0]       | append[0] <<  8;
      block3[1] = append[0] >> 24 | append[1] <<  8;
      block3[2] = append[1] >> 24;
      break;

    case 50:
      block3[0] = block3[0]       | append[0] << 16;
      block3[1] = append[0] >> 16 | append[1] << 16;
      block3[2] = append[1] >> 16;
      break;

    case 51:
      block3[0] = block3[0]       | append[0] << 24;
      block3[1] = append[0] >>  8 | append[1] << 24;
      block3[2] = append[1] >>  8;
      break;

    case 52:
      block3[1] = append[0];
      block3[2] = append[1];
      break;

    case 53:
      block3[1] = block3[1]       | append[0] <<  8;
      block3[2] = append[0] >> 24 | append[1] <<  8;
      block3[3] = append[1] >> 24;
      break;

    case 54:
      block3[1] = block3[1]       | append[0] << 16;
      block3[2] = append[0] >> 16 | append[1] << 16;
      block3[3] = append[1] >> 16;
      break;

    case 55:
      block3[1] = block3[1]       | append[0] << 24;
      block3[2] = append[0] >>  8 | append[1] << 24;
      block3[3] = append[1] >>  8;
      break;

    case 56:
      block3[2] = append[0];
      block3[3] = append[1];
      break;
  }
}

__device__ static uintm swap_workaround (uintm v)
{
  return __byte_perm (v, 0, 0x0123);
}

#define GET_SCRYPT_CNT(r,p) (2 * (r) * 16 * (p))
#define GET_SMIX_CNT(r,N)   (2 * (r) * 16 * (N))
#define GET_STATE_CNT(r)    (2 * (r) * 16)

#define ADD_ROTATE_XOR(r,i1,i2,s) (r) ^= rotate ((i1) + (i2), (s));

#define SALSA20_2R()                    \
{                                       \
  ADD_ROTATE_XOR (X1, X0, X3,  7);      \
  ADD_ROTATE_XOR (X2, X1, X0,  9);      \
  ADD_ROTATE_XOR (X3, X2, X1, 13);      \
  ADD_ROTATE_XOR (X0, X3, X2, 18);      \
                                        \
  X1 = uintm (X1.w, X1.x, X1.y, X1.z);  \
  X2 = uintm (X2.z, X2.w, X2.x, X2.y);  \
  X3 = uintm (X3.y, X3.z, X3.w, X3.x);  \
                                        \
  ADD_ROTATE_XOR (X3, X0, X1,  7);      \
  ADD_ROTATE_XOR (X2, X3, X0,  9);      \
  ADD_ROTATE_XOR (X1, X2, X3, 13);      \
  ADD_ROTATE_XOR (X0, X1, X2, 18);      \
                                        \
  X1 = uintm (X1.y, X1.z, X1.w, X1.x);  \
  X2 = uintm (X2.z, X2.w, X2.x, X2.y);  \
  X3 = uintm (X3.w, X3.x, X3.y, X3.z);  \
}

#define SALSA20_8_XOR() \
{                       \
  R0 = R0 ^ Y0;         \
  R1 = R1 ^ Y1;         \
  R2 = R2 ^ Y2;         \
  R3 = R3 ^ Y3;         \
                        \
  uintm X0 = R0;        \
  uintm X1 = R1;        \
  uintm X2 = R2;        \
  uintm X3 = R3;        \
                        \
  SALSA20_2R ();        \
  SALSA20_2R ();        \
  SALSA20_2R ();        \
  SALSA20_2R ();        \
                        \
  R0 = R0 + X0;         \
  R1 = R1 + X1;         \
  R2 = R2 + X2;         \
  R3 = R3 + X3;         \
}

__device__ static void salsa_r (uintm T[8], const u32 r)
{
  const u32 state_cnt = GET_STATE_CNT (r);

  const u32 state_cnt4 = state_cnt / 4;

  uintm R0 = T[state_cnt4 - 4];
  uintm R1 = T[state_cnt4 - 3];
  uintm R2 = T[state_cnt4 - 2];
  uintm R3 = T[state_cnt4 - 1];

  for (u32 i = 0; i < state_cnt4; i += 8)
  {
    uintm Y0;
    uintm Y1;
    uintm Y2;
    uintm Y3;

    Y0 = T[i + 0];
    Y1 = T[i + 1];
    Y2 = T[i + 2];
    Y3 = T[i + 3];

    SALSA20_8_XOR ();

    T[i + 0] = R0;
    T[i + 1] = R1;
    T[i + 2] = R2;
    T[i + 3] = R3;

    Y0 = T[i + 4];
    Y1 = T[i + 5];
    Y2 = T[i + 6];
    Y3 = T[i + 7];

    SALSA20_8_XOR ();

    T[i + 4] = R0;
    T[i + 5] = R1;
    T[i + 6] = R2;
    T[i + 7] = R3;
  }

  #define exchg(x,y) { const uintm t = T[(x)]; T[(x)] = T[(y)]; T[(y)] = t; }

  #define exchg4(x,y)         \
  {                           \
    const u32 x4 = (x) * 4;  \
    const u32 y4 = (y) * 4;  \
                              \
    exchg (x4 + 0, y4 + 0);   \
    exchg (x4 + 1, y4 + 1);   \
    exchg (x4 + 2, y4 + 2);   \
    exchg (x4 + 3, y4 + 3);   \
  }

  for (u32 i = 1; i < r / 1; i++)
  {
    const u32 x = i * 1;
    const u32 y = i * 2;

    exchg4 (x, y);
  }

  for (u32 i = 1; i < r / 2; i++)
  {
    const u32 x = i * 1;
    const u32 y = i * 2;

    const u32 xr1 = (r * 2) - 1 - x;
    const u32 yr1 = (r * 2) - 1 - y;

    exchg4 (xr1, yr1);
  }
}

__device__ static void scrypt_smix (uintm *X, uintm *T, const u32 N, const u32 r, const u32 tmto, const u32 phy, uintm *V)
{
  const u32 state_cnt = GET_STATE_CNT (r);

  const u32 state_cnt4 = state_cnt / 4;

  #if __CUDA_ARCH__ >= 500
  #define Coord(x,y,z) (((y) * zSIZE) + ((x) * zSIZE * ySIZE) + (z))
  #define CO Coord(x,y,z)
  #else
  #define Coord(x,y,z) (((x) * zSIZE) + ((y) * zSIZE * xSIZE) + (z))
  #define CO Coord(x,y,z)
  #endif

  const u32 xSIZE = phy;
  const u32 ySIZE = N / tmto;
  const u32 zSIZE = state_cnt4;

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  const u32 x = gid % xSIZE;

  for (u32 i = 0; i < state_cnt4; i += 4)
  {
    T[0] = uintm (X[i + 0].x, X[i + 1].y, X[i + 2].z, X[i + 3].w);
    T[1] = uintm (X[i + 1].x, X[i + 2].y, X[i + 3].z, X[i + 0].w);
    T[2] = uintm (X[i + 2].x, X[i + 3].y, X[i + 0].z, X[i + 1].w);
    T[3] = uintm (X[i + 3].x, X[i + 0].y, X[i + 1].z, X[i + 2].w);

    X[i + 0] = T[0];
    X[i + 1] = T[1];
    X[i + 2] = T[2];
    X[i + 3] = T[3];
  }

  for (u32 y = 0; y < ySIZE; y++)
  {
    for (u32 z = 0; z < zSIZE; z++) V[CO] = X[z];

    for (u32 i = 0; i < tmto; i++) salsa_r (X, r);
  }

  for (u32 i = 0; i < N; i++)
  {
    const u32 k = X[zSIZE - 4].x & (N - 1);

    const u32 y = k / tmto;

    const u32 km = k - (y * tmto);

    for (u32 z = 0; z < zSIZE; z++) T[z] = V[CO];

    for (u32 i = 0; i < km; i++) salsa_r (T, r);

    for (u32 z = 0; z < zSIZE; z++) X[z] ^= T[z];

    salsa_r (X, r);
  }

  for (u32 i = 0; i < state_cnt4; i += 4)
  {
    T[0] = uintm (X[i + 0].x, X[i + 3].y, X[i + 2].z, X[i + 1].w);
    T[1] = uintm (X[i + 1].x, X[i + 0].y, X[i + 3].z, X[i + 2].w);
    T[2] = uintm (X[i + 2].x, X[i + 1].y, X[i + 0].z, X[i + 3].w);
    T[3] = uintm (X[i + 3].x, X[i + 2].y, X[i + 1].z, X[i + 0].w);

    X[i + 0] = T[0];
    X[i + 1] = T[1];
    X[i + 2] = T[2];
    X[i + 3] = T[3];
  }
}

extern "C" __global__ void __launch_bounds__ (64, 1) m08900_init (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, scrypt_tmp_t *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, uintm *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32x w0[4];

  w0[0] = pws[gid].i[ 0];
  w0[1] = pws[gid].i[ 1];
  w0[2] = pws[gid].i[ 2];
  w0[3] = pws[gid].i[ 3];

  u32x w1[4];

  w1[0] = pws[gid].i[ 4];
  w1[1] = pws[gid].i[ 5];
  w1[2] = pws[gid].i[ 6];
  w1[3] = pws[gid].i[ 7];

  u32x w2[4];

  w2[0] = pws[gid].i[ 8];
  w2[1] = pws[gid].i[ 9];
  w2[2] = pws[gid].i[10];
  w2[3] = pws[gid].i[11];

  u32x w3[4];

  w3[0] = pws[gid].i[12];
  w3[1] = pws[gid].i[13];
  w3[2] = pws[gid].i[14];
  w3[3] = pws[gid].i[15];

  /**
   * salt
   */

  u32 salt_buf0[4];

  salt_buf0[0] = salt_bufs[salt_pos].salt_buf[0];
  salt_buf0[1] = salt_bufs[salt_pos].salt_buf[1];
  salt_buf0[2] = salt_bufs[salt_pos].salt_buf[2];
  salt_buf0[3] = salt_bufs[salt_pos].salt_buf[3];

  u32 salt_buf1[4];

  salt_buf1[0] = salt_bufs[salt_pos].salt_buf[4];
  salt_buf1[1] = salt_bufs[salt_pos].salt_buf[5];
  salt_buf1[2] = salt_bufs[salt_pos].salt_buf[6];
  salt_buf1[3] = salt_bufs[salt_pos].salt_buf[7];

  const u32 salt_len = salt_bufs[salt_pos].salt_len;

  /**
   * memory buffers
   */

  const u32 scrypt_r = SCRYPT_R;
  const u32 scrypt_p = SCRYPT_P;
  //const u32 scrypt_N = SCRYPT_N;

  //const u32 state_cnt  = GET_STATE_CNT  (scrypt_r);
  const u32 scrypt_cnt = GET_SCRYPT_CNT (scrypt_r, scrypt_p);
  //const u32 smix_cnt   = GET_SMIX_CNT   (scrypt_r, scrypt_N);

  /**
   * 1st pbkdf2, creates B
   */

  w0[0] = swap_workaround (w0[0]);
  w0[1] = swap_workaround (w0[1]);
  w0[2] = swap_workaround (w0[2]);
  w0[3] = swap_workaround (w0[3]);
  w1[0] = swap_workaround (w1[0]);
  w1[1] = swap_workaround (w1[1]);
  w1[2] = swap_workaround (w1[2]);
  w1[3] = swap_workaround (w1[3]);
  w2[0] = swap_workaround (w2[0]);
  w2[1] = swap_workaround (w2[1]);
  w2[2] = swap_workaround (w2[2]);
  w2[3] = swap_workaround (w2[3]);
  w3[0] = swap_workaround (w3[0]);
  w3[1] = swap_workaround (w3[1]);
  w3[2] = swap_workaround (w3[2]);
  w3[3] = swap_workaround (w3[3]);

  u32 ipad[8];
  u32 opad[8];

  hmac_sha256_pad (w0, w1, w2, w3, ipad, opad);

  for (u32 i = 0, j = 0, k = 0; i < scrypt_cnt; i += 8, j += 1, k += 2)
  {
    w0[0] = salt_buf0[0];
    w0[1] = salt_buf0[1];
    w0[2] = salt_buf0[2];
    w0[3] = salt_buf0[3];
    w1[0] = salt_buf1[0];
    w1[1] = salt_buf1[1];
    w1[2] = salt_buf1[2];
    w1[3] = salt_buf1[3];
    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;
    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = 0;

    u32 append[2];

    append[0] = swap_workaround (j + 1);
    append[1] = 0x80;

    memcat8 (w0, w1, w2, w3, salt_len, append);

    w0[0] = swap_workaround (w0[0]);
    w0[1] = swap_workaround (w0[1]);
    w0[2] = swap_workaround (w0[2]);
    w0[3] = swap_workaround (w0[3]);
    w1[0] = swap_workaround (w1[0]);
    w1[1] = swap_workaround (w1[1]);
    w1[2] = swap_workaround (w1[2]);
    w1[3] = swap_workaround (w1[3]);
    w2[0] = swap_workaround (w2[0]);
    w2[1] = swap_workaround (w2[1]);
    w2[2] = swap_workaround (w2[2]);
    w2[3] = swap_workaround (w2[3]);
    w3[0] = swap_workaround (w3[0]);
    w3[1] = swap_workaround (w3[1]);
    w3[2] = 0;
    w3[3] = (64 + salt_len + 4) * 8;

    u32x digest[8];

    hmac_sha256_run (w0, w1, w2, w3, ipad, opad, digest);

    const uintm tmp0 = uintm (digest[0], digest[1], digest[2], digest[3]);
    const uintm tmp1 = uintm (digest[4], digest[5], digest[6], digest[7]);

    __syncthreads ();

    tmps[gid].P[k + 0] = tmp0;
    tmps[gid].P[k + 1] = tmp1;
  }
}

extern "C" __global__ void __launch_bounds__ (64, 1) m08900_loop (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, scrypt_tmp_t *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, uintm *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  const u32 scrypt_phy   = salt_bufs[salt_pos].scrypt_phy;

  const u32 state_cnt    = GET_STATE_CNT  (SCRYPT_R);
  const u32 scrypt_cnt   = GET_SCRYPT_CNT (SCRYPT_R, SCRYPT_P);

  const u32 state_cnt4   = state_cnt  / 4;
  const u32 scrypt_cnt4  = scrypt_cnt / 4;

  uintm X[state_cnt4];
  uintm T[state_cnt4];

  for (int z = 0; z < state_cnt4; z++) X[z] = swap_workaround (tmps[gid].P[z]);

  scrypt_smix (X, T, SCRYPT_N, SCRYPT_R, SCRYPT_TMTO, scrypt_phy, d_scryptV_buf);

  for (int z = 0; z < state_cnt4; z++) tmps[gid].P[z] = swap_workaround (X[z]);

  #if SCRYPT_P >= 1
  for (int i = state_cnt4; i < scrypt_cnt4; i += state_cnt4)
  {
    for (int z = 0; z < state_cnt4; z++) X[z] = swap_workaround (tmps[gid].P[i + z]);

    scrypt_smix (X, T, SCRYPT_N, SCRYPT_R, SCRYPT_TMTO, scrypt_phy, d_scryptV_buf);

    for (int z = 0; z < state_cnt4; z++) tmps[gid].P[i + z] = swap_workaround (X[z]);
  }
  #endif
}

extern "C" __global__ void __launch_bounds__ (64, 1) m08900_comp (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, scrypt_tmp_t *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, uintm *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 rules_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;;
  const u32 lid = threadIdx.x;

  if (gid >= gid_max) return;

  u32x w0[4];

  w0[0] = pws[gid].i[ 0];
  w0[1] = pws[gid].i[ 1];
  w0[2] = pws[gid].i[ 2];
  w0[3] = pws[gid].i[ 3];

  u32x w1[4];

  w1[0] = pws[gid].i[ 4];
  w1[1] = pws[gid].i[ 5];
  w1[2] = pws[gid].i[ 6];
  w1[3] = pws[gid].i[ 7];

  u32x w2[4];

  w2[0] = pws[gid].i[ 8];
  w2[1] = pws[gid].i[ 9];
  w2[2] = pws[gid].i[10];
  w2[3] = pws[gid].i[11];

  u32x w3[4];

  w3[0] = pws[gid].i[12];
  w3[1] = pws[gid].i[13];
  w3[2] = pws[gid].i[14];
  w3[3] = pws[gid].i[15];

  /**
   * memory buffers
   */

  const u32 scrypt_r = SCRYPT_R;
  const u32 scrypt_p = SCRYPT_P;

  const u32 scrypt_cnt = GET_SCRYPT_CNT (scrypt_r, scrypt_p);

  const u32 scrypt_cnt4  = scrypt_cnt / 4;

  /**
   * 2nd pbkdf2, creates B
   */

  w0[0] = swap_workaround (w0[0]);
  w0[1] = swap_workaround (w0[1]);
  w0[2] = swap_workaround (w0[2]);
  w0[3] = swap_workaround (w0[3]);
  w1[0] = swap_workaround (w1[0]);
  w1[1] = swap_workaround (w1[1]);
  w1[2] = swap_workaround (w1[2]);
  w1[3] = swap_workaround (w1[3]);
  w2[0] = swap_workaround (w2[0]);
  w2[1] = swap_workaround (w2[1]);
  w2[2] = swap_workaround (w2[2]);
  w2[3] = swap_workaround (w2[3]);
  w3[0] = swap_workaround (w3[0]);
  w3[1] = swap_workaround (w3[1]);
  w3[2] = swap_workaround (w3[2]);
  w3[3] = swap_workaround (w3[3]);

  u32 ipad[8];
  u32 opad[8];

  hmac_sha256_pad (w0, w1, w2, w3, ipad, opad);

  for (u32 l = 0; l < scrypt_cnt4; l += 4)
  {
    __syncthreads ();

    uintm tmp;

    tmp = tmps[gid].P[l + 0];

    w0[0] = tmp.x;
    w0[1] = tmp.y;
    w0[2] = tmp.z;
    w0[3] = tmp.w;

    tmp = tmps[gid].P[l + 1];

    w1[0] = tmp.x;
    w1[1] = tmp.y;
    w1[2] = tmp.z;
    w1[3] = tmp.w;

    tmp = tmps[gid].P[l + 2];

    w2[0] = tmp.x;
    w2[1] = tmp.y;
    w2[2] = tmp.z;
    w2[3] = tmp.w;

    tmp = tmps[gid].P[l + 3];

    w3[0] = tmp.x;
    w3[1] = tmp.y;
    w3[2] = tmp.z;
    w3[3] = tmp.w;

    sha256_transform (w0, w1, w2, w3, ipad);
  }

  w0[0] = 0x00000001;
  w0[1] = 0x80000000;
  w0[2] = 0;
  w0[3] = 0;
  w1[0] = 0;
  w1[1] = 0;
  w1[2] = 0;
  w1[3] = 0;
  w2[0] = 0;
  w2[1] = 0;
  w2[2] = 0;
  w2[3] = 0;
  w3[0] = 0;
  w3[1] = 0;
  w3[2] = 0;
  w3[3] = (64 + (scrypt_cnt * 4) + 4) * 8;

  u32x digest[8];

  hmac_sha256_run (w0, w1, w2, w3, ipad, opad, digest);

  const u32x r0 = swap_workaround (digest[DGST_R0]);
  const u32x r1 = swap_workaround (digest[DGST_R1]);
  const u32x r2 = swap_workaround (digest[DGST_R2]);
  const u32x r3 = swap_workaround (digest[DGST_R3]);

  #define il_pos 0

  #include VECT_COMPARE_M
}
