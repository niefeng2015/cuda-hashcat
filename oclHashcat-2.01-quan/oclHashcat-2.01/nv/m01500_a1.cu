#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _DES_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 0
#define DGST_R1 1
#define DGST_R2 2
#define DGST_R3 3

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_S "check_single_vect1_comp4.c"
#define VECT_COMPARE_M "check_multi_vect1_comp4.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_S "check_single_vect2_comp4.c"
#define VECT_COMPARE_M "check_multi_vect2_comp4.c"
#endif

#ifdef  VECT_SIZE4
#define VECT_COMPARE_S "check_single_vect4_comp4.c"
#define VECT_COMPARE_M "check_multi_vect4_comp4.c"
#endif

__device__ __constant__ comb_t c_combs[1024];

#define PERM_OP(a,b,tt,n,m) \
{                           \
  tt = a >> n;              \
  tt = tt ^ b;              \
  tt = tt & m;              \
  b = b ^ tt;               \
  tt = tt << n;             \
  a = a ^ tt;               \
}

#define HPERM_OP(a,tt,n,m)  \
{                           \
  tt = a << (16 + n);       \
  tt = tt ^ a;              \
  tt = tt & m;              \
  a  = a ^ tt;              \
  tt = tt >> (16 + n);      \
  a  = a ^ tt;              \
}

__device__ __constant__ u32 c_SPtrans[8][64] =
{
  /* nibble 0 */
  0x00820200, 0x00020000, 0x80800000, 0x80820200,
  0x00800000, 0x80020200, 0x80020000, 0x80800000,
  0x80020200, 0x00820200, 0x00820000, 0x80000200,
  0x80800200, 0x00800000, 0x00000000, 0x80020000,
  0x00020000, 0x80000000, 0x00800200, 0x00020200,
  0x80820200, 0x00820000, 0x80000200, 0x00800200,
  0x80000000, 0x00000200, 0x00020200, 0x80820000,
  0x00000200, 0x80800200, 0x80820000, 0x00000000,
  0x00000000, 0x80820200, 0x00800200, 0x80020000,
  0x00820200, 0x00020000, 0x80000200, 0x00800200,
  0x80820000, 0x00000200, 0x00020200, 0x80800000,
  0x80020200, 0x80000000, 0x80800000, 0x00820000,
  0x80820200, 0x00020200, 0x00820000, 0x80800200,
  0x00800000, 0x80000200, 0x80020000, 0x00000000,
  0x00020000, 0x00800000, 0x80800200, 0x00820200,
  0x80000000, 0x80820000, 0x00000200, 0x80020200,
  /* nibble 1 */
  0x10042004, 0x00000000, 0x00042000, 0x10040000,
  0x10000004, 0x00002004, 0x10002000, 0x00042000,
  0x00002000, 0x10040004, 0x00000004, 0x10002000,
  0x00040004, 0x10042000, 0x10040000, 0x00000004,
  0x00040000, 0x10002004, 0x10040004, 0x00002000,
  0x00042004, 0x10000000, 0x00000000, 0x00040004,
  0x10002004, 0x00042004, 0x10042000, 0x10000004,
  0x10000000, 0x00040000, 0x00002004, 0x10042004,
  0x00040004, 0x10042000, 0x10002000, 0x00042004,
  0x10042004, 0x00040004, 0x10000004, 0x00000000,
  0x10000000, 0x00002004, 0x00040000, 0x10040004,
  0x00002000, 0x10000000, 0x00042004, 0x10002004,
  0x10042000, 0x00002000, 0x00000000, 0x10000004,
  0x00000004, 0x10042004, 0x00042000, 0x10040000,
  0x10040004, 0x00040000, 0x00002004, 0x10002000,
  0x10002004, 0x00000004, 0x10040000, 0x00042000,
  /* nibble 2 */
  0x41000000, 0x01010040, 0x00000040, 0x41000040,
  0x40010000, 0x01000000, 0x41000040, 0x00010040,
  0x01000040, 0x00010000, 0x01010000, 0x40000000,
  0x41010040, 0x40000040, 0x40000000, 0x41010000,
  0x00000000, 0x40010000, 0x01010040, 0x00000040,
  0x40000040, 0x41010040, 0x00010000, 0x41000000,
  0x41010000, 0x01000040, 0x40010040, 0x01010000,
  0x00010040, 0x00000000, 0x01000000, 0x40010040,
  0x01010040, 0x00000040, 0x40000000, 0x00010000,
  0x40000040, 0x40010000, 0x01010000, 0x41000040,
  0x00000000, 0x01010040, 0x00010040, 0x41010000,
  0x40010000, 0x01000000, 0x41010040, 0x40000000,
  0x40010040, 0x41000000, 0x01000000, 0x41010040,
  0x00010000, 0x01000040, 0x41000040, 0x00010040,
  0x01000040, 0x00000000, 0x41010000, 0x40000040,
  0x41000000, 0x40010040, 0x00000040, 0x01010000,
  /* nibble 3 */
  0x00100402, 0x04000400, 0x00000002, 0x04100402,
  0x00000000, 0x04100000, 0x04000402, 0x00100002,
  0x04100400, 0x04000002, 0x04000000, 0x00000402,
  0x04000002, 0x00100402, 0x00100000, 0x04000000,
  0x04100002, 0x00100400, 0x00000400, 0x00000002,
  0x00100400, 0x04000402, 0x04100000, 0x00000400,
  0x00000402, 0x00000000, 0x00100002, 0x04100400,
  0x04000400, 0x04100002, 0x04100402, 0x00100000,
  0x04100002, 0x00000402, 0x00100000, 0x04000002,
  0x00100400, 0x04000400, 0x00000002, 0x04100000,
  0x04000402, 0x00000000, 0x00000400, 0x00100002,
  0x00000000, 0x04100002, 0x04100400, 0x00000400,
  0x04000000, 0x04100402, 0x00100402, 0x00100000,
  0x04100402, 0x00000002, 0x04000400, 0x00100402,
  0x00100002, 0x00100400, 0x04100000, 0x04000402,
  0x00000402, 0x04000000, 0x04000002, 0x04100400,
  /* nibble 4 */
  0x02000000, 0x00004000, 0x00000100, 0x02004108,
  0x02004008, 0x02000100, 0x00004108, 0x02004000,
  0x00004000, 0x00000008, 0x02000008, 0x00004100,
  0x02000108, 0x02004008, 0x02004100, 0x00000000,
  0x00004100, 0x02000000, 0x00004008, 0x00000108,
  0x02000100, 0x00004108, 0x00000000, 0x02000008,
  0x00000008, 0x02000108, 0x02004108, 0x00004008,
  0x02004000, 0x00000100, 0x00000108, 0x02004100,
  0x02004100, 0x02000108, 0x00004008, 0x02004000,
  0x00004000, 0x00000008, 0x02000008, 0x02000100,
  0x02000000, 0x00004100, 0x02004108, 0x00000000,
  0x00004108, 0x02000000, 0x00000100, 0x00004008,
  0x02000108, 0x00000100, 0x00000000, 0x02004108,
  0x02004008, 0x02004100, 0x00000108, 0x00004000,
  0x00004100, 0x02004008, 0x02000100, 0x00000108,
  0x00000008, 0x00004108, 0x02004000, 0x02000008,
  /* nibble 5 */
  0x20000010, 0x00080010, 0x00000000, 0x20080800,
  0x00080010, 0x00000800, 0x20000810, 0x00080000,
  0x00000810, 0x20080810, 0x00080800, 0x20000000,
  0x20000800, 0x20000010, 0x20080000, 0x00080810,
  0x00080000, 0x20000810, 0x20080010, 0x00000000,
  0x00000800, 0x00000010, 0x20080800, 0x20080010,
  0x20080810, 0x20080000, 0x20000000, 0x00000810,
  0x00000010, 0x00080800, 0x00080810, 0x20000800,
  0x00000810, 0x20000000, 0x20000800, 0x00080810,
  0x20080800, 0x00080010, 0x00000000, 0x20000800,
  0x20000000, 0x00000800, 0x20080010, 0x00080000,
  0x00080010, 0x20080810, 0x00080800, 0x00000010,
  0x20080810, 0x00080800, 0x00080000, 0x20000810,
  0x20000010, 0x20080000, 0x00080810, 0x00000000,
  0x00000800, 0x20000010, 0x20000810, 0x20080800,
  0x20080000, 0x00000810, 0x00000010, 0x20080010,
  /* nibble 6 */
  0x00001000, 0x00000080, 0x00400080, 0x00400001,
  0x00401081, 0x00001001, 0x00001080, 0x00000000,
  0x00400000, 0x00400081, 0x00000081, 0x00401000,
  0x00000001, 0x00401080, 0x00401000, 0x00000081,
  0x00400081, 0x00001000, 0x00001001, 0x00401081,
  0x00000000, 0x00400080, 0x00400001, 0x00001080,
  0x00401001, 0x00001081, 0x00401080, 0x00000001,
  0x00001081, 0x00401001, 0x00000080, 0x00400000,
  0x00001081, 0x00401000, 0x00401001, 0x00000081,
  0x00001000, 0x00000080, 0x00400000, 0x00401001,
  0x00400081, 0x00001081, 0x00001080, 0x00000000,
  0x00000080, 0x00400001, 0x00000001, 0x00400080,
  0x00000000, 0x00400081, 0x00400080, 0x00001080,
  0x00000081, 0x00001000, 0x00401081, 0x00400000,
  0x00401080, 0x00000001, 0x00001001, 0x00401081,
  0x00400001, 0x00401080, 0x00401000, 0x00001001,
  /* nibble 7 */
  0x08200020, 0x08208000, 0x00008020, 0x00000000,
  0x08008000, 0x00200020, 0x08200000, 0x08208020,
  0x00000020, 0x08000000, 0x00208000, 0x00008020,
  0x00208020, 0x08008020, 0x08000020, 0x08200000,
  0x00008000, 0x00208020, 0x00200020, 0x08008000,
  0x08208020, 0x08000020, 0x00000000, 0x00208000,
  0x08000000, 0x00200000, 0x08008020, 0x08200020,
  0x00200000, 0x00008000, 0x08208000, 0x00000020,
  0x00200000, 0x00008000, 0x08000020, 0x08208020,
  0x00008020, 0x08000000, 0x00000000, 0x00208000,
  0x08200020, 0x08008020, 0x08008000, 0x00200020,
  0x08208000, 0x00000020, 0x00200020, 0x08008000,
  0x08208020, 0x00200000, 0x08200000, 0x08000020,
  0x00208000, 0x00008020, 0x08008020, 0x08200000,
  0x00000020, 0x08208000, 0x00208020, 0x00000000,
  0x08000000, 0x08200020, 0x00008000, 0x00208020
};

__device__ __constant__ u32 c_skb[8][64] =
{
  /* for C bits (numbered as per FIPS 46) 1 2 3 4 5 6 */
  0x00000000, 0x00000010, 0x20000000, 0x20000010,
  0x00010000, 0x00010010, 0x20010000, 0x20010010,
  0x00000800, 0x00000810, 0x20000800, 0x20000810,
  0x00010800, 0x00010810, 0x20010800, 0x20010810,
  0x00000020, 0x00000030, 0x20000020, 0x20000030,
  0x00010020, 0x00010030, 0x20010020, 0x20010030,
  0x00000820, 0x00000830, 0x20000820, 0x20000830,
  0x00010820, 0x00010830, 0x20010820, 0x20010830,
  0x00080000, 0x00080010, 0x20080000, 0x20080010,
  0x00090000, 0x00090010, 0x20090000, 0x20090010,
  0x00080800, 0x00080810, 0x20080800, 0x20080810,
  0x00090800, 0x00090810, 0x20090800, 0x20090810,
  0x00080020, 0x00080030, 0x20080020, 0x20080030,
  0x00090020, 0x00090030, 0x20090020, 0x20090030,
  0x00080820, 0x00080830, 0x20080820, 0x20080830,
  0x00090820, 0x00090830, 0x20090820, 0x20090830,
  /* for C bits (numbered as per FIPS 46) 7 8 10 11 12 13 */
  0x00000000, 0x02000000, 0x00002000, 0x02002000,
  0x00200000, 0x02200000, 0x00202000, 0x02202000,
  0x00000004, 0x02000004, 0x00002004, 0x02002004,
  0x00200004, 0x02200004, 0x00202004, 0x02202004,
  0x00000400, 0x02000400, 0x00002400, 0x02002400,
  0x00200400, 0x02200400, 0x00202400, 0x02202400,
  0x00000404, 0x02000404, 0x00002404, 0x02002404,
  0x00200404, 0x02200404, 0x00202404, 0x02202404,
  0x10000000, 0x12000000, 0x10002000, 0x12002000,
  0x10200000, 0x12200000, 0x10202000, 0x12202000,
  0x10000004, 0x12000004, 0x10002004, 0x12002004,
  0x10200004, 0x12200004, 0x10202004, 0x12202004,
  0x10000400, 0x12000400, 0x10002400, 0x12002400,
  0x10200400, 0x12200400, 0x10202400, 0x12202400,
  0x10000404, 0x12000404, 0x10002404, 0x12002404,
  0x10200404, 0x12200404, 0x10202404, 0x12202404,
  /* for C bits (numbered as per FIPS 46) 14 15 16 17 19 20 */
  0x00000000, 0x00000001, 0x00040000, 0x00040001,
  0x01000000, 0x01000001, 0x01040000, 0x01040001,
  0x00000002, 0x00000003, 0x00040002, 0x00040003,
  0x01000002, 0x01000003, 0x01040002, 0x01040003,
  0x00000200, 0x00000201, 0x00040200, 0x00040201,
  0x01000200, 0x01000201, 0x01040200, 0x01040201,
  0x00000202, 0x00000203, 0x00040202, 0x00040203,
  0x01000202, 0x01000203, 0x01040202, 0x01040203,
  0x08000000, 0x08000001, 0x08040000, 0x08040001,
  0x09000000, 0x09000001, 0x09040000, 0x09040001,
  0x08000002, 0x08000003, 0x08040002, 0x08040003,
  0x09000002, 0x09000003, 0x09040002, 0x09040003,
  0x08000200, 0x08000201, 0x08040200, 0x08040201,
  0x09000200, 0x09000201, 0x09040200, 0x09040201,
  0x08000202, 0x08000203, 0x08040202, 0x08040203,
  0x09000202, 0x09000203, 0x09040202, 0x09040203,
  /* for C bits (numbered as per FIPS 46) 21 23 24 26 27 28 */
  0x00000000, 0x00100000, 0x00000100, 0x00100100,
  0x00000008, 0x00100008, 0x00000108, 0x00100108,
  0x00001000, 0x00101000, 0x00001100, 0x00101100,
  0x00001008, 0x00101008, 0x00001108, 0x00101108,
  0x04000000, 0x04100000, 0x04000100, 0x04100100,
  0x04000008, 0x04100008, 0x04000108, 0x04100108,
  0x04001000, 0x04101000, 0x04001100, 0x04101100,
  0x04001008, 0x04101008, 0x04001108, 0x04101108,
  0x00020000, 0x00120000, 0x00020100, 0x00120100,
  0x00020008, 0x00120008, 0x00020108, 0x00120108,
  0x00021000, 0x00121000, 0x00021100, 0x00121100,
  0x00021008, 0x00121008, 0x00021108, 0x00121108,
  0x04020000, 0x04120000, 0x04020100, 0x04120100,
  0x04020008, 0x04120008, 0x04020108, 0x04120108,
  0x04021000, 0x04121000, 0x04021100, 0x04121100,
  0x04021008, 0x04121008, 0x04021108, 0x04121108,
  /* for D bits (numbered as per FIPS 46) 1 2 3 4 5 6 */
  0x00000000, 0x10000000, 0x00010000, 0x10010000,
  0x00000004, 0x10000004, 0x00010004, 0x10010004,
  0x20000000, 0x30000000, 0x20010000, 0x30010000,
  0x20000004, 0x30000004, 0x20010004, 0x30010004,
  0x00100000, 0x10100000, 0x00110000, 0x10110000,
  0x00100004, 0x10100004, 0x00110004, 0x10110004,
  0x20100000, 0x30100000, 0x20110000, 0x30110000,
  0x20100004, 0x30100004, 0x20110004, 0x30110004,
  0x00001000, 0x10001000, 0x00011000, 0x10011000,
  0x00001004, 0x10001004, 0x00011004, 0x10011004,
  0x20001000, 0x30001000, 0x20011000, 0x30011000,
  0x20001004, 0x30001004, 0x20011004, 0x30011004,
  0x00101000, 0x10101000, 0x00111000, 0x10111000,
  0x00101004, 0x10101004, 0x00111004, 0x10111004,
  0x20101000, 0x30101000, 0x20111000, 0x30111000,
  0x20101004, 0x30101004, 0x20111004, 0x30111004,
  /* for D bits (numbered as per FIPS 46) 8 9 11 12 13 14 */
  0x00000000, 0x08000000, 0x00000008, 0x08000008,
  0x00000400, 0x08000400, 0x00000408, 0x08000408,
  0x00020000, 0x08020000, 0x00020008, 0x08020008,
  0x00020400, 0x08020400, 0x00020408, 0x08020408,
  0x00000001, 0x08000001, 0x00000009, 0x08000009,
  0x00000401, 0x08000401, 0x00000409, 0x08000409,
  0x00020001, 0x08020001, 0x00020009, 0x08020009,
  0x00020401, 0x08020401, 0x00020409, 0x08020409,
  0x02000000, 0x0A000000, 0x02000008, 0x0A000008,
  0x02000400, 0x0A000400, 0x02000408, 0x0A000408,
  0x02020000, 0x0A020000, 0x02020008, 0x0A020008,
  0x02020400, 0x0A020400, 0x02020408, 0x0A020408,
  0x02000001, 0x0A000001, 0x02000009, 0x0A000009,
  0x02000401, 0x0A000401, 0x02000409, 0x0A000409,
  0x02020001, 0x0A020001, 0x02020009, 0x0A020009,
  0x02020401, 0x0A020401, 0x02020409, 0x0A020409,
  /* for D bits (numbered as per FIPS 46) 16 17 18 19 20 21 */
  0x00000000, 0x00000100, 0x00080000, 0x00080100,
  0x01000000, 0x01000100, 0x01080000, 0x01080100,
  0x00000010, 0x00000110, 0x00080010, 0x00080110,
  0x01000010, 0x01000110, 0x01080010, 0x01080110,
  0x00200000, 0x00200100, 0x00280000, 0x00280100,
  0x01200000, 0x01200100, 0x01280000, 0x01280100,
  0x00200010, 0x00200110, 0x00280010, 0x00280110,
  0x01200010, 0x01200110, 0x01280010, 0x01280110,
  0x00000200, 0x00000300, 0x00080200, 0x00080300,
  0x01000200, 0x01000300, 0x01080200, 0x01080300,
  0x00000210, 0x00000310, 0x00080210, 0x00080310,
  0x01000210, 0x01000310, 0x01080210, 0x01080310,
  0x00200200, 0x00200300, 0x00280200, 0x00280300,
  0x01200200, 0x01200300, 0x01280200, 0x01280300,
  0x00200210, 0x00200310, 0x00280210, 0x00280310,
  0x01200210, 0x01200310, 0x01280210, 0x01280310,
  /* for D bits (numbered as per FIPS 46) 22 23 24 25 27 28 */
  0x00000000, 0x04000000, 0x00040000, 0x04040000,
  0x00000002, 0x04000002, 0x00040002, 0x04040002,
  0x00002000, 0x04002000, 0x00042000, 0x04042000,
  0x00002002, 0x04002002, 0x00042002, 0x04042002,
  0x00000020, 0x04000020, 0x00040020, 0x04040020,
  0x00000022, 0x04000022, 0x00040022, 0x04040022,
  0x00002020, 0x04002020, 0x00042020, 0x04042020,
  0x00002022, 0x04002022, 0x00042022, 0x04042022,
  0x00000800, 0x04000800, 0x00040800, 0x04040800,
  0x00000802, 0x04000802, 0x00040802, 0x04040802,
  0x00002800, 0x04002800, 0x00042800, 0x04042800,
  0x00002802, 0x04002802, 0x00042802, 0x04042802,
  0x00000820, 0x04000820, 0x00040820, 0x04040820,
  0x00000822, 0x04000822, 0x00040822, 0x04040822,
  0x00002820, 0x04002820, 0x00042820, 0x04042820,
  0x00002822, 0x04002822, 0x00042822, 0x04042822
};

#ifdef VECT_SIZE1
#define BOX(i,n,S) u32x ((S)[(n)][(i)])
#endif

#ifdef VECT_SIZE2
#define BOX(i,n,S) u32x ((S)[(n)][(i).x], (S)[(n)][(i).y])
#endif

__device__ static void _des_crypt_keysetup (u32x c, u32x d, u32x Kc[16], u32x Kd[16], u32 s_skb[8][64])
{
  u32x tt;

  PERM_OP  (d, c, tt, 4, 0x0f0f0f0f);
  HPERM_OP (c,    tt, 2, 0xcccc0000);
  HPERM_OP (d,    tt, 2, 0xcccc0000);
  PERM_OP  (d, c, tt, 1, 0x55555555);
  PERM_OP  (c, d, tt, 8, 0x00ff00ff);
  PERM_OP  (d, c, tt, 1, 0x55555555);

  d = ((d & 0x000000ff) << 16)
    | ((d & 0x0000ff00) <<  0)
    | ((d & 0x00ff0000) >> 16)
    | ((c & 0xf0000000) >>  4);

  c = c & 0x0fffffff;

  #pragma unroll
  for (u32 i = 0; i < 16; i++)
  {
    if ((i < 2) || (i == 8) || (i == 15))
    {
      c = ((c >> 1) | (c << 27));
      d = ((d >> 1) | (d << 27));
    }
    else
    {
      c = ((c >> 2) | (c << 26));
      d = ((d >> 2) | (d << 26));
    }

    c = c & 0x0fffffff;
    d = d & 0x0fffffff;

    const u32x c00 = (c >>  0) & 0x0000003f;
    const u32x c06 = (c >>  6) & 0x00383003;
    const u32x c07 = (c >>  7) & 0x0000003c;
    const u32x c13 = (c >> 13) & 0x0000060f;
    const u32x c20 = (c >> 20) & 0x00000001;

    u32x s = BOX (((c00 >>  0) & 0xff), 0, s_skb)
            | BOX (((c06 >>  0) & 0xff)
                  |((c07 >>  0) & 0xff), 1, s_skb)
            | BOX (((c13 >>  0) & 0xff)
                  |((c06 >>  8) & 0xff), 2, s_skb)
            | BOX (((c20 >>  0) & 0xff)
                  |((c13 >>  8) & 0xff)
                  |((c06 >> 16) & 0xff), 3, s_skb);

    const u32x d00 = (d >>  0) & 0x00003c3f;
    const u32x d07 = (d >>  7) & 0x00003f03;
    const u32x d21 = (d >> 21) & 0x0000000f;
    const u32x d22 = (d >> 22) & 0x00000030;

    u32x t = BOX (((d00 >>  0) & 0xff), 4, s_skb)
            | BOX (((d07 >>  0) & 0xff)
                  |((d00 >>  8) & 0xff), 5, s_skb)
            | BOX (((d07 >>  8) & 0xff), 6, s_skb)
            | BOX (((d21 >>  0) & 0xff)
                  |((d22 >>  0) & 0xff), 7, s_skb);

    #if __CUDA_ARCH__ >= 200
    Kc[i] = __byte_perm (s, t, 0x5410);
    Kd[i] = __byte_perm (s, t, 0x7632);
    #else
    Kc[i] = ((t << 16) | (s & 0x0000ffff));
    Kd[i] = ((s >> 16) | (t & 0xffff0000));
    #endif
  }
}

__device__ static void _des_crypt_encrypt (u32x iv[2], u32 mask, u32x Kc[16], u32x Kd[16], u32 s_SPtrans[8][64])
{
  const u32 E1 = (mask >> 2) & 0x3f0;

  const u32 E0 = mask & 0x3f;

  u32x r = 0;
  u32x l = 0;

  for (u32 i = 0; i < 25; i++)
  {
    #pragma unroll
    for (u32 j = 0; j < 16; j += 2)
    {
      u32x t;
      u32x u;

      t = r ^ (r >> 16);
      u = t & E0;
      t = t & E1;
      u = u ^ (u << 16);
      u = u ^ r;
      u = u ^ Kc[j + 0];
      t = t ^ (t << 16);
      t = t ^ r;
      t = rotl32 (t, 28u);
      t = t ^ Kd[j + 0];

      l ^= BOX (((u >>  0) & 0x3f), 0, s_SPtrans)
         | BOX (((u >>  8) & 0x3f), 2, s_SPtrans)
         | BOX (((u >> 16) & 0x3f), 4, s_SPtrans)
         | BOX (((u >> 24) & 0x3f), 6, s_SPtrans)
         | BOX (((t >>  0) & 0x3f), 1, s_SPtrans)
         | BOX (((t >>  8) & 0x3f), 3, s_SPtrans)
         | BOX (((t >> 16) & 0x3f), 5, s_SPtrans)
         | BOX (((t >> 24) & 0x3f), 7, s_SPtrans);

      t = l ^ (l >> 16);
      u = t & E0;
      t = t & E1;
      u = u ^ (u << 16);
      u = u ^ l;
      u = u ^ Kc[j + 1];
      t = t ^ (t << 16);
      t = t ^ l;
      t = rotl32 (t, 28u);
      t = t ^ Kd[j + 1];

      r ^= BOX (((u >>  0) & 0x3f), 0, s_SPtrans)
         | BOX (((u >>  8) & 0x3f), 2, s_SPtrans)
         | BOX (((u >> 16) & 0x3f), 4, s_SPtrans)
         | BOX (((u >> 24) & 0x3f), 6, s_SPtrans)
         | BOX (((t >>  0) & 0x3f), 1, s_SPtrans)
         | BOX (((t >>  8) & 0x3f), 3, s_SPtrans)
         | BOX (((t >> 16) & 0x3f), 5, s_SPtrans)
         | BOX (((t >> 24) & 0x3f), 7, s_SPtrans);
    }

    u32x tt;

    tt = l;
    l  = r;
    r  = tt;
  }

  iv[0] = rotl32 (r, 31);
  iv[1] = rotl32 (l, 31);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m01500_m04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 combs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  u32x wordl0[4];

  wordl0[0] = pws[gid].i[ 0];
  wordl0[1] = pws[gid].i[ 1];
  wordl0[2] = 0;
  wordl0[3] = 0;

  u32x wordl1[4];

  wordl1[0] = 0;
  wordl1[1] = 0;
  wordl1[2] = 0;
  wordl1[3] = 0;

  u32x wordl2[4];

  wordl2[0] = 0;
  wordl2[1] = 0;
  wordl2[2] = 0;
  wordl2[3] = 0;

  u32x wordl3[4];

  wordl3[0] = 0;
  wordl3[1] = 0;
  wordl3[2] = 0;
  wordl3[3] = 0;

  const u32 pw_l_len = pws[gid].pw_len;

  if (combs_mode == COMBINATOR_MODE_BASE_RIGHT)
  {
    switch_buffer_by_offset (wordl0, wordl1, wordl2, wordl3, c_combs[0].pw_len);
  }

  /**
   * sbox, kbox
   */

  __shared__ u32 s_skb[8][64];
  __shared__ u32 s_SPtrans[8][64];

  if (lid < 64)
  {
    s_skb[0][lid] = c_skb[0][lid];
    s_skb[1][lid] = c_skb[1][lid];
    s_skb[2][lid] = c_skb[2][lid];
    s_skb[3][lid] = c_skb[3][lid];
    s_skb[4][lid] = c_skb[4][lid];
    s_skb[5][lid] = c_skb[5][lid];
    s_skb[6][lid] = c_skb[6][lid];
    s_skb[7][lid] = c_skb[7][lid];

    s_SPtrans[0][lid] = c_SPtrans[0][lid];
    s_SPtrans[1][lid] = c_SPtrans[1][lid];
    s_SPtrans[2][lid] = c_SPtrans[2][lid];
    s_SPtrans[3][lid] = c_SPtrans[3][lid];
    s_SPtrans[4][lid] = c_SPtrans[4][lid];
    s_SPtrans[5][lid] = c_SPtrans[5][lid];
    s_SPtrans[6][lid] = c_SPtrans[6][lid];
    s_SPtrans[7][lid] = c_SPtrans[7][lid];
  }

  __syncthreads ();

  if (gid >= gid_max) return;

  /**
   * salt
   */

  const u32 mask = salt_bufs[salt_pos].salt_buf[0];

  /**
   * main
   */

  for (u32 il_pos = 0; il_pos < combs_cnt; il_pos++)
  {
    const u32 pw_r_len = c_combs[il_pos].pw_len;

    u32 pw_len = pw_l_len + pw_r_len;

    pw_len = (pw_len >= 8) ? 8 : pw_len;

    u32 wordr0[4];

    wordr0[0] = c_combs[il_pos].i[0];
    wordr0[1] = c_combs[il_pos].i[1];
    wordr0[2] = 0;
    wordr0[3] = 0;

    u32 wordr1[4];

    wordr1[0] = 0;
    wordr1[1] = 0;
    wordr1[2] = 0;
    wordr1[3] = 0;

    u32 wordr2[4];

    wordr2[0] = 0;
    wordr2[1] = 0;
    wordr2[2] = 0;
    wordr2[3] = 0;

    u32 wordr3[4];

    wordr3[0] = 0;
    wordr3[1] = 0;
    wordr3[2] = 0;
    wordr3[3] = 0;

    if (combs_mode == COMBINATOR_MODE_BASE_LEFT)
    {
      switch_buffer_by_offset (wordr0, wordr1, wordr2, wordr3, pw_l_len);
    }

    u32x w0[4];

    w0[0] = wordl0[0] | wordr0[0];
    w0[1] = wordl0[1] | wordr0[1];
    w0[2] = 0;
    w0[3] = 0;

    u32x w1[4];

    w1[0] = 0;
    w1[1] = 0;
    w1[2] = 0;
    w1[3] = 0;

    u32x w2[4];

    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = 0;

    u32x data[2];

    data[0] = (w0[0] << 1) & 0xfefefefe;
    data[1] = (w0[1] << 1) & 0xfefefefe;

    u32x Kc[16];
    u32x Kd[16];

    _des_crypt_keysetup (data[0], data[1], Kc, Kd, s_skb);

    u32x iv[2];

    _des_crypt_encrypt (iv, mask, Kc, Kd, s_SPtrans);

    const u32x r0 = iv[0];
    const u32x r1 = iv[1];
    const u32x r2 = 0;
    const u32x r3 = 0;

    #include VECT_COMPARE_M
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m01500_m08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 combs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m01500_m16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 combs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m01500_s04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 combs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * modifier
   */

  const u32 lid = threadIdx.x;

  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  u32x wordl0[4];

  wordl0[0] = pws[gid].i[ 0];
  wordl0[1] = pws[gid].i[ 1];
  wordl0[2] = 0;
  wordl0[3] = 0;

  u32x wordl1[4];

  wordl1[0] = 0;
  wordl1[1] = 0;
  wordl1[2] = 0;
  wordl1[3] = 0;

  u32x wordl2[4];

  wordl2[0] = 0;
  wordl2[1] = 0;
  wordl2[2] = 0;
  wordl2[3] = 0;

  u32x wordl3[4];

  wordl3[0] = 0;
  wordl3[1] = 0;
  wordl3[2] = 0;
  wordl3[3] = 0;

  const u32 pw_l_len = pws[gid].pw_len;

  if (combs_mode == COMBINATOR_MODE_BASE_RIGHT)
  {
    switch_buffer_by_offset (wordl0, wordl1, wordl2, wordl3, c_combs[0].pw_len);
  }

  /**
   * sbox, kbox
   */

  __shared__ u32 s_skb[8][64];
  __shared__ u32 s_SPtrans[8][64];

  if (lid < 64)
  {
    s_skb[0][lid] = c_skb[0][lid];
    s_skb[1][lid] = c_skb[1][lid];
    s_skb[2][lid] = c_skb[2][lid];
    s_skb[3][lid] = c_skb[3][lid];
    s_skb[4][lid] = c_skb[4][lid];
    s_skb[5][lid] = c_skb[5][lid];
    s_skb[6][lid] = c_skb[6][lid];
    s_skb[7][lid] = c_skb[7][lid];

    s_SPtrans[0][lid] = c_SPtrans[0][lid];
    s_SPtrans[1][lid] = c_SPtrans[1][lid];
    s_SPtrans[2][lid] = c_SPtrans[2][lid];
    s_SPtrans[3][lid] = c_SPtrans[3][lid];
    s_SPtrans[4][lid] = c_SPtrans[4][lid];
    s_SPtrans[5][lid] = c_SPtrans[5][lid];
    s_SPtrans[6][lid] = c_SPtrans[6][lid];
    s_SPtrans[7][lid] = c_SPtrans[7][lid];
  }

  __syncthreads ();

  if (gid >= gid_max) return;

  /**
   * salt
   */

  const u32 mask = salt_bufs[salt_pos].salt_buf[0];

  /**
   * digest
   */

  const u32 search[4] =
  {
    digests_buf[digests_offset].digest_buf[DGST_R0],
    digests_buf[digests_offset].digest_buf[DGST_R1],
    digests_buf[digests_offset].digest_buf[DGST_R2],
    digests_buf[digests_offset].digest_buf[DGST_R3]
  };

  /**
   * main
   */

  for (u32 il_pos = 0; il_pos < combs_cnt; il_pos++)
  {
    const u32 pw_r_len = c_combs[il_pos].pw_len;

    u32 pw_len = pw_l_len + pw_r_len;

    pw_len = (pw_len >= 8) ? 8 : pw_len;

    u32 wordr0[4];

    wordr0[0] = c_combs[il_pos].i[0];
    wordr0[1] = c_combs[il_pos].i[1];
    wordr0[2] = 0;
    wordr0[3] = 0;

    u32 wordr1[4];

    wordr1[0] = 0;
    wordr1[1] = 0;
    wordr1[2] = 0;
    wordr1[3] = 0;

    u32 wordr2[4];

    wordr2[0] = 0;
    wordr2[1] = 0;
    wordr2[2] = 0;
    wordr2[3] = 0;

    u32 wordr3[4];

    wordr3[0] = 0;
    wordr3[1] = 0;
    wordr3[2] = 0;
    wordr3[3] = 0;

    if (combs_mode == COMBINATOR_MODE_BASE_LEFT)
    {
      switch_buffer_by_offset (wordr0, wordr1, wordr2, wordr3, pw_l_len);
    }

    u32x w0[4];

    w0[0] = wordl0[0] | wordr0[0];
    w0[1] = wordl0[1] | wordr0[1];
    w0[2] = 0;
    w0[3] = 0;

    u32x w1[4];

    w1[0] = 0;
    w1[1] = 0;
    w1[2] = 0;
    w1[3] = 0;

    u32x w2[4];

    w2[0] = 0;
    w2[1] = 0;
    w2[2] = 0;
    w2[3] = 0;

    u32x w3[4];

    w3[0] = 0;
    w3[1] = 0;
    w3[2] = 0;
    w3[3] = 0;

    u32x data[2];

    data[0] = (w0[0] << 1) & 0xfefefefe;
    data[1] = (w0[1] << 1) & 0xfefefefe;

    u32x Kc[16];
    u32x Kd[16];

    _des_crypt_keysetup (data[0], data[1], Kc, Kd, s_skb);

    u32x iv[2];

    _des_crypt_encrypt (iv, mask, Kc, Kd, s_SPtrans);

    const u32x r0 = iv[0];
    const u32x r1 = iv[1];
    const u32x r2 = 0;
    const u32x r3 = 0;

    #include VECT_COMPARE_S
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m01500_s08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 combs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}

extern "C" __global__ void __launch_bounds__ (256, 1) m01500_s16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const bf_t *bfs_buf, const void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 combs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
}
