#include "hip/hip_runtime.h"
/**
 * Author......: Jens Steube <jens.steube@gmail.com>
 * License.....: MIT
 */

#define _SIPHASH_
#define _SCALAR_

#include "include/constants.h"
#include "include/kernel_vendor.h"

#ifdef  VLIW1
#define VECT_SIZE1
#endif

#ifdef  VLIW2
#define VECT_SIZE1
#endif

#define DGST_R0 0
#define DGST_R1 1
#define DGST_R2 2
#define DGST_R3 3

#include "include/kernel_functions.c"
#include "types_nv.c"
#include "common_nv.c"

#ifdef  VECT_SIZE1
#define VECT_COMPARE_S "check_single_vect1_comp4_warp.c"
#define VECT_COMPARE_M "check_multi_vect1_comp4_warp.c"
#endif

#ifdef  VECT_SIZE2
#define VECT_COMPARE_S "check_single_vect2_comp4_warp.c"
#define VECT_COMPARE_M "check_multi_vect2_comp4_warp.c"
#endif

#ifdef  VECT_SIZE4
#define VECT_COMPARE_S "check_single_vect4_comp4_warp.c"
#define VECT_COMPARE_M "check_multi_vect4_comp4_warp.c"
#endif

#define SIPROUND(v0,v1,v2,v3) \
  (v0) += (v1);               \
  (v1)  = rotl64 ((v1), 13);  \
  (v1) ^= (v0);               \
  (v0)  = rotl64 ((v0), 32);  \
  (v2) += (v3);               \
  (v3)  = rotl64 ((v3), 16);  \
  (v3) ^= (v2);               \
  (v0) += (v3);               \
  (v3)  = rotl64 ((v3), 21);  \
  (v3) ^= (v0);               \
  (v2) += (v1);               \
  (v1)  = rotl64 ((v1), 17);  \
  (v1) ^= (v2);               \
  (v2)  = rotl64 ((v2), 32);

__device__ __constant__ u32x c_bfs[1024];

__device__ static void m10100m (u32 w[16], const u32 pw_len, const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x * words_buf_r, void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset)
{
  /**
   * modifier
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;
  const u32 lid = threadIdx.x;

  /**
   * base
   */

  u64 v0p = SIPHASHM_0;
  u64 v1p = SIPHASHM_1;
  u64 v2p = SIPHASHM_2;
  u64 v3p = SIPHASHM_3;

  v0p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[1], salt_bufs[salt_pos].salt_buf[0]);
  v1p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[3], salt_bufs[salt_pos].salt_buf[2]);
  v2p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[1], salt_bufs[salt_pos].salt_buf[0]);
  v3p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[3], salt_bufs[salt_pos].salt_buf[2]);

  u64 *w_ptr = (u64 *) w;

  w_ptr[pw_len / 8] |= (u64) pw_len << 56;

  /**
   * loop
   */

  const u32 bf_loops = ceil ((float) bfs_cnt / VECT_DIV);

  u32x w0l = w[0];

  for (u32 il_pos = 0; il_pos < bf_loops; il_pos++)
  {
    const u32x w0r = c_bfs[il_pos];

    const u32x w0 = w0l | w0r;

    u64x v0 = v0p;
    u64x v1 = v1p;
    u64x v2 = v2p;
    u64x v3 = v3p;

    u64x m = hl32_to_64 (w[1], w0);

    v3 ^= m;

    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);

    v0 ^= m;

    int i;
    int j;

    for (i = 8, j = 2; i <= pw_len; i += 8, j += 2)
    {
      m = hl32_to_64 (w[j + 1], w[j + 0]);

      v3 ^= m;

      SIPROUND (v0, v1, v2, v3);
      SIPROUND (v0, v1, v2, v3);

      v0 ^= m;
    }

    v2 ^= 0xff;

    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);

    const u64x v = v0 ^ v1 ^ v2 ^ v3;

    const u32x a = l32_from_64 (v);
    const u32x b = h32_from_64 (v);

    const u32x r0 = a;
    const u32x r1 = b;
    const u32x r2 = 0;
    const u32x r3 = 0;

    #include VECT_COMPARE_M
  }
}

__device__ static void m10100s (u32 w[16], const u32 pw_len, const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x * words_buf_r, void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset)
{
  /**
   * modifier
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;
  const u32 lid = threadIdx.x;

  /**
   * base
   */

  u64 v0p = SIPHASHM_0;
  u64 v1p = SIPHASHM_1;
  u64 v2p = SIPHASHM_2;
  u64 v3p = SIPHASHM_3;

  v0p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[1], salt_bufs[salt_pos].salt_buf[0]);
  v1p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[3], salt_bufs[salt_pos].salt_buf[2]);
  v2p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[1], salt_bufs[salt_pos].salt_buf[0]);
  v3p ^= hl32_to_64 (salt_bufs[salt_pos].salt_buf[3], salt_bufs[salt_pos].salt_buf[2]);

  u64 *w_ptr = (u64 *) w;

  w_ptr[pw_len / 8] |= (u64) pw_len << 56;

  /**
   * digest
   */

  const u32 search[4] =
  {
    digests_buf[digests_offset].digest_buf[DGST_R0],
    digests_buf[digests_offset].digest_buf[DGST_R1],
    digests_buf[digests_offset].digest_buf[DGST_R2],
    digests_buf[digests_offset].digest_buf[DGST_R3]
  };

  /**
   * loop
   */

  const u32 bf_loops = ceil ((float) bfs_cnt / VECT_DIV);

  u32x w0l = w[0];

  for (u32 il_pos = 0; il_pos < bf_loops; il_pos++)
  {
    const u32x w0r = c_bfs[il_pos];

    const u32x w0 = w0l | w0r;

    u64x v0 = v0p;
    u64x v1 = v1p;
    u64x v2 = v2p;
    u64x v3 = v3p;

    u64x m = hl32_to_64 (w[1], w0);

    v3 ^= m;

    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);

    v0 ^= m;

    int i;
    int j;

    for (i = 8, j = 2; i <= pw_len; i += 8, j += 2)
    {
      m = hl32_to_64 (w[j + 1], w[j + 0]);

      v3 ^= m;

      SIPROUND (v0, v1, v2, v3);
      SIPROUND (v0, v1, v2, v3);

      v0 ^= m;
    }

    v2 ^= 0xff;

    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);
    SIPROUND (v0, v1, v2, v3);

    const u64x v = v0 ^ v1 ^ v2 ^ v3;

    const u32x a = l32_from_64 (v);
    const u32x b = h32_from_64 (v);

    const u32x r0 = a;
    const u32x r1 = b;
    const u32x r2 = 0;
    const u32x r3 = 0;

    #include VECT_COMPARE_S
  }
}

extern "C" __global__ void __launch_bounds__ (256, 1) m10100_m04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r,void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = 0;
  w[ 5] = 0;
  w[ 6] = 0;
  w[ 7] = 0;
  w[ 8] = 0;
  w[ 9] = 0;
  w[10] = 0;
  w[11] = 0;
  w[12] = 0;
  w[13] = 0;
  w[14] = pws[gid].i[14];
  w[15] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m10100m (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m10100_m08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r,void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = pws[gid].i[ 4];
  w[ 5] = pws[gid].i[ 5];
  w[ 6] = pws[gid].i[ 6];
  w[ 7] = pws[gid].i[ 7];
  w[ 8] = 0;
  w[ 9] = 0;
  w[10] = 0;
  w[11] = 0;
  w[12] = 0;
  w[13] = 0;
  w[14] = pws[gid].i[14];
  w[15] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m10100m (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m10100_m16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r,void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = pws[gid].i[ 4];
  w[ 5] = pws[gid].i[ 5];
  w[ 6] = pws[gid].i[ 6];
  w[ 7] = pws[gid].i[ 7];
  w[ 8] = pws[gid].i[ 8];
  w[ 9] = pws[gid].i[ 9];
  w[10] = pws[gid].i[10];
  w[11] = pws[gid].i[11];
  w[12] = pws[gid].i[12];
  w[13] = pws[gid].i[13];
  w[14] = pws[gid].i[14];
  w[15] = pws[gid].i[15];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m10100m (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m10100_s04 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r,void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = 0;
  w[ 5] = 0;
  w[ 6] = 0;
  w[ 7] = 0;
  w[ 8] = 0;
  w[ 9] = 0;
  w[10] = 0;
  w[11] = 0;
  w[12] = 0;
  w[13] = 0;
  w[14] = pws[gid].i[14];
  w[15] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m10100s (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m10100_s08 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r,void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = pws[gid].i[ 4];
  w[ 5] = pws[gid].i[ 5];
  w[ 6] = pws[gid].i[ 6];
  w[ 7] = pws[gid].i[ 7];
  w[ 8] = 0;
  w[ 9] = 0;
  w[10] = 0;
  w[11] = 0;
  w[12] = 0;
  w[13] = 0;
  w[14] = pws[gid].i[14];
  w[15] = 0;

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m10100s (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}

extern "C" __global__ void __launch_bounds__ (256, 1) m10100_s16 (const pw_t *pws, const gpu_rule_t *rules_buf, const comb_t *combs_buf, const u32x *words_buf_r,void *tmps, void *hooks, const u32 *bitmaps_buf_s1_a, const u32 *bitmaps_buf_s1_b, const u32 *bitmaps_buf_s1_c, const u32 *bitmaps_buf_s1_d, const u32 *bitmaps_buf_s2_a, const u32 *bitmaps_buf_s2_b, const u32 *bitmaps_buf_s2_c, const u32 *bitmaps_buf_s2_d, plain_t *plains_buf, const digest_t *digests_buf, u32 *hashes_shown, const salt_t *salt_bufs, const void *esalt_bufs, u32 *d_return_buf, u32 *d_scryptV_buf, const u32 bitmap_mask, const u32 bitmap_shift1, const u32 bitmap_shift2, const u32 salt_pos, const u32 loop_pos, const u32 loop_cnt, const u32 bfs_cnt, const u32 digests_cnt, const u32 digests_offset, const u32 combs_mode, const u32 gid_max)
{
  /**
   * base
   */

  const u32 gid = (blockIdx.x * blockDim.x) + threadIdx.x;

  if (gid >= gid_max) return;

  u32 w[16];

  w[ 0] = pws[gid].i[ 0];
  w[ 1] = pws[gid].i[ 1];
  w[ 2] = pws[gid].i[ 2];
  w[ 3] = pws[gid].i[ 3];
  w[ 4] = pws[gid].i[ 4];
  w[ 5] = pws[gid].i[ 5];
  w[ 6] = pws[gid].i[ 6];
  w[ 7] = pws[gid].i[ 7];
  w[ 8] = pws[gid].i[ 8];
  w[ 9] = pws[gid].i[ 9];
  w[10] = pws[gid].i[10];
  w[11] = pws[gid].i[11];
  w[12] = pws[gid].i[12];
  w[13] = pws[gid].i[13];
  w[14] = pws[gid].i[14];
  w[15] = pws[gid].i[15];

  const u32 pw_len = pws[gid].pw_len;

  /**
   * main
   */

  m10100s (w, pw_len, pws, rules_buf, combs_buf, words_buf_r, tmps, hooks, bitmaps_buf_s1_a, bitmaps_buf_s1_b, bitmaps_buf_s1_c, bitmaps_buf_s1_d, bitmaps_buf_s2_a, bitmaps_buf_s2_b, bitmaps_buf_s2_c, bitmaps_buf_s2_d, plains_buf, digests_buf, hashes_shown, salt_bufs, esalt_bufs, d_return_buf, d_scryptV_buf, bitmap_mask, bitmap_shift1, bitmap_shift2, salt_pos, loop_pos, loop_cnt, bfs_cnt, digests_cnt, digests_offset);
}
